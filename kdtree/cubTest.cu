#include "hip/hip_runtime.h"
// #include <hipcub/hipcub.hpp>
// #include <iostream>
// #include <stdio.h>      /* printf, scanf, puts, NULL */
// #include <stdlib.h>     /* srand, rand */
// #include <time.h>       /* time */

// int main(){
//     int  num_items=7;          // e.g., 7
//     int  num_segments=2;       // e.g., 3
    
//     int offsets[3] = {1, 3, 6};
//     int  *d_offsets;         // e.g., [0, 3, 3, 7]

//     int keys_in[7] = {8, 6, 7, 5, 3, 0, 9};
//     int  *d_keys_in;         // e.g., [8, 6, 7, 5, 3, 0, 9]

//     int keys_out[7];
//     int  *d_keys_out;        // e.g., [-, -, -, -, -, -, -]

//     int values_in[7] = {0, 1, 2, 3, 4, 5, 6};
//     int  *d_values_in;       // e.g., [0, 1, 2, 3, 4, 5, 6]

//     int values_out[7];
//     int  *d_values_out;      // e.g., [-, -, -, -, -, -, -]
    
//     void     *d_temp_storage = NULL;
//     size_t   temp_storage_bytes = 0;

//     hipMalloc((void**) &d_offsets, 4*sizeof(int));
//     hipMalloc((void**) &d_keys_in,  num_items*sizeof(int));
//     hipMalloc((void**) &d_keys_out,  num_items*sizeof(int));
//     hipMalloc((void**) &d_values_in, num_items*sizeof(int));
//     hipMalloc((void**) &d_values_out,  num_items*sizeof(int));

//     hipMemcpy(d_offsets, offsets, 3*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_keys_in, keys_in, num_items*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_keys_out, keys_out, num_items*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_values_in, values_in, num_items*sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_values_out, values_out, num_items*sizeof(int), hipMemcpyHostToDevice);

//     hipDeviceSynchronize();
//     for(int i=0; i<num_items; ++i){
//         printf("%d ", keys_in[i]);
//     }
//     printf("\n");

//     for(int i=0; i<num_items; ++i){
//         printf("%d ", values_in[i]);
//     }
//     printf("\n");
//     printf("\n");

//     hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
//         d_keys_in, d_keys_out, d_values_in, d_values_out,
//         num_items, num_segments, d_offsets, d_offsets + 1);

//     // Allocate temporary storage
//     hipMalloc(&d_temp_storage, temp_storage_bytes);
//     printf("temp_storage_array: %zu\n", temp_storage_bytes);
//     // Run sorting operation
//     hipDeviceSynchronize();
    
//     hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
//         d_keys_in, d_keys_out, d_values_in, d_values_out,
//         num_items, num_segments, d_offsets, d_offsets + 1);
//     // d_keys_out            <-- [6, 7, 8, 0, 3, 5, 9]
//     // d_values_out          <-- [1, 2, 0, 5, 4, 3, 6]

//     hipDeviceSynchronize();


//     hipMemcpy(keys_out, d_keys_out, num_items*sizeof(int), hipMemcpyDeviceToHost);
//     hipMemcpy(values_out, d_values_out, num_items*sizeof(int), hipMemcpyDeviceToHost);
//     hipDeviceSynchronize();

//     for(int i=0; i<num_items; ++i){
//         printf("%d ", keys_out[i]);
//     }
//     printf("\n");

//     for(int i=0; i<num_items; ++i){
//         printf("%d ", values_out[i]);
//     }
//     printf("\n");
//     printf("\n");
// }

#include <hipcub/hipcub.hpp>   // or equivalently <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

int main(){
    // Declare, allocate, and initialize device-accessible pointers for input and output
    int  num_items=7;      // e.g., 7
    int  *d_in;          // e.g., [8, 6, 7, 5, 3, 0, 9]
    int  *d_out;         // e.g., [-]
    
    int *in = (int*)malloc(7*sizeof(int));
    int *out = (int*)malloc(sizeof(int));
    
    for(int i=0; i<7; ++i){
        in[i] = i;
    }
    hipMalloc((void**) &d_in, 7*sizeof(int));
    hipMalloc((void**) &d_out, sizeof(int));
    hipMemcpy(d_in, in, 7*sizeof(int), hipMemcpyHostToDevice);
    
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d", out);
}