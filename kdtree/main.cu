#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "TLR_Matrix.cuh"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "SVD.cuh"
#include "config.h"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include "hipblas.h"
#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>

#define BLOCK_SIZE 32
#define PRINT_OUTPUT 0
using namespace std;

// TODO: create a struct for the tiled matrix that has u_tiled, v_tiled, k, k_scan
// TODO: generate pointcloud and copy values of the pointcloud to ptr on GPU
// TODO: fix makefile so main.cu depends on helperKerlens.cuh
// TODO: make sure that everything that is malloced is freed

int main(int argc, char *argv[]){
    hipEvent_t startCode, stopCode;
    hipEventCreate(&startCode);
    hipEventCreate(&stopCode);
    hipEventRecord(startCode);

    Config config = parseArgs(argc,argv);
    printf("n: %d\n", config.n);
    printf("bucket size: %d\n", config.bucket_size);
    printf("epsilon: %f\n", config.epsilon);
    printf("dim: %d\n", config.dim);

    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    timer_arr[0] = (float)config.n;
    timer_arr[1] = (float)config.bucket_size;
    timer_arr[2] = (float)config.dim;
    timer_arr[3] = (float)config.epsilon;

    // Create point cloud
    PointCloud<H2Opus_Real> pt_cloud(config.dim, (size_t)config.n);
    generateGrid<H2Opus_Real>(pt_cloud, config.n);

    #if PRINT_OUTPUT
    printf("created point cloud\n");
    for(int i=0; i<config.n; ++i){
        for(int j=0; j<config.dim;++j){
            printf("%lf ", pt_cloud.pts[j][i]);
        }
        printf("\n");
    }
    printf("\n\n");
    #endif

    H2Opus_Real *dataset;
    dataset = (H2Opus_Real*)malloc(config.n*config.dim*(uint64_t)sizeof(H2Opus_Real));
    assert(dataset != NULL);

    // TODO: move this to a kernel
    for (unsigned long long i = 0; i < config.dim; ++i){
        for(unsigned long long j = 0; j < config.n; ++j){
            dataset[i*config.n+j] = pt_cloud.getDataPoint((size_t)j, (int)i);
        }
    }

    H2Opus_Real *d_dataset;
    hipError_t cudaErr = hipMalloc((void**) &d_dataset, config.n*config.dim*(uint64_t)sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMemcpy(d_dataset, dataset, config.n*config.dim*(uint64_t)sizeof(H2Opus_Real*), hipMemcpyHostToDevice);
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    free(dataset);

    uint64_t num_segments = 1;
    uint64_t num_segments_reduce = num_segments*config.dim;
    uint64_t segment_size = upper_power_of_two(config.n);

    int *d_offsets_sort;         // e.g., [0, 3, 3, 7]
    int *d_offsets_reduce;
    H2Opus_Real *d_keys_in;         // e.g., [8, 6, 7, 5, 3, 0, 9]
    H2Opus_Real *d_keys_out;        // e.g., [-, -, -, -, -, -, -]
    int  *d_values_in;       // e.g., [0, 1, 2, 3, 4, 5, 6]
    int  *d_values_out;      // e.g., [-, -, -, -, -, -, -]
    int *d_curr_dim;
    H2Opus_Real *d_reduce_in;
    H2Opus_Real *d_reduce_min_out;
    H2Opus_Real *d_reduce_max_out;
    int *d_temp;
    H2Opus_Real *d_span;
    int* d_span_offsets;
    hipcub::KeyValuePair<int, H2Opus_Real> *d_span_reduce_out;

    bool workDone= false;
    bool* d_workDone;
    uint64_t* d_bit_vector;
    short int* d_popc_bit_vector;
    short int* d_popc_scan;
    unsigned int* d_new_num_segments;
    unsigned int* new_num_segments;
    int max_num_segments;

    int* A;
    int* B;
    int* d_bin_search_output;
    int* d_thrust_v_bin_search_output;
    int* d_input_search;
    int* d_aux_offsets_sort;

    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        max_num_segments = 1<<(getMaxSegmentSize(config.n, config.bucket_size).second);
    } else {
        max_num_segments = (config.n+config.bucket_size-1)/config.bucket_size;
    }
    printf("max num segments: %d\n", max_num_segments);

    unsigned int largest_segment_size = config.n;

    // TODO: fix memory allocated
    cudaErr = hipMalloc((void**) &d_offsets_sort, (max_num_segments + 1)*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_offsets_reduce, (long long)((max_num_segments*config.dim + 1)*(long long)sizeof(int)));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_keys_in, config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_keys_out, config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_values_in, config.n*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_values_out, config.n*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_curr_dim, (max_num_segments + 1)*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_reduce_in, (long long)config.n*(long long)config.dim*(long long)sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_reduce_min_out, (long long)((max_num_segments+1)*config.dim)*(long long)sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_reduce_max_out, (long long)((max_num_segments+1)*config.dim)*(long long)sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_span, (long long)((max_num_segments+1)*config.dim)*(long long)sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_span_offsets, (max_num_segments + 1)*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_span_reduce_out, (max_num_segments+1)*sizeof(hipcub::KeyValuePair<int, H2Opus_Real>));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    if(config.div_method == DIVIDE_IN_HALF){
        cudaErr = hipMalloc((void**) &d_bit_vector, (((max_num_segments+1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8)) *sizeof(uint64_t));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_popc_bit_vector, (((max_num_segments + 1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8))*sizeof(short int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_popc_scan, (((max_num_segments + 1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8))*sizeof(short int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_new_num_segments, sizeof(unsigned int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_workDone, sizeof(bool));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        new_num_segments = (unsigned int*)malloc(sizeof(unsigned int));
    }

    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        cudaErr = hipMalloc((void**) &d_aux_offsets_sort, (max_num_segments + 1) * sizeof(int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &A, (max_num_segments + 1)*sizeof(int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &B, config.n*sizeof(int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_bin_search_output, config.n*sizeof(int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_input_search, config.n*sizeof(int));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    }

    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (config.n+numThreadsPerBlock-1)/numThreadsPerBlock;

    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        initializeArrays<<<numBlocks, numThreadsPerBlock>>>(config.n, config.dim, d_values_in, d_curr_dim, d_offsets_sort, d_offsets_reduce, d_input_search, max_num_segments);
    } else {
        initializeArrays<<<numBlocks, numThreadsPerBlock>>>(config.n, d_values_in, d_curr_dim, max_num_segments);
    }
    hipDeviceSynchronize();

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipEvent_t startKDtree, stopKDtree;
    hipEventCreate(&startKDtree);
    hipEventCreate(&stopKDtree);
    hipEventRecord(startKDtree);

    unsigned int iteration = 0;

    // TODO: fix the while loop if statement
    #if 0
    if(config.div_method == DIVIDE_IN_HALF){
        while(!workDone)
    } else if (config.div_method == POWER_OF_TWO_ON_LEFT) {
        while(segment_size > config.bucket_size)
    } else {
        while(largest_segment_size > config.bucket_size)
    }
    #endif

    while(segment_size > config.bucket_size)
    {
        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+1+numThreadsPerBlock-1)/numThreadsPerBlock;
        if(config.div_method == POWER_OF_TWO_ON_LEFT){
            fillOffsets<<<numBlocks, numThreadsPerBlock>>>(config.n, config.dim, num_segments, segment_size, d_offsets_sort, d_offsets_reduce);
        }
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (long long)((long long)config.n*(long long)config.dim + numThreadsPerBlock-1)/numThreadsPerBlock;

        fillReductionArray<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, d_dataset, d_values_in, d_reduce_in);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        findSpan<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, num_segments, d_reduce_min_out, d_reduce_max_out, d_span, d_span_offsets);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        if(config.div_method == DIVIDE_IN_HALF){
            fillCurrDim<<<numBlocks, numThreadsPerBlock>>> (config.n, num_segments, d_curr_dim, d_span_reduce_out, d_bit_vector);
        } else {
            fillCurrDim<<<numBlocks, numThreadsPerBlock>>> (config.n, num_segments, d_curr_dim, d_span_reduce_out);
        }
        hipDeviceSynchronize();

        // fill keys_in array
        numThreadsPerBlock = 1024;
        numBlocks = (config.n+numThreadsPerBlock-1)/numThreadsPerBlock;

        if(config.div_method != POWER_OF_TWO_ON_LEFT){
            thrust::device_ptr<int> A = thrust::device_pointer_cast((int *)d_offsets_sort), B = thrust::device_pointer_cast((int *)d_input_search);
            thrust::device_vector<int> d_bin_search_output(config.n);
            thrust::upper_bound(A, A + num_segments + 1, B, B + config.n, d_bin_search_output.begin(), thrust::less<int>());
            d_thrust_v_bin_search_output = thrust::raw_pointer_cast(&d_bin_search_output[0]);
            fillKeysIn<<<numBlocks, numThreadsPerBlock>>> (config.n, d_keys_in, d_curr_dim, d_values_in, d_dataset, d_offsets_sort, d_thrust_v_bin_search_output);
        } else {
            fillKeysIn<<<numBlocks, numThreadsPerBlock>>> (config.n, segment_size, d_keys_in, d_curr_dim, d_values_in, d_dataset);
        }
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        config.n, num_segments, d_offsets_sort, d_offsets_sort + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        config.n, num_segments, d_offsets_sort, d_offsets_sort + 1);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        d_temp = d_values_in;
        d_values_in = d_values_out;
        d_values_out = d_temp;
        ++iteration;

        // TODO: fix cuda-memcheck bug in divide_in_half
        if(config.div_method == DIVIDE_IN_HALF){
            numThreadsPerBlock = 1024;
            numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
            fillBitVector<<<numBlocks, numThreadsPerBlock>>>(num_segments, d_bit_vector, d_offsets_sort, config.bucket_size);
            hipDeviceSynchronize();

            unsigned int num_threads = (num_segments + sizeof(uint64_t)*8 - 1)/(sizeof(uint64_t)*8);
            numThreadsPerBlock = 1024;
            numBlocks = (num_threads + numThreadsPerBlock-1)/numThreadsPerBlock;
            fillPopCount<<<numBlocks, numThreadsPerBlock>>>(num_threads, d_bit_vector, d_popc_bit_vector);
            hipDeviceSynchronize();

            d_temp_storage = NULL;
            temp_storage_bytes = 0;
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_popc_bit_vector, d_popc_scan, num_segments);
            // Allocate temporary storage
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            // Run exclusive prefix sum
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_popc_bit_vector, d_popc_scan, num_segments);
            hipFree(d_temp_storage);

            numThreadsPerBlock = 1024;
            numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
            fillOffsetsSort<<<numBlocks, numThreadsPerBlock>>>(config.n, config.dim, num_segments, d_offsets_sort, d_aux_offsets_sort, d_bit_vector, d_popc_scan, d_new_num_segments, d_workDone, config.bucket_size);
            hipDeviceSynchronize();
            hipMemcpy(new_num_segments, d_new_num_segments, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&workDone, d_workDone, sizeof(bool), hipMemcpyDeviceToHost);
            num_segments = *new_num_segments;
            d_temp = d_aux_offsets_sort;
            d_aux_offsets_sort = d_offsets_sort;
            d_offsets_sort = d_temp;

            if(workDone){
                break;
            }

            numThreadsPerBlock = 1024;
            numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
            fillOffsetsReduce<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, num_segments, d_offsets_sort, d_offsets_reduce);
            hipDeviceSynchronize();

        } else if(config.div_method == POWER_OF_TWO_ON_LEFT){
            segment_size /= 2;
            num_segments = (config.n+segment_size-1)/segment_size;

        } else if(config.div_method == FULL_TREE){
            numThreadsPerBlock = 1024;
            numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
            fillOffsetsSort<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, num_segments, d_offsets_sort, d_aux_offsets_sort);
            hipDeviceSynchronize();

            d_temp = d_aux_offsets_sort;
            d_aux_offsets_sort = d_offsets_sort;
            d_offsets_sort = d_temp;
            num_segments *= 2;

            numThreadsPerBlock = 1024;
            numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
            fillOffsetsReduce<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, num_segments, d_offsets_sort, d_offsets_reduce);
            hipDeviceSynchronize();

            ++largest_segment_size;
            largest_segment_size /= 2;
        }

        num_segments_reduce = num_segments*config.dim;
    }

    hipEventRecord(stopKDtree);
    hipEventSynchronize(stopKDtree);
    float KDtree_time = 0;
    hipEventElapsedTime(&KDtree_time, startKDtree, stopKDtree);
    timer_arr[4] = KDtree_time;
    hipEventDestroy(startKDtree);
    hipEventDestroy(stopKDtree);

    #if 0
    int *index_map = (int*)malloc(config.n*sizeof(int));
    hipMemcpy(index_map, d_values_in, config.n*sizeof(int), hipMemcpyDeviceToHost);
    
    FILE *fp;
    fp = fopen("results/pointcloud.csv", "a");// "w" means that we are going to write on this file
    fprintf(fp, "bucket size: %d, n: %d, num segments: %d,\n", config.bucket_size, config.n, num_segments);
    for(int i=0; i<config.n; ++i){
        for(int j=0; j<config.dim; ++j){
            fprintf(fp, "%lf, ", pt_cloud.pts[j][index_map[i]]);
        }
        fprintf(fp, "\n");
    }
    fclose(fp); //Don't forget to close the file when finished    
    free(index_map);
    #endif

    #if 0
    printf("index max\n");
    for(int i=0; i<config.n; ++i){
        printf("%d ", index_map[i]);
    }
    printf("\n");
    #endif

    if(config.div_method == POWER_OF_TWO_ON_LEFT){
        printf("num segments :%d\n", num_segments);
        printf("segment size :%d\n", segment_size);
        fillOffsets<<<numBlocks, numThreadsPerBlock>>>(config.n, config.dim, num_segments, segment_size, d_offsets_sort, d_offsets_reduce);
        hipDeviceSynchronize();
    }

    hipFree(d_offsets_reduce);
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_values_out);
    hipFree(d_curr_dim);
    hipFree(d_reduce_in);
    hipFree(d_reduce_min_out);
    hipFree(d_reduce_max_out);
    hipFree(d_span_reduce_out);
    hipFree(d_span);
    hipFree(d_span_offsets);
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        hipFree(d_aux_offsets_sort);
        hipFree(A);
        hipFree(B);
        hipFree(d_bin_search_output);
        hipFree(d_input_search);
    }
    if(config.div_method == DIVIDE_IN_HALF){
        free(new_num_segments);
        hipFree(d_bit_vector);
        hipFree(d_popc_bit_vector);
        hipFree(d_popc_scan);
        hipFree(d_new_num_segments);
        hipFree(d_workDone);
    }

    uint64_t maxSegmentSize;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        maxSegmentSize = getMaxSegmentSize(config.n, config.bucket_size).first;
    } else {
        maxSegmentSize = config.bucket_size;
    }
    printf("max segment size: %lu\n", maxSegmentSize);

    H2Opus_Real* d_input_matrix_segmented;
    H2Opus_Real* input_matrix_segmented = (H2Opus_Real*)malloc(maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    printf("mem allocated to input matrix: %lu\n", maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    cudaErr = hipMalloc((void**) &d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    H2Opus_Real *h_S = (H2Opus_Real *)malloc(maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    H2Opus_Real *h_U = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    H2Opus_Real *h_V = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));

    H2Opus_Real* d_S;
    H2Opus_Real* d_U;
    H2Opus_Real* d_V;
    cudaErr = hipMalloc((void**) &d_S, maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_U, maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_V, maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    int* d_scan_K_segmented;
    cudaErr = hipMalloc((void**) &d_scan_K_segmented, num_segments*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    H2Opus_Real** d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    H2Opus_Real** d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    TLR_Matrix matrix;
    cudaErr = hipMalloc((void**) &matrix.blockRanks, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix.diagonal, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    int k_sum = 0;

    hipEvent_t startGenerateInputMatrix, stopGenerateInputMatrix;
    hipEventCreate(&startGenerateInputMatrix);
    hipEventCreate(&stopGenerateInputMatrix);
    hipEventRecord(startGenerateInputMatrix);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        dim3 m_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
        dim3 m_numBlocks(1, num_segments);

        generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(config.n, num_segments, maxSegmentSize, config.dim, d_values_in, d_input_matrix_segmented, d_dataset, d_offsets_sort, segment, matrix.diagonal);
        hipDeviceSynchronize();

        hipMemcpy(input_matrix_segmented, d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);

        #if PRINT_OUTPUT
        printf("input matrix\n");
        for(unsigned int i=0; i<num_segments*maxSegmentSize*maxSegmentSize; ++i){
            printf("%lf ", input_matrix_segmented[i]);
        }
        #endif

        hipEvent_t startSVD, stopSVD;
        hipEventCreate(&startSVD);
        hipEventCreate(&stopSVD);
        hipEventRecord(startSVD);
        SVD(config.n, num_segments, input_matrix_segmented, maxSegmentSize, h_S, h_U, h_V);
        hipEventRecord(stopSVD);
        hipEventSynchronize(stopSVD);
        hipEventDestroy(startSVD);
        hipEventDestroy(stopSVD);
        hipDeviceSynchronize();

        hipMemcpy(d_S, h_S, maxSegmentSize * num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);
        hipMemcpy(d_U, h_U, maxSegmentSize*maxSegmentSize*num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);
        hipMemcpy(d_V, h_V, maxSegmentSize*maxSegmentSize*num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);

        // ----------------------------ARA----------------------------
        #if 0
        kblasHandle_t kblas_handle;
        kblasRandState_t rand_state;
        kblasCreate(&kblas_handle);
		kblasInitRandState(kblas_handle, &rand_state, 16384*2, 0);
		kblasEnableMagma(kblas_handle);
        kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle, BLOCK_SIZE, num_segments);
        check_kblas_error( kblasAllocateWorkspace(kblas_handle) );

        int* d_rows_batch, d_cols_batch;
        hipMalloc((void**) &d_rows_batch, num_segments*sizeof(int));
        hipMalloc((void**) &d_cols_batch, num_segments*sizeof(int));

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments + numThreadsPerBlock - 1)/numThreadsPerBlock;
        fillBatch<<<numBlocks, numThreadsPerBlock>>>(num_segments, d_rows_batch, d_cols_batch);
        hipDeviceSynchronize();

        hipEvent_t startARA, stopARA;
        hipEventCreate(&startARA);
        hipEventCreate(&stopARA);
        hipEventRecord(startARA);
        kblas_ara_batch( kblas_handles[g], d_rows_batch[g], d_cols_batch[g], d_M_ptrs[g], d_ldm_batch[g], 
								d_A_ptrs[g], d_lda_batch[g], d_B_ptrs[g], d_ldb_batch[g], d_ranks[g], 
								tol, max_rows, max_cols, max_rank, BLOCK_SIZE, ARA_R, rand_state[g], 0, batchCount_gpu);
        hipEventRecord(stopARA);
        hipEventSynchronize(stopARA);
        hipEventDestroy(startARA);
        hipEventDestroy(stopARA);
        hipDeviceSynchronize();

        hipFree(d_rows_batch);
        hipFree(d_cols_batch);
        #endif

        #if 0
        printSigmas(h_S, num_segments, maxSegmentSize, config.bucket_size, config.n, segment);  
        #endif

        numThreadsPerBlock = maxSegmentSize; //TODO: make sure that bucket_size is less than 1024
        numBlocks = num_segments;
        calcMemNeeded<<<numBlocks, numThreadsPerBlock>>> (maxSegmentSize, matrix.blockRanks + segment*num_segments, d_S, config.epsilon);
        hipDeviceSynchronize();

        #if 0
        unsigned int *h_k = (unsigned int *)malloc(num_segments*sizeof(unsigned int));
        hipMemcpy(h_k, matrix.blockRanks + segment*num_segments, num_segments*sizeof(unsigned int), hipMemcpyDeviceToHost);
        printKs(h_k, num_segments, maxSegmentSize, config.bucket_size, config.n, segment, config.epsilon);
        free(h_k);
        #endif

        d_temp_storage = NULL;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* totalMem = (int*)malloc(sizeof(int));
        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, matrix.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();

        cudaErr = hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        hipFree(d_totalMem);

        #if 0
        printf("max mem: %d\n", maxSegmentSize*num_segments);
        printf("total mem: %d\n", (*totalMem));
        #endif

        H2Opus_Real* d_U_tiled_segmented;
        cudaErr = hipMalloc((void**) &d_U_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        H2Opus_Real* d_V_tiled_segmented;
        cudaErr = hipMalloc((void**) &d_V_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

        dim3 d_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
        dim3 d_numBlocks(1, num_segments);
        tileMatrix<<<d_numBlocks, d_numThreadsPerBlock>>> (config.n, num_segments, maxSegmentSize, d_S, d_U, d_V, d_U_tiled_segmented, d_V_tiled_segmented, matrix.blockRanks + segment*num_segments, d_scan_K_segmented, segment);
        hipDeviceSynchronize();

        #if 0
        H2Opus_Real* d_expMatrix;
        cudaErr = hipMalloc((void**) &d_expMatrix, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

        expandMatrix<<<d_numBlocks, d_numThreadsPerBlock>>> (num_segments, maxSegmentSize, matrix.blockRanks + segment*num_segments, d_scan_K_segmented, d_U_tiled_segmented, d_V_tiled_segmented, d_expMatrix);
        hipDeviceSynchronize();

        // #if 0
        // printExpM<<<1, 1>>> (num_segments, maxSegmentSize, d_expMatrix, d_input_matrix_segmented);
        // hipDeviceSynchronize();
        // #endif

        H2Opus_Real* d_error;
        H2Opus_Real* error = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
        hipMalloc((void**) &d_error, sizeof(H2Opus_Real));

        H2Opus_Real* d_tmp;
        H2Opus_Real* tmp = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
        hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));

        *error = 0;
        *tmp = 0;
        hipMemcpy(d_error, error, sizeof(H2Opus_Real), hipMemcpyHostToDevice);
        hipMemcpy(d_tmp, tmp, sizeof(H2Opus_Real), hipMemcpyHostToDevice);

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments*maxSegmentSize*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
        calcError<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_expMatrix, d_input_matrix_segmented, d_error, d_tmp);
        hipDeviceSynchronize();
        hipMemcpy(error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        hipMemcpy(tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        hipFree(d_error);
        hipFree(d_tmp);
        // printf("error: %lf\n", sqrt(*error)/sqrt(*tmp));
        free(tmp);
        free(error);
        hipFree(d_expMatrix);
        #endif

        cudaErr = hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        hipMemcpy(d_U_tiled_temp[segment], d_U_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        hipMemcpy(d_V_tiled_temp[segment], d_V_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);

        k_sum += (*totalMem);

        free(totalMem);
        hipFree(d_U_tiled_segmented);
        hipFree(d_V_tiled_segmented);
    }
    printf("total mem %d\n", k_sum);
    timer_arr[5] =k_sum;

    hipEventRecord(stopGenerateInputMatrix);
    hipEventSynchronize(stopGenerateInputMatrix);
    float GenMatrix_time = 0;
    hipEventElapsedTime(&GenMatrix_time, startGenerateInputMatrix, stopGenerateInputMatrix);
    timer_arr[6] = GenMatrix_time;
    hipEventDestroy(startGenerateInputMatrix);
    hipEventDestroy(stopGenerateInputMatrix);

    free(h_S);
    free(h_U);
    free(h_V);
    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_scan_K_segmented);
    free(input_matrix_segmented);
    hipFree(d_values_in);
    hipFree(d_offsets_sort);
    hipFree(d_dataset);
    hipFree(d_input_matrix_segmented);
    
    cudaErr = hipMalloc((void**) &matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix.blockOffsets, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipDeviceSynchronize();
    hipFree(d_temp_storage);

    unsigned int* h_scan_K = (unsigned int*)malloc(num_segments*num_segments*sizeof(unsigned int));
    cudaErr = hipMemcpy(h_scan_K, matrix.blockOffsets, num_segments*num_segments*sizeof(unsigned int), hipMemcpyDeviceToHost);
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    for(unsigned int segment = 0; segment < num_segments-1; ++segment){
        cudaErr = hipMemcpy(&matrix.U[h_scan_K[num_segments*segment]*maxSegmentSize], d_U_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(int), hipMemcpyDeviceToDevice);
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMemcpy(&matrix.V[h_scan_K[num_segments*segment]*maxSegmentSize], d_V_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(int), hipMemcpyDeviceToDevice);
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    }

    printf("copied data\n");
    for(unsigned int segment = 0; segment < num_segments; ++segment){
        hipFree(d_U_tiled_temp[segment]);
        hipFree(d_V_tiled_temp[segment]);
    }
    free(h_scan_K);
    free(d_U_tiled_temp);
    free(d_V_tiled_temp);
    
    // TODO: cudafree d_U_tiled_temp and d_V_tiled_temp

    H2Opus_Real* d_buffer_vector;
    H2Opus_Real* d_input_vector;
    H2Opus_Real* d_output_vector;
    H2Opus_Real* d_output_vector_org;
    cudaErr = hipMalloc((void**) &d_buffer_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_buffer_vector: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_input_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_input_vector: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_output_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_output_vector_org, maxSegmentSize*num_segments*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }

    numThreadsPerBlock = 1024;
    numBlocks = (num_segments*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
    fillVector<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_input_vector, d_output_vector, d_output_vector_org);
    cudaErr = hipGetLastError();
    if ( cudaErr != hipSuccess ){ printf("CUDA Error fillVector: %s\n", hipGetErrorString(cudaErr)); }
    hipDeviceSynchronize();
    printf("filled vector\n");

    numThreadsPerBlock = 2*upper_power_of_two(maxSegmentSize);
    numBlocks = (num_segments+1)/2;

    hipEvent_t startGEMV, stopGEMV;
    hipEventCreate(&startGEMV);
    hipEventCreate(&stopGEMV);
    hipEventRecord(startGEMV);
    GEMV<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, matrix.blockRanks, matrix.blockOffsets, matrix.U, matrix.V, matrix.diagonal, d_input_vector, d_output_vector, d_buffer_vector);
    cudaErr = hipGetLastError();
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    hipDeviceSynchronize();
    hipEventRecord(stopGEMV);
    hipEventSynchronize(stopGEMV);
    float GEMV_time = 0;
    hipEventElapsedTime(&GEMV_time, startGEMV, stopGEMV);
    hipEventDestroy(startGEMV);
    hipEventDestroy(stopGEMV);

    // dense matrix * vector
    #if 1
    // generate n*n dense matrix
    H2Opus_Real* d_denseMatrix;
    cudaErr = hipMalloc((void**) &d_denseMatrix, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }
    H2Opus_Real* d_vector;
    cudaErr = hipMalloc((void**) &d_vector, config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }

    numThreadsPerBlock = 1024; //TODO: make sure that bucket_size is less than 1024   
    numBlocks = (config.n + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix);
    filltmpVector<<<numBlocks, numThreadsPerBlock>>>(config.n, d_vector);
    hipDeviceSynchronize();

    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    H2Opus_Real alfa=1, beta=0;
    
    hipEvent_t startDenseGEMV, stopDenseGEMV;
    hipEventCreate(&startDenseGEMV);
    hipEventCreate(&stopDenseGEMV);
    hipEventRecord(startDenseGEMV);
    hipblasDgemv(handle, HIPBLAS_OP_T,
                           config.n, config.n,
                           &alfa,
                           d_denseMatrix, config.n,
                           d_vector, 1,
                           &beta,
                           d_vector, 1);
    hipDeviceSynchronize();
    hipEventRecord(stopDenseGEMV);
    hipEventSynchronize(stopDenseGEMV);
    float DenseGEMV_time = 0;
    hipEventElapsedTime(&DenseGEMV_time, startDenseGEMV, stopDenseGEMV);
    printf("DenseGEMV time: %f\n", DenseGEMV_time);
    timer_arr[8] = DenseGEMV_time;
    hipEventDestroy(startDenseGEMV);
    hipEventDestroy(stopDenseGEMV);

    hipblasDestroy(handle);
    hipFree(d_denseMatrix);
    hipFree(d_vector);
    #endif

    printf("GEMV time: %f\n", GEMV_time);
    timer_arr[7] = GEMV_time;

    hipFree(d_buffer_vector);
    hipFree(d_input_vector);
    hipFree(d_output_vector);
    hipFree(d_output_vector_org);

    // H2Opus_Real* d_U_tiled_2;
    // H2Opus_Real* d_V_tiled_2;
    // int* d_scan_K_2;
    // int* d_K_2;
    TLR_Matrix matrix2;
    cudaErr = hipMalloc((void**) &matrix2.blockRanks, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix2.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix2.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix2.diagonal, maxSegmentSize*maxSegmentSize*num_segments*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &matrix2.blockOffsets, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    hipMemcpy(matrix2.blockRanks, matrix.blockRanks, num_segments*num_segments*sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.blockOffsets, matrix.blockOffsets, num_segments*num_segments*sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.U, matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.V, matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.diagonal, matrix.diagonal, maxSegmentSize*maxSegmentSize*num_segments*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);

    H2Opus_Real* d_U_tiled_output;
    H2Opus_Real* d_V_tiled_output;
    int* d_scan_K_output;
    TLR_Matrix matrix_gemm_output;

    cudaErr = hipMalloc((void**) &matrix_gemm_output.blockRanks, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_U_tiled_output, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_V_tiled_output, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_scan_K_output, num_segments*num_segments*sizeof(unsigned int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    h_S = (H2Opus_Real *)malloc(maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    h_U = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    h_V = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));

    d_S;
    d_U;
    d_V;
    cudaErr = hipMalloc((void**) &d_S, maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_U, maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    cudaErr = hipMalloc((void**) &d_V, maxSegmentSize * maxSegmentSize * num_segments * sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    H2Opus_Real* d_gemm_matrix_segmented;
    H2Opus_Real* gemm_matrix_segmented = (H2Opus_Real*)malloc(maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    cudaErr = hipMalloc((void**) &d_gemm_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

    d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    d_scan_K_segmented;
    cudaErr = hipMalloc((void**) &d_scan_K_segmented, num_segments*sizeof(int));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
    k_sum=0;

    dim3 mm_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
    dim3 mm_numBlocks(1, num_segments);
    numThreadsPerBlock = maxSegmentSize; //TODO: make sure that bucket_size is less than 1024   
    numBlocks = num_segments;

    hipEvent_t startGEMM, stopGEMM;
    hipEventCreate(&startGEMM);
    hipEventCreate(&stopGEMM);
    hipEventRecord(startGEMM);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        GEMM<<<mm_numBlocks, mm_numThreadsPerBlock, 2*config.bucket_size*config.bucket_size*sizeof(H2Opus_Real)>>>(num_segments, maxSegmentSize, matrix.U, matrix.V, matrix.diagonal, matrix.blockRanks, matrix.blockOffsets, matrix2.U, matrix2.V, matrix2.diagonal, matrix2.blockRanks, matrix2.blockOffsets, d_gemm_matrix_segmented, segment, config.bucket_size);
        hipDeviceSynchronize();
        hipMemcpy(gemm_matrix_segmented, d_gemm_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);

        // SVD on gemm_matrix
        SVD(config.n, num_segments, gemm_matrix_segmented, maxSegmentSize, h_S, h_U, h_V);
        hipDeviceSynchronize();

        hipMemcpy(d_S, h_S, maxSegmentSize * num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);
        hipMemcpy(d_U, h_U, maxSegmentSize*maxSegmentSize*num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);
        hipMemcpy(d_V, h_V, maxSegmentSize*maxSegmentSize*num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);

        // calcmemneeded and truncation on gemm matrix
        calcMemNeeded<<<numBlocks, numThreadsPerBlock>>> (maxSegmentSize, matrix_gemm_output.blockRanks + segment*num_segments, d_S, config.epsilon);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix_gemm_output.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix_gemm_output.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* totalMem = (int*)malloc(sizeof(int));
        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, matrix_gemm_output.blockRanks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();

        cudaErr = hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        hipFree(d_totalMem);

        H2Opus_Real* d_U_tiled_segmented;
        cudaErr = hipMalloc((void**) &d_U_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        H2Opus_Real* d_V_tiled_segmented;
        cudaErr = hipMalloc((void**) &d_V_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }

        dim3 d_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
        dim3 d_numBlocks(1, num_segments);
        tileMatrix<<<d_numBlocks, d_numThreadsPerBlock>>> (config.n, num_segments, maxSegmentSize, d_S, d_U, d_V, d_U_tiled_segmented, d_V_tiled_segmented, matrix_gemm_output.blockRanks + segment*num_segments, d_scan_K_segmented, segment);
        hipDeviceSynchronize();

        cudaErr = hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        cudaErr = hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
        if ( cudaErr != hipSuccess ){ printf("CUDA Error: %s\n", hipGetErrorString(cudaErr)); }
        hipMemcpy(d_U_tiled_temp[segment], d_U_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        hipMemcpy(d_V_tiled_temp[segment], d_V_tiled_segmented, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);

        k_sum += (*totalMem);

        free(totalMem);
        hipFree(d_U_tiled_segmented);
        hipFree(d_V_tiled_segmented);
        // save output in a double pointer array
    }

    hipDeviceSynchronize();
    hipEventRecord(stopGEMM);
    hipEventSynchronize(stopGEMM);
    float GEMM_time = 0;
    hipEventElapsedTime(&GEMM_time, startGEMM, stopGEMM);
    hipEventDestroy(startGEMM);
    hipEventDestroy(stopGEMM);
    printf("GEMM time: %f\n", GEMM_time);
    timer_arr[9] = GEMM_time;
    // TODO: copy from double pointer array to a single pointer array

    #if 1
    // generate n*n dense matrix
    H2Opus_Real* d_denseMatrix1;
    cudaErr = hipMalloc((void**) &d_denseMatrix1, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }
    H2Opus_Real* d_denseMatrix2;
    cudaErr = hipMalloc((void**) &d_denseMatrix2, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }
    H2Opus_Real* d_denseMatrix3;
    cudaErr = hipMalloc((void**) &d_denseMatrix3, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error d_output_vector_org: %s\n", hipGetErrorString(cudaErr)); }

    numThreadsPerBlock = 1024; //TODO: make sure that bucket_size is less than 1024   
    numBlocks = (config.n + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix1);
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix2);
    hipDeviceSynchronize();

    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    hipEvent_t startDenseGEMM, stopDenseGEMM;
    hipEventCreate(&startDenseGEMM);
    hipEventCreate(&stopDenseGEMM);
    hipEventRecord(startDenseGEMM);
    hipDeviceSynchronize();
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                           config.n, config.n, config.n,
                           &alfa,
                           d_denseMatrix1, config.n,
                           d_denseMatrix2, config.n,
                           &beta,
                           d_denseMatrix3, config.n);
    hipDeviceSynchronize();
    hipEventRecord(stopDenseGEMM);
    hipEventSynchronize(stopDenseGEMM);
    float DenseGEMM_time = 0;
    hipEventElapsedTime(&DenseGEMM_time, startDenseGEMM, stopDenseGEMM);
    printf("DenseGEMM time: %f\n", DenseGEMM_time);
    timer_arr[10] = DenseGEMM_time;
    hipEventDestroy(startDenseGEMM);
    hipEventDestroy(stopDenseGEMM);

    hipblasDestroy(handle);
    hipFree(d_denseMatrix1);
    hipFree(d_denseMatrix2);
    hipFree(d_denseMatrix3);
    #endif

    free(gemm_matrix_segmented);
    hipFree(d_scan_K_segmented);
    hipFree(matrix_gemm_output.blockRanks);
    hipFree(d_gemm_matrix_segmented);
    
    cudaFreeMatrix(matrix);
    cudaFreeMatrix(matrix2);

    hipDeviceSynchronize();
    hipEventRecord(stopCode);
    hipEventSynchronize(stopCode);
    float code_time = 0;
    float Code_time=0;
    hipEventElapsedTime(&Code_time, startCode, stopCode);
    hipEventDestroy(startCode);
    hipEventDestroy(stopCode);
    printf("total time: %f\n", Code_time);
    timer_arr[11] = Code_time;
    printCountersInFile(timer_arr);
    free(timer_arr);
    
}