#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "helperFunctions.h"
#include "helperKernels.cuh"

#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>

#define eps 1e-4
#define PRINT_OUTPUT 1
#define USE_SVD 0
#define DIVIDE_IN_HALF 0
using namespace std;

// TODO: generate pointcloud and copy values of the pointcloud to ptr on GPU
// TODO: fix makefile so main.cu depends on helperKerlens.cuh

int main(){
    int n = 21;
    int dim = 2;
    printf("N = %d\n", n);

    // Create point cloud
    PointCloud<H2Opus_Real> pt_cloud(dim, (size_t)n);
    generateGrid<H2Opus_Real>(pt_cloud, n);
    printf("dimension: %d\n", pt_cloud.getDimension());
    printf("bucket size: %d\n", BUCKET_SIZE);

    #if PRINT_OUTPUT
    printf("created point cloud\n");
    for(int i=0; i<n; ++i){
        for(int j=0; j<dim;++j){
            printf("%f ", pt_cloud.pts[j][i]);
        }
        printf("\n");
    }
    printf("\n\n");
    #endif

    H2Opus_Real *dataset;
    dataset = (H2Opus_Real*)malloc((long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    assert(dataset != NULL);

    // TODO: move this to a kernel
    for (unsigned long long i = 0; i < dim; ++i){
        for(unsigned long long j = 0; j < n; ++j){
            dataset[i*n+j] = pt_cloud.getDataPoint((size_t)j, (int)i);
        }
    }

    H2Opus_Real *d_dataset;
    hipMalloc((void**) &d_dataset, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    hipMemcpy(d_dataset, dataset, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real*), hipMemcpyHostToDevice);

    unsigned long long segment_size = upper_power_of_two(n);
    unsigned long long  num_segments = 1;
    unsigned long long num_segments_reduce = num_segments*dim;

    int *d_offsets_sort;         // e.g., [0, 3, 3, 7]
    int *d_offsets_reduce;
    H2Opus_Real *d_keys_in;         // e.g., [8, 6, 7, 5, 3, 0, 9]
    H2Opus_Real *d_keys_out;        // e.g., [-, -, -, -, -, -, -]
    int  *d_values_in;       // e.g., [0, 1, 2, 3, 4, 5, 6]
    int  *d_values_out;      // e.g., [-, -, -, -, -, -, -]
    int *currDimArray_d;
    H2Opus_Real *d_reduce_in;
    H2Opus_Real *d_reduce_min_out;
    H2Opus_Real *d_reduce_max_out;
    int *d_temp;
    H2Opus_Real *d_span;
    int* d_span_offsets;
    hipcub::KeyValuePair<int, H2Opus_Real> *d_span_reduce_out;
    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    H2Opus_Real* d_input_matrix;

    hipMalloc((void**) &d_temp, n*sizeof(int));
    hipMalloc((void**) &d_offsets_sort, ((n+BUCKET_SIZE-1)/BUCKET_SIZE)*sizeof(int));
    hipMalloc((void**) &d_offsets_reduce, (long long)((n+BUCKET_SIZE-1)/BUCKET_SIZE)*dim*(long long)sizeof(int));
    hipMalloc((void**) &d_keys_in, n*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_keys_out, n*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_values_in, n*sizeof(int));
    hipMalloc((void**) &d_values_out, n*sizeof(int));
    hipMalloc((void**) &currDimArray_d, ((n+BUCKET_SIZE-1)/BUCKET_SIZE)*sizeof(int));
    hipMalloc((void**) &d_reduce_in, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    hipMalloc((void**) &d_reduce_min_out, (long long)((n+BUCKET_SIZE-1)/BUCKET_SIZE)*dim*(long long)sizeof(int));
    hipMalloc((void**) &d_reduce_max_out, (long long)((n+BUCKET_SIZE-1)/BUCKET_SIZE)*dim*(long long)sizeof(int));
    hipMalloc((void**) &d_span, (long long)((n+BUCKET_SIZE-1)/BUCKET_SIZE)*dim*(long long)sizeof(int));
    hipMalloc((void**) &d_span_offsets, ((n+BUCKET_SIZE-1)/BUCKET_SIZE)*sizeof(int));
    hipMalloc((void**) &d_span_reduce_out, ((n+BUCKET_SIZE-1)/BUCKET_SIZE)*sizeof(hipcub::KeyValuePair<int, H2Opus_Real>));
    hipMalloc((void**) &d_input_matrix, (long long)n*(long long)n*(long long)sizeof(H2Opus_Real));

    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (n+numThreadsPerBlock-1)/numThreadsPerBlock;
    initializeArrays<<<numBlocks, numThreadsPerBlock>>>(n, d_values_in, currDimArray_d);
    hipDeviceSynchronize();

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipEvent_t startWhileLoop, stopWhileLoop;
    hipEventCreate(&startWhileLoop);
    hipEventCreate(&stopWhileLoop);
    hipEventRecord(startWhileLoop);

    unsigned int iteration = 0;

    while(segment_size > BUCKET_SIZE) {
        for(unsigned int i=0; i<numTimers; ++i){
            timer_arr[i]=0;
        }

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+1+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startFillOffsets_k, stopFillOffsets_k;
        hipEventCreate(&startFillOffsets_k);
        hipEventCreate(&stopFillOffsets_k);
        hipEventRecord(startFillOffsets_k);
        fillOffsetsArrays<<<numBlocks, numThreadsPerBlock>>>(n, dim, num_segments, segment_size, d_offsets_sort, d_offsets_reduce);
        hipEventRecord(stopFillOffsets_k);
        hipEventSynchronize(stopFillOffsets_k);
        hipEventElapsedTime(&timer_arr[0], startFillOffsets_k, stopFillOffsets_k);
        hipEventDestroy(startFillOffsets_k);
        hipEventDestroy(stopFillOffsets_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (long long)((long long)n*(long long)dim + numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startFillReduction_k, stopFillReduction_k;
        hipEventCreate(&startFillReduction_k);
        hipEventCreate(&stopFillReduction_k);
        hipEventRecord(startFillReduction_k);
        fillReductionArray<<<numBlocks, numThreadsPerBlock>>> (n, dim, d_dataset, d_values_in, d_reduce_in);
        hipEventRecord(stopFillReduction_k);
        hipEventSynchronize(stopFillReduction_k);
        hipEventElapsedTime(&timer_arr[1], startFillReduction_k, stopFillReduction_k);
        hipEventDestroy(startFillReduction_k);
        hipEventDestroy(stopFillReduction_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startMinReduce_k, stopMinReduce_k;
        hipEventCreate(&startMinReduce_k);
        hipEventCreate(&stopMinReduce_k);
        hipEventRecord(startMinReduce_k);
        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipEventRecord(stopMinReduce_k);
        hipEventSynchronize(stopMinReduce_k);
        hipEventElapsedTime(&timer_arr[2], startMinReduce_k, stopMinReduce_k);
        hipEventDestroy(startMinReduce_k);
        hipEventDestroy(stopMinReduce_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startMaxReduce_k, stopMaxReduce_k;
        hipEventCreate(&startMaxReduce_k);
        hipEventCreate(&stopMaxReduce_k);
        hipEventRecord(startMaxReduce_k);
        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipEventRecord(stopMaxReduce_k);
        hipEventSynchronize(stopMaxReduce_k);
        hipEventElapsedTime(&timer_arr[3], startMaxReduce_k, stopMaxReduce_k);
        hipEventDestroy(startMaxReduce_k);
        hipEventDestroy(stopMaxReduce_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startFindSpan_k, stopFindSpan_k;
        hipEventCreate(&startFindSpan_k);
        hipEventCreate(&stopFindSpan_k);
        hipEventRecord(startFindSpan_k);
        findSpan<<<numBlocks, numThreadsPerBlock>>> (n, dim, num_segments, segment_size, d_reduce_min_out, d_reduce_max_out, d_span, d_span_offsets);
        hipEventRecord(stopFindSpan_k);
        hipEventSynchronize(stopFindSpan_k);
        hipEventElapsedTime(&timer_arr[4], startFindSpan_k, stopFindSpan_k);
        hipEventDestroy(startFindSpan_k);
        hipEventDestroy(stopFindSpan_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startArgMaxReduce_k, stopArgMaxReduce_k;
        hipEventCreate(&startArgMaxReduce_k);
        hipEventCreate(&stopArgMaxReduce_k);
        hipEventRecord(startArgMaxReduce_k);
        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        hipEventRecord(stopArgMaxReduce_k);
        hipEventSynchronize(stopArgMaxReduce_k);
        hipEventElapsedTime(&timer_arr[5], startArgMaxReduce_k, stopArgMaxReduce_k);
        hipEventDestroy(startArgMaxReduce_k);
        hipEventDestroy(stopArgMaxReduce_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startfillCurrDim_k, stopFillCurrDim_k;
        hipEventCreate(&startfillCurrDim_k);
        hipEventCreate(&stopFillCurrDim_k);
        hipEventRecord(startfillCurrDim_k);
        fillCurDimArray<<<numBlocks, numThreadsPerBlock>>> (n, num_segments, currDimArray_d, d_span_reduce_out);
        hipEventRecord(stopFillCurrDim_k);
        hipEventSynchronize(stopFillCurrDim_k);
        hipEventElapsedTime(&timer_arr[6], startfillCurrDim_k, stopFillCurrDim_k);
        hipEventDestroy(startfillCurrDim_k);
        hipEventDestroy(stopFillCurrDim_k);
        hipDeviceSynchronize();

        // fill keys_in array
        numThreadsPerBlock = 1024;
        numBlocks = (n+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startfillKeysIn_k, stopKeysIn_k;
        hipEventCreate(&startfillKeysIn_k);
        hipEventCreate(&stopKeysIn_k);
        hipEventRecord(startfillKeysIn_k);
        fillKeysIn<<<numBlocks, numThreadsPerBlock>>> (n, segment_size, d_keys_in, currDimArray_d, d_values_in, d_dataset);
        hipEventRecord(stopKeysIn_k);
        hipEventSynchronize(stopKeysIn_k);
        hipEventElapsedTime(&timer_arr[7], startfillKeysIn_k, stopKeysIn_k);
        hipEventDestroy(startfillKeysIn_k);
        hipEventDestroy(stopKeysIn_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startSort_k, stopSort_k;
        hipEventCreate(&startSort_k);
        hipEventCreate(&stopSort_k);
        hipEventRecord(startSort_k);
        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        n, num_segments, d_offsets_sort, d_offsets_sort + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        n, num_segments, d_offsets_sort, d_offsets_sort + 1);
        hipEventRecord(stopSort_k);
        hipEventSynchronize(stopSort_k);
        hipEventElapsedTime(&timer_arr[8], startSort_k, stopSort_k);
        hipEventDestroy(startSort_k);
        hipEventDestroy(stopSort_k);
        hipDeviceSynchronize();

        hipFree(d_temp_storage);

        d_temp = d_values_in;
        d_values_in = d_values_out;
        d_values_out = d_temp;

        printCountersInFile(iteration, segment_size, num_segments, timer_arr);

        ++iteration;
        segment_size /= 2;
        num_segments = (n/segment_size) + 1;
        num_segments_reduce = num_segments*dim;
    }

    hipEventRecord(stopWhileLoop);
    hipEventSynchronize(stopWhileLoop);
    float whileLoop_time = 0;
    hipEventElapsedTime(&whileLoop_time, startWhileLoop, stopWhileLoop);
    printf("total time taken for while loop: %f\n", whileLoop_time);
    hipEventDestroy(startWhileLoop);
    hipEventDestroy(stopWhileLoop);

    #if 0
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (n*n+numThreadsPerBlock-1)/numThreadsPerBlock;
    // TODO: rewrite padded numbers as INT_MAX
    generateInputMatrix<<<numBlocks, numThreadsPerBlock>>>(n, dim, d_values_in, d_input_matrix, d_dataset);
    cudaDeivceSynchronize();

    int nBlocks = (n+BUCKET_SIZE-1)/BUCKET_SIZE;

    int NRows=BUCKET_SIZE;
    int NCols=BUCKET_SIZE;

    H2Opus_Real* S = (H2Opus_Real*)malloc(min(Nrows, Ncols) * nBlocks*nBlocks * sizeof(H2Opus_Real));
    H2Opus_Real* U = (H2Opus_Real*)malloc(Nrows * Ncols * nBlocks*nBlocks * sizeof(H2Opus_Real));
    H2Opus_Real* V = (H2Opus_Real*)malloc(Nrows * Ncols * nBlocks*nBlocks * sizeof(H2Opus_Real));

    SVD(n, BUCKET_SIZE, d_input_matrix, S, U, V);

    int* d_K;
    H2Opus_Real* d_S;
    H2Opus_Real* d_U;
    H2Opus_Real* d_V;
    hipMalloc((void**) &d_S, min(Nrows, Ncols) * nBlocks*nBlocks * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_U, Nrows*Ncols * nBlocks*nBlocks * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_V, Nrows*Ncols * nBlocks*nBlocks * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_K, nBlocks*nBlocks * sizeof(int));

    hipMemcpy(d_S, S, min(Nrows, Ncols) * nBlocks*nBlocks * sizeof(H2Opus_Real), hipMemcpyHostToDevice);

    // TODO: call a kernel to figure out size of each tile
    numThreadsPerBlock = BUCKET_SIZE; //TODO: make sure that bucket_size is less than 1024
    numBlocks = (nBlocks*nBlocks+numThreadsPerBlock-1)/numThreadsPerBlock;
    calcMemNeeded<<<numBlocks, numThreadsPerBlock>>> (n, d_K, d_S, eps);
    hipDeviceSynchronize();

    int* totalMem = (int*)malloc(sizeof(int));
    int* d_totalMem;
    hipMalloc((void**) &d_totalMem, sizeof(int));
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_K, d_totalMem, nBlocks*nBlocks);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run sum-reduction
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_K, d_totalMem, nBlocks*nBlocks);

    hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);

    // TODO: call a kernel to allocate the tiles, copy them from Us, Ss and Vs
    H2Opus_Real* d_STiled;
    H2Opus_Real* d_UTiled;
    H2Opus_Real* d_VTiled;
    hipMalloc((void**) &d_STiles, totalMemNeeded * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_UTiles, Nrows*totalMemNeeded * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_VTiled, Nrows*totalMemNeeded * sizeof(H2Opus_Real));

    int* maxMem = (int*)malloc(sizeof(int));
    int* d_maxMem;
    hipMalloc((void**) &d_maxMem, sizeof(int));
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_K, d_maxMem, nBlocks*nBlocks);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_K, d_maxMem, nBlocks*nBlocks);
    hipMemcpy(maxMem, d_maxMem, sizeof(int), hipMemcpyDeviceToHost);

    int* d_KScan;
    hipMalloc((void**) &d_maxMem, nBlocks*nBlocks*sizeof(int));
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_K, d_out, nBlocks*nBlocks);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_K, d_out, nBlocks*nBlocks);

    numThreadsPerBlock = maxMem;
    numBlocks = (nBlocks*nBlocks+numThreadsPerBlock-1)/numThreadsPerBlock;
    tileMatrix<<<numBlocks, numThreadsPerBlock>>> (n, d_S, d_U, d_V, d_Siled, d_UTiled, d_VTiled, d_K, d_KScan);

    free(Ss);
    free(Us);
    free(Vs);
    #endif

    #if PRINT_OUTPUT
    int *index_map = (int*)malloc(n*sizeof(int));
    hipMemcpy(index_map, d_values_in, n*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0; i<n; ++i){
        for(int j=0; j<dim; ++j){
            printf("%f ", pt_cloud.pts[j][index_map[i]]);
        }
        printf("\n");
    }
    free(index_map);
    #endif

    free(dataset);

    hipFree(d_dataset);
    hipFree(d_offsets_sort);
    hipFree(d_offsets_reduce);
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    hipFree(d_values_in);
    hipFree(d_values_out);
    hipFree(currDimArray_d);
    hipFree(d_reduce_in);
    hipFree(d_reduce_min_out);
    hipFree(d_reduce_max_out);
    hipFree(d_temp);
    hipFree(d_span_reduce_out);
    hipFree(d_span);
    hipFree(d_span_offsets);
    hipFree(d_input_matrix);
}