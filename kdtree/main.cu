#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "SVD.cuh"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>

#define eps 1e-4
#define PRINT_OUTPUT 0
#define USE_SVD 0
#define DIVISION_METHOD 2
using namespace std;

// TODO: generate pointcloud and copy values of the pointcloud to ptr on GPU
// TODO: fix makefile so main.cu depends on helperKerlens.cuh

int main(){
    int n = 1<<6;
    int dim = 2;
    printf("N = %d\n", n);

    // Create point cloud
    PointCloud<H2Opus_Real> pt_cloud(dim, (size_t)n);
    generateGrid<H2Opus_Real>(pt_cloud, n);
    printf("dimension: %d\n", pt_cloud.getDimension());
    printf("bucket size: %d\n", BUCKET_SIZE);

    #if PRINT_OUTPUT
    printf("created point cloud\n");
    for(int i=0; i<n; ++i){
        for(int j=0; j<dim;++j){
            printf("%f ", pt_cloud.pts[j][i]);
        }
        printf("\n");
    }
    printf("\n\n");
    #endif

    H2Opus_Real *dataset;
    dataset = (H2Opus_Real*)malloc((long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    assert(dataset != NULL);

    // TODO: move this to a kernel
    for (unsigned long long i = 0; i < dim; ++i){
        for(unsigned long long j = 0; j < n; ++j){
            dataset[i*n+j] = pt_cloud.getDataPoint((size_t)j, (int)i);
        }
    }

    H2Opus_Real *d_dataset;
    hipMalloc((void**) &d_dataset, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    hipMemcpy(d_dataset, dataset, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real*), hipMemcpyHostToDevice);

    unsigned int  num_segments = 1;
    unsigned long long num_segments_reduce = num_segments*dim;

    #if DIVISION_METHOD == 0
    unsigned int segment_size = upper_power_of_two(n);
    #endif

    int *d_offsets_sort;         // e.g., [0, 3, 3, 7]
    int *d_offsets_reduce;
    H2Opus_Real *d_keys_in;         // e.g., [8, 6, 7, 5, 3, 0, 9]
    H2Opus_Real *d_keys_out;        // e.g., [-, -, -, -, -, -, -]
    int  *d_values_in;       // e.g., [0, 1, 2, 3, 4, 5, 6]
    int  *d_values_out;      // e.g., [-, -, -, -, -, -, -]
    int *d_curr_dim;
    H2Opus_Real *d_reduce_in;
    H2Opus_Real *d_reduce_min_out;
    H2Opus_Real *d_reduce_max_out;
    int *d_temp;
    H2Opus_Real *d_span;
    int* d_span_offsets;
    hipcub::KeyValuePair<int, H2Opus_Real> *d_span_reduce_out;
    float* timer_arr = (float*)malloc(numTimers*sizeof(float));

    #if DIVISION_METHOD == 1
    bool workDone= false;
    bool* d_workDone;
    uint64_t* d_bit_vector;
    short int* d_popc_bit_vector;
    short int* d_popc_scan;
    unsigned int* d_new_num_segments;
    unsigned int* new_num_segments = (unsigned int*)malloc(sizeof(unsigned int));
    #endif

    int max_num_segments;

    #if DIVISION_METHOD !=0 
    int* A;
    int* B;
    int* output;
    int* d_output;
    int* d_input_search;
    int* d_aux_offsets_sort;
    max_num_segments = 1<<(getMaxSegmentSize(n, BUCKET_SIZE).second);
    #else
    max_num_segments = (n+BUCKET_SIZE-1)/BUCKET_SIZE;
    #endif
    printf("max num segments: %d\n", max_num_segments);

    #if DIVISION_METHOD == 2
    int largest_segment_size = n;
    #endif

    // TODO: fix memory allocated
    hipMalloc((void**) &d_temp, n*sizeof(int));
    hipMalloc((void**) &d_offsets_sort, (max_num_segments + 1)*sizeof(int));
    hipMalloc((void**) &d_offsets_reduce, (long long)((max_num_segments*dim + 1)*(long long)sizeof(int)));
    hipMalloc((void**) &d_keys_in, n*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_keys_out, n*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_values_in, n*sizeof(int));
    hipMalloc((void**) &d_values_out, n*sizeof(int));
    hipMalloc((void**) &d_curr_dim, (max_num_segments + 1)*sizeof(int));
    hipMalloc((void**) &d_reduce_in, (long long)n*(long long)dim*(long long)sizeof(H2Opus_Real));
    hipMalloc((void**) &d_reduce_min_out, (long long)((max_num_segments+1)*dim)*(long long)sizeof(int));
    hipMalloc((void**) &d_reduce_max_out, (long long)((max_num_segments+1)*dim)*(long long)sizeof(int));
    hipMalloc((void**) &d_span, (long long)((max_num_segments+1)*dim)*(long long)sizeof(int));
    hipMalloc((void**) &d_span_offsets, (max_num_segments + 1)*sizeof(int));
    hipMalloc((void**) &d_span_reduce_out, (max_num_segments+1)*sizeof(hipcub::KeyValuePair<int, H2Opus_Real>));

    #if DIVISION_METHOD == 1
    hipMalloc((void**) &d_bit_vector, (((max_num_segments+1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8)) *sizeof(uint64_t));
    hipMalloc((void**) &d_popc_bit_vector, (((max_num_segments + 1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8))*sizeof(short int));
    hipMalloc((void**) &d_popc_scan, (((max_num_segments + 1) + sizeof(uint64_t)*8-1)/(sizeof(uint64_t)*8))*sizeof(short int));
    hipMalloc((void**) &d_new_num_segments, sizeof(unsigned int));
    hipMalloc((void**) &d_workDone, sizeof(bool));
    #endif

    #if DIVISION_METHOD !=0
    hipMalloc((void**) &d_aux_offsets_sort, (max_num_segments + 1) * sizeof(int));
    hipMalloc((void**) &A, (max_num_segments + 1)*sizeof(int));
    hipMalloc((void**) &B, n*sizeof(int));
    hipMalloc((void**) &output, n*sizeof(int));
    hipMalloc((void**) &d_output, n*sizeof(int));
    hipMalloc((void**) &d_input_search, n*sizeof(int));
    #endif

    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (n+numThreadsPerBlock-1)/numThreadsPerBlock;

    #if DIVISION_METHOD != 0
    initializeArrays<<<numBlocks, numThreadsPerBlock>>>(n, dim, d_values_in, d_curr_dim, d_offsets_sort, d_offsets_reduce, d_input_search, max_num_segments);
    #elif DIVISION_METHOD == 0
    initializeArrays<<<numBlocks, numThreadsPerBlock>>>(n, d_values_in, d_curr_dim, max_num_segments);
    #endif
    hipDeviceSynchronize();

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipEvent_t startWhileLoop, stopWhileLoop;
    hipEventCreate(&startWhileLoop);
    hipEventCreate(&stopWhileLoop);
    hipEventRecord(startWhileLoop);

    unsigned int iteration = 0;

    #if DIVISION_METHOD == 1
    while(!workDone) {
    #elif DIVISION_METHOD == 0
    while(segment_size > BUCKET_SIZE) {
    #else
    while(largest_segment_size > BUCKET_SIZE) {
    #endif
        // printf("begin\n");
        for(unsigned int i=0; i<numTimers; ++i){
            timer_arr[i]=0;
        }

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+1+numThreadsPerBlock-1)/numThreadsPerBlock;
        hipEvent_t startFillOffsets_k, stopFillOffsets_k;
        hipEventCreate(&startFillOffsets_k);
        hipEventCreate(&stopFillOffsets_k);
        hipEventRecord(startFillOffsets_k);
        #if DIVISION_METHOD==0
        fillOffsets<<<numBlocks, numThreadsPerBlock>>>(n, dim, num_segments, segment_size, d_offsets_sort, d_offsets_reduce);
        #endif
        hipEventRecord(stopFillOffsets_k);
        hipEventSynchronize(stopFillOffsets_k);
        hipEventElapsedTime(&timer_arr[0], startFillOffsets_k, stopFillOffsets_k);
        hipEventDestroy(startFillOffsets_k);
        hipEventDestroy(stopFillOffsets_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (long long)((long long)n*(long long)dim + numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startFillReduction_k, stopFillReduction_k;
        hipEventCreate(&startFillReduction_k);
        hipEventCreate(&stopFillReduction_k);
        hipEventRecord(startFillReduction_k);
        fillReductionArray<<<numBlocks, numThreadsPerBlock>>> (n, dim, d_dataset, d_values_in, d_reduce_in);
        hipEventRecord(stopFillReduction_k);
        hipEventSynchronize(stopFillReduction_k);
        hipEventElapsedTime(&timer_arr[1], startFillReduction_k, stopFillReduction_k);
        hipEventDestroy(startFillReduction_k);
        hipEventDestroy(stopFillReduction_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startMinReduce_k, stopMinReduce_k;
        hipEventCreate(&startMinReduce_k);
        hipEventCreate(&stopMinReduce_k);
        hipEventRecord(startMinReduce_k);
        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_min_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipEventRecord(stopMinReduce_k);
        hipEventSynchronize(stopMinReduce_k);
        hipEventElapsedTime(&timer_arr[2], startMinReduce_k, stopMinReduce_k);
        hipEventDestroy(startMinReduce_k);
        hipEventDestroy(stopMinReduce_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startMaxReduce_k, stopMaxReduce_k;
        hipEventCreate(&startMaxReduce_k);
        hipEventCreate(&stopMaxReduce_k);
        hipEventRecord(startMaxReduce_k);
        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_reduce_in, d_reduce_max_out,
            num_segments_reduce, d_offsets_reduce, d_offsets_reduce + 1);
        hipEventRecord(stopMaxReduce_k);
        hipEventSynchronize(stopMaxReduce_k);
        hipEventElapsedTime(&timer_arr[3], startMaxReduce_k, stopMaxReduce_k);
        hipEventDestroy(startMaxReduce_k);
        hipEventDestroy(stopMaxReduce_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startFindSpan_k, stopFindSpan_k;
        hipEventCreate(&startFindSpan_k);
        hipEventCreate(&stopFindSpan_k);
        hipEventRecord(startFindSpan_k);
        findSpan<<<numBlocks, numThreadsPerBlock>>> (n, dim, num_segments, d_reduce_min_out, d_reduce_max_out, d_span, d_span_offsets);
        hipEventRecord(stopFindSpan_k);
        hipEventSynchronize(stopFindSpan_k);
        hipEventElapsedTime(&timer_arr[4], startFindSpan_k, stopFindSpan_k);
        hipEventDestroy(startFindSpan_k);
        hipEventDestroy(stopFindSpan_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startArgMaxReduce_k, stopArgMaxReduce_k;
        hipEventCreate(&startArgMaxReduce_k);
        hipEventCreate(&stopArgMaxReduce_k);
        hipEventRecord(startArgMaxReduce_k);
        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run argmax-reduction
        hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_span, d_span_reduce_out,
            num_segments, d_span_offsets, d_span_offsets + 1);
        hipEventRecord(stopArgMaxReduce_k);
        hipEventSynchronize(stopArgMaxReduce_k);
        hipEventElapsedTime(&timer_arr[5], startArgMaxReduce_k, stopArgMaxReduce_k);
        hipEventDestroy(startArgMaxReduce_k);
        hipEventDestroy(stopArgMaxReduce_k);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startfillCurrDim_k, stopFillCurrDim_k;
        hipEventCreate(&startfillCurrDim_k);
        hipEventCreate(&stopFillCurrDim_k);
        hipEventRecord(startfillCurrDim_k);
        #if DIVISION_METHOD == 1
        fillCurrDim<<<numBlocks, numThreadsPerBlock>>> (n, num_segments, d_curr_dim, d_span_reduce_out, d_bit_vector);
        #else
        fillCurrDim<<<numBlocks, numThreadsPerBlock>>> (n, num_segments, d_curr_dim, d_span_reduce_out);
        #endif
        hipDeviceSynchronize();
        hipEventRecord(stopFillCurrDim_k);
        hipEventSynchronize(stopFillCurrDim_k);
        hipEventElapsedTime(&timer_arr[6], startfillCurrDim_k, stopFillCurrDim_k);
        hipEventDestroy(startfillCurrDim_k);
        hipEventDestroy(stopFillCurrDim_k);

        // fill keys_in array
        numThreadsPerBlock = 1024;
        numBlocks = (n+numThreadsPerBlock-1)/numThreadsPerBlock;

        hipEvent_t startfillKeysIn_k, stopKeysIn_k;
        hipEventCreate(&startfillKeysIn_k);
        hipEventCreate(&stopKeysIn_k);
        hipEventRecord(startfillKeysIn_k);

        #if DIVISION_METHOD != 0
        thrust::device_ptr<int> A = thrust::device_pointer_cast((int *)d_offsets_sort), B = thrust::device_pointer_cast((int *)d_input_search);
        thrust::device_vector<int> output(n);
        thrust::upper_bound(A, A + num_segments + 1, B, B + n, output.begin(), thrust::less<int>());
        d_output = thrust::raw_pointer_cast(&output[0]);
        fillKeysIn<<<numBlocks, numThreadsPerBlock>>> (n, d_keys_in, d_curr_dim, d_values_in, d_dataset, d_offsets_sort, d_output);
        #else
        fillKeysIn<<<numBlocks, numThreadsPerBlock>>> (n, segment_size, d_keys_in, d_curr_dim, d_values_in, d_dataset);
        #endif

        hipEventRecord(stopKeysIn_k);
        hipEventSynchronize(stopKeysIn_k);
        hipEventElapsedTime(&timer_arr[7], startfillKeysIn_k, stopKeysIn_k);
        hipEventDestroy(startfillKeysIn_k);
        hipEventDestroy(stopKeysIn_k);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;

        hipEvent_t startSort_k, stopSort_k;
        hipEventCreate(&startSort_k);
        hipEventCreate(&stopSort_k);
        hipEventRecord(startSort_k);
        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        n, num_segments, d_offsets_sort, d_offsets_sort + 1);

        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
        d_keys_in, d_keys_out, d_values_in, d_values_out,
        n, num_segments, d_offsets_sort, d_offsets_sort + 1);
        hipEventRecord(stopSort_k);
        hipEventSynchronize(stopSort_k);
        hipEventElapsedTime(&timer_arr[8], startSort_k, stopSort_k);
        hipEventDestroy(startSort_k);
        hipEventDestroy(stopSort_k);
        hipDeviceSynchronize();
        
        hipFree(d_temp_storage);

        d_temp = d_values_in;
        d_values_in = d_values_out;
        d_values_out = d_temp;
        printCountersInFile(iteration, num_segments, timer_arr);
        ++iteration;

        #if DIVISION_METHOD == 1
        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
        fillBitVector<<<numBlocks, numThreadsPerBlock>>>(num_segments, d_bit_vector, d_offsets_sort);
        hipDeviceSynchronize();
        unsigned int num_threads = (num_segments + sizeof(uint64_t)*8 - 1)/(sizeof(uint64_t)*8);
        numThreadsPerBlock = 1024;
        numBlocks = (num_threads + numThreadsPerBlock-1)/numThreadsPerBlock;
        fillPopCount<<<numBlocks, numThreadsPerBlock>>>(num_threads, d_bit_vector, d_popc_bit_vector);
        hipDeviceSynchronize();

        d_temp_storage = NULL;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_popc_bit_vector, d_popc_scan, num_segments);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run exclusive prefix sum
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_popc_bit_vector, d_popc_scan, num_segments);

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
        fillOffsetsSort<<<numBlocks, numThreadsPerBlock>>>(n, dim, num_segments, d_offsets_sort, d_aux_offsets_sort, d_bit_vector, d_popc_scan, d_new_num_segments, d_workDone);
        hipDeviceSynchronize();
        hipMemcpy(new_num_segments, d_new_num_segments, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(&workDone, d_workDone, sizeof(bool), hipMemcpyDeviceToHost);
        num_segments = *new_num_segments;
        d_temp = d_aux_offsets_sort;
        d_aux_offsets_sort = d_offsets_sort;
        d_offsets_sort = d_temp;

        if(workDone){
            break;
        }

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
        fillOffsetsReduce<<<numBlocks, numThreadsPerBlock>>> (n, dim, num_segments, d_offsets_sort, d_offsets_reduce);
        hipDeviceSynchronize();

        #elif DIVISION_METHOD == 0
        segment_size /= 2;
        num_segments = (n+segment_size-1)/segment_size;

        #elif DIVISION_METHOD == 2
        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
        fillOffsetsSort<<<numBlocks, numThreadsPerBlock>>> (n, dim, num_segments, d_offsets_sort, d_aux_offsets_sort);
        hipDeviceSynchronize();
        d_temp = d_aux_offsets_sort;
        d_aux_offsets_sort = d_offsets_sort;
        d_offsets_sort = d_temp;
        num_segments *= 2;

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments+numThreadsPerBlock-1)/numThreadsPerBlock;
        fillOffsetsReduce<<<numBlocks, numThreadsPerBlock>>> (n, dim, num_segments, d_offsets_sort, d_offsets_reduce);
        hipDeviceSynchronize();
        ++largest_segment_size;
        largest_segment_size /= 2;
        #endif

        num_segments_reduce = num_segments*dim;
    }

    hipEventRecord(stopWhileLoop);
    hipEventSynchronize(stopWhileLoop);
    float whileLoop_time = 0;
    hipEventElapsedTime(&whileLoop_time, startWhileLoop, stopWhileLoop);
    printf("total time taken for while loop: %f\n", whileLoop_time);
    hipEventDestroy(startWhileLoop);
    hipEventDestroy(stopWhileLoop);

    #if DIVISION_METHOD == 0
    printf("num segments :%d\n", num_segments);
    printf("segment size :%d\n", segment_size);
    fillOffsets<<<numBlocks, numThreadsPerBlock>>>(n, dim, num_segments, segment_size, d_offsets_sort, d_offsets_reduce);
    hipDeviceSynchronize();
    #endif

    #if PRINT_OUTPUT
    int *index_map = (int*)malloc(n*sizeof(int));
    hipMemcpy(index_map, d_values_in, n*sizeof(int), hipMemcpyDeviceToHost);
    printf("inex max\n");
    for(int i=0; i<n; ++i){
        printf("%d ", index_map[i]);
    }
    printf("\n");

    for(int i=0; i<n; ++i){
        for(int j=0; j<dim; ++j){
            printf("%f ", pt_cloud.pts[j][index_map[i]]);
        }
        printf("\n");
    }
    free(index_map);
    #endif

    // // _______________________________________________________________________________________________
    // #if 0
    // int maxSegmentSize;
    // #if DIVISION_METHOD != 0
    // maxSegmentSize = getMaxSegmentSize(n, BUCKET_SIZE).first;
    // printf("max segment size: %d\n", maxSegmentSize);
    // printf("n: %d\n", n);
    // #else
    // maxSegmentSize = BUCKET_SIZE;
    // #endif

    // H2Opus_Real* d_input_matrix;
    // H2Opus_Real* input_matrix = (H2Opus_Real*)malloc((long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real));
    // hipMalloc((void**) &d_input_matrix, (long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real));
    
    // dim3 m_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
    // dim3 m_numBlocks(num_segments, num_segments);
    // generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(n, num_segments, maxSegmentSize, dim, d_values_in, d_input_matrix, d_dataset, d_offsets_sort);
    // hipDeviceSynchronize();
    // hipMemcpy(input_matrix, d_input_matrix, (long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);

    // int NRows = maxSegmentSize;
    // int NCols = maxSegmentSize;
    // int numMatrices = num_segments*num_segments;

    // H2Opus_Real *h_S = (H2Opus_Real *)malloc(maxSegmentSize * numMatrices * sizeof(H2Opus_Real));
    // H2Opus_Real *h_U = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * numMatrices * sizeof(H2Opus_Real));
    // H2Opus_Real *h_V = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * numMatrices * sizeof(H2Opus_Real));
    // SVD(n, num_segments, input_matrix, NRows, NCols, maxSegmentSize, h_S, h_U, h_V);
    // #endif
    // // _______________________________________________________________________________________________

    int maxSegmentSize;
    #if DIVISION_METHOD != 0
    maxSegmentSize = getMaxSegmentSize(n, BUCKET_SIZE).first;
    printf("max segment size: %d\n", maxSegmentSize);
    printf("n: %d\n", n);
    #else
    maxSegmentSize = BUCKET_SIZE;
    #endif

    H2Opus_Real* d_input_matrix;
    H2Opus_Real* input_matrix = (H2Opus_Real*)malloc((long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real));
    hipMalloc((void**) &d_input_matrix, (long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real));
    
    dim3 m_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
    dim3 m_numBlocks(num_segments, num_segments);
    generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(n, num_segments, maxSegmentSize, dim, d_values_in, d_input_matrix, d_dataset, d_offsets_sort);
    hipDeviceSynchronize();
    hipMemcpy(input_matrix, d_input_matrix, (long long)maxSegmentSize*num_segments*(long long)maxSegmentSize*num_segments*(long long)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);

    #if PRINT_OUTPUT
    for(unsigned int i=0; i<num_segments*maxSegmentSize; ++i){
        for(unsigned int j=0; j<num_segments*maxSegmentSize; ++j){
            printf("%f ", input_matrix[i*num_segments*maxSegmentSize + j]);
        }
        printf("\n");
    }
    #endif

    int nBlocks = num_segments;
    int NRows = maxSegmentSize;
    int NCols = maxSegmentSize;

    H2Opus_Real *h_S = (H2Opus_Real *)malloc(maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));
    H2Opus_Real *h_U = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));
    H2Opus_Real *h_V = (H2Opus_Real *)malloc(maxSegmentSize * maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));

    SVD(n, num_segments, input_matrix, NRows, NCols, maxSegmentSize, h_S, h_U, h_V);
    hipDeviceSynchronize();

    int* d_K;
    H2Opus_Real* d_S;
    H2Opus_Real* d_U;
    H2Opus_Real* d_V;
    hipMalloc((void**) &d_S, maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_U, maxSegmentSize * maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_V, maxSegmentSize * maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real));
    hipMalloc((void**) &d_K, num_segments*num_segments * sizeof(int));
    hipMemcpy(d_S, h_S, maxSegmentSize * num_segments*num_segments * sizeof(H2Opus_Real), hipMemcpyHostToDevice);
    
    printf("num segments: %d\n", num_segments);
    // TODO: call a kernel to figure out size of each tile
    numThreadsPerBlock = maxSegmentSize; //TODO: make sure that bucket_size is less than 1024
    numBlocks = num_segments*num_segments;
    calcMemNeeded<<<numBlocks, numThreadsPerBlock>>> (n, maxSegmentSize, d_K, d_S, eps, d_offsets_sort, num_segments);
    hipDeviceSynchronize();

    int* d_scan_K;
    hipMalloc((void**) &d_scan_K, num_segments*num_segments*sizeof(int));
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_K, d_scan_K, num_segments*num_segments);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_K, d_scan_K, num_segments*num_segments);

    int* totalMem = (int*)malloc(sizeof(int));
    int* d_totalMem;
    hipMalloc((void**) &d_totalMem, sizeof(int));
    getTotalMem<<<1, 1>>> (d_totalMem, d_K, d_scan_K, num_segments);
    hipDeviceSynchronize();
    hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);

    H2Opus_Real* d_U_tiled;
    H2Opus_Real* d_V_tiled;
    hipMalloc((void**) &d_U_tiled, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_V_tiled, maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real));

    // TODO: find the maximum rank using a max reduction
    dim3 d_numThreadsPerBlock(upper_power_of_two(maxSegmentSize), upper_power_of_two(maxSegmentSize));
    dim3 d_numBlocks(num_segments, num_segments);
    tileMatrix<<<d_numBlocks, d_numThreadsPerBlock>>> (n, num_segments, maxSegmentSize, d_S, d_U, d_V, d_U_tiled, d_V_tiled, d_K, d_scan_K);
    hipDeviceSynchronize();

    H2Opus_Real* expMatrix;
    hipMalloc((void**) &expMatrix, num_segments*num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));
    expandMatrix<<<d_numBlocks, d_numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_K, d_scan_K, d_U_tiled, d_V_tiled, expMatrix);
    hipDeviceSynchronize();

    H2Opus_Real* d_error;
    H2Opus_Real* error = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_error, sizeof(H2Opus_Real));

    H2Opus_Real* d_tmp;
    H2Opus_Real* tmp = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));

    numThreadsPerBlock = 1024;
    numBlocks = (num_segments*num_segments*maxSegmentSize*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
    calcError<<<d_numBlocks, d_numThreadsPerBlock>>> (num_segments, maxSegmentSize, expMatrix, d_input_matrix, d_error, d_tmp);
    hipDeviceSynchronize();
    hipMemcpy(error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipMemcpy(tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    printf("error: %f\n", sqrt(*error)/sqrt(*tmp));
    // free(dataset);
    // hipFree(d_dataset);
    // hipFree(d_offsets_sort);
    // hipFree(d_offsets_reduce);
    // hipFree(d_keys_in);
    // hipFree(d_keys_out);
    // hipFree(d_values_in);
    // hipFree(d_values_out);
    // hipFree(d_curr_dim);
    // hipFree(d_reduce_in);
    // hipFree(d_reduce_min_out);
    // hipFree(d_reduce_max_out);
    // hipFree(d_temp);
    // hipFree(d_span_reduce_out);
    // hipFree(d_span);
    // hipFree(d_span_offsets);
    // hipFree(d_input_matrix);
    // free(input_matrix);

    // #if DIVISION_METHOD == 1
    // hipFree(A);
    // hipFree(B);
    // hipFree(output);
    // hipFree(d_output);
    // hipFree(d_input_search);
    // hipFree(d_bit_vector);
    // hipFree(d_aux_offsets_sort);
    // hipFree(d_new_num_segments);
    // free(new_num_segments);
    // hipFree(d_workDone);
    // #endif
}