
struct Result {

	double runtime_ms;
	double initialization_time_ms;
	double gflops;
	cutlass::Status status;
	hipError_t error;
	bool passed;

	Result(
		double runtime_ms = 0,
		double initialization_time_ms = 0,
		double gflops = 0,
		cutlass::Status status = cutlass::Status::kSuccess,
		hipError_t error = hipSuccess
	):
	runtime_ms(runtime_ms), initialization_time_ms(initialization_time_ms), gflops(gflops),
	status(status), error(error), passed(true) { }
};

hipError_t cutlass_grouped_dgemm() {
}

hipError_t cutlassHierarchicalXVec(
    unsigned int numberOfInputPoints, unsigned int  bucketSize, 
    unsigned int  numSegments, unsigned int  vectorWidth, HMatrix hierarchicalMatrix,
    H2Opus_Real *inputVectors, H2Opus_Real *resultVectors) {

		using ElementA = double;
		using ElementB = double;
		using ElementOutput = double;
		using ElementAccumulator = double;
		using LayoutA = cutlass::layout::ColumnMajor;
		using LayoutB = cutlass::layout::ColumnMajor;
		using LayoutC = cutlass::layout::ColumnMajor;

		using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
			ElementA, LayoutA,
			cutlass::ComplexTransform::kNone, 8,
			ElementB, LayoutB,
			cutlass::ComplexTransform::kNone, 8,
			ElementOutput, LayoutC,
			ElementAccumulator,
			cutlass::arch::OpClassTensorOp,
			cutlass::arch::Sm80,
			cutlass::gemm::GemmShape<128, 128, 32>,
			cutlass::gemm::GemmShape<64, 64, 32>,
			cutlass::gemm::GemmShape<16, 8, 16>,
			cutlass::epilogue::thread::LinearCombination<
				ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
				ElementAccumulator, ElementAccumulator>,
			cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
			4>::GemmKernel;

    	using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;
		
		typename Gemm::EpilogueOutputOp::Params epilogue_1(1.0f, 0.0f);
		typename Gemm::EpilogueOutputOp::Params epilogue_2(1.0f, 1.0f);

      	// loop over levels
		for(unsigned int level = WAStruct.numLevels - 2; level > 0; --level) {
        	// TODO: preprocess each level

			int threadblock_count = Gemm::sufficient(problem_sizes.data(), problem_count);
			if (!threadblock_count) {
				printf("Active CUDA device lacks hardware resources to run CUTLASS Grouped GEMM kernel.");
				return result;
			}

			typename Gemm::Arguments args(
				problem_sizes_device.get(),
				problem_count(),
				threadblock_count,
				epilogue_1,
				ptr_A.get(),
				ptr_B.get(),
				ptr_C.get(),
				ptr_D.get(),
				lda.get(),
				ldb.get(),
				ldc.get(),
				ldd.get(),
				options.problem_sizes.data()
			);

			Gemm gemm;
			size_t workspace_size = gemm.get_workspace_size(args);
			cutlass::DeviceAllocation<uint8_t> workspace(workspace_size);

			result.status = gemm.initialize(args, workspace.get());

			gemm.run();
		}
}