#include "hip/hip_runtime.h"

#include "helperFunctions.cuh"
#include <hiprand.h>

void convertColumnMajorToMorton(unsigned int numSegments, unsigned int maxSegmentSize, uint64_t rankSum, TLR_Matrix matrix, TLR_Matrix &mortonMatrix) {

    hipMalloc((void**) &mortonMatrix.U, rankSum*maxSegmentSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &mortonMatrix.V, rankSum*maxSegmentSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &mortonMatrix.blockOffsets, numSegments*numSegments*sizeof(int));
    hipMalloc((void**) &mortonMatrix.blockRanks, numSegments*numSegments*sizeof(int));
    hipMalloc((void**) &mortonMatrix.diagonal, static_cast<uint64_t>(numSegments)*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));

    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (numSegments*numSegments + 1024 - 1)/1024;
    copyCMRanksToMORanks <<< numBlocks, numThreadsPerBlock >>> (numSegments, maxSegmentSize, matrix.blockRanks, mortonMatrix.blockRanks);

    // scan mortonMatrix ranks
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, mortonMatrix.blockRanks, mortonMatrix.blockOffsets, numSegments*numSegments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, mortonMatrix.blockRanks, mortonMatrix.blockOffsets, numSegments*numSegments);
    hipFree(d_temp_storage);

    int* h_matrix_offsets = (int*)malloc(numSegments*numSegments*sizeof(int));
    int* h_mortonMatrix_offsets = (int*)malloc(numSegments*numSegments*sizeof(int));
    hipMemcpy(h_matrix_offsets, matrix.blockOffsets, numSegments*numSegments*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_mortonMatrix_offsets, mortonMatrix.blockOffsets, numSegments*numSegments*sizeof(int), hipMemcpyDeviceToHost);

    int* h_matrix_ranks = (int*)malloc(numSegments*numSegments*sizeof(int));
    hipMemcpy(h_matrix_ranks, matrix.blockRanks, numSegments*numSegments*sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned int i=0; i<numSegments*numSegments; ++i){
        int MOIndex = CMIndextoMOIndex(numSegments, i);
        unsigned int numThreadsPerBlock = 1024;
        unsigned int numBlocks = (h_matrix_ranks[i]*maxSegmentSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
        assert(h_matrix_ranks[i] >= 0);
        if(h_matrix_ranks[i] > 0){
            hipMemcpy(&mortonMatrix.U[static_cast<uint64_t>(h_mortonMatrix_offsets[MOIndex])*maxSegmentSize], &matrix.U[static_cast<uint64_t>(h_matrix_offsets[i])*maxSegmentSize], static_cast<uint64_t>(h_matrix_ranks[i])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
            hipMemcpy(&mortonMatrix.V[static_cast<uint64_t>(h_mortonMatrix_offsets[MOIndex])*maxSegmentSize], &matrix.V[static_cast<uint64_t>(h_matrix_offsets[i])*maxSegmentSize], static_cast<uint64_t>(h_matrix_ranks[i])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        }
    }

    hipMemcpy(mortonMatrix.diagonal, matrix.diagonal, numSegments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void copyCMRanksToMORanks(int num_segments, int maxSegmentSize, int* matrixRanks, int* mortonMatrixRanks){
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i<num_segments*num_segments){
        int MOIndex = CMIndextoMOIndex(num_segments, i);
        mortonMatrixRanks[MOIndex] = matrixRanks[i];
    }
}

void generateRandomVector(unsigned int vectorWidth, unsigned int vectorHeight, H2Opus_Real *vector) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniformDouble(gen, vector, vectorWidth*vectorHeight);
    hiprandDestroyGenerator(gen);
}

void generateMaxRanks(unsigned int numLevels, unsigned int bucketSize, unsigned int *maxRanks) {
    for(unsigned int i = 0; i < numLevels - 2; ++i) {
        maxRanks[i] = bucketSize*(1 << i);
        if(i > 5) {
            maxRanks[i]/=4;
        }
    }
}