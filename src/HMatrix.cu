
#include "HMatrix.cuh"
#include "admissibilityCondition.cuh"
#include "boundingBoxes.h"
#include "config.h"
#include "helperKernels.cuh"
#include "kDTreeHelpers.cuh"
#include <hipcub/hipcub.hpp>
#include <functional>

void constructWeakAdmissibilityStruct(
    HMatrixStructure *HMatrixStruct, 
    unsigned int numberOfInputPoints, 
    unsigned int bucketSize,
    ADMISSIBILITY_CONDITION admissibilityCondition) {

        unsigned int dim = 2;
        for(unsigned int level = 0; level < HMatrixStruct->numLevels - 1; ++level) {
            unsigned int numTiles = 1 << (level + 1);
            HMatrixStruct->numTiles[level] = numTiles;

            for(unsigned int j = 0; j < numTiles; ++j) {
                int x;
                if(j%2 == 0) {
                    x = 1;
                }
                else {
                    x = -1;
                }
                unsigned int tileIndex = j*dim + j + x;
                HMatrixStruct->tileIndices[level][j + x] = CMIndextoMOIndex(dim, tileIndex);
            }
            
            dim <<= 1;
        }
}

void constructMatrixStruct_recursive(
    HMatrixStructure *HMatrixStruct,
    KDTreeBoundingBoxes BBox_u,
    KDTreeBoundingBoxes BBox_v,
    BoundingBox node_u,
    BoundingBox node_v,
    unsigned int dimensionOfInputPoints,
    unsigned int currentLevel,
    unsigned int maxDepth,
    float epsilon,
    std::function<bool(
        BoundingBox,
        BoundingBox,
        unsigned int,
        unsigned int,
        unsigned int,
        float)> isAdmissible) {

            bool isDiagonal = (node_u.index == node_v.index);
            bool isLeafNode = (currentLevel == maxDepth);

            if(isDiagonal && isLeafNode) {
                return;
            }
            else if(isLeafNode || isAdmissible(node_u, node_v, dimensionOfInputPoints, currentLevel, maxDepth, epsilon)) {
                // TODO: write to HMatrixStruct
                unsigned int numRows = 1<<currentLevel;
                unsigned int tileIndex = CMIndextoMOIndex(numRows, node_u.index*numRows + node_v.index);
                HMatrixStruct->tileIndices[currentLevel - 1][HMatrixStruct->numTiles[currentLevel - 1]++] = tileIndex;
                return;
            }
            else {
                constructMatrixStruct_recursive(
                    HMatrixStruct,
                    BBox_u,
                    BBox_v,
                    BBox_u.levels[currentLevel + 1].boundingBoxes[2*node_u.index],
                    BBox_v.levels[currentLevel + 1].boundingBoxes[2*node_v.index],
                    dimensionOfInputPoints,
                    currentLevel + 1,
                    maxDepth,
                    epsilon,
                    isAdmissible);

                constructMatrixStruct_recursive(
                    HMatrixStruct,
                    BBox_u,
                    BBox_v,
                    BBox_u.levels[currentLevel + 1].boundingBoxes[2*node_u.index + 1],
                    BBox_v.levels[currentLevel + 1].boundingBoxes[2*node_v.index],
                    dimensionOfInputPoints,
                    currentLevel + 1,
                    maxDepth,
                    epsilon,
                    isAdmissible);

                constructMatrixStruct_recursive(
                    HMatrixStruct,
                    BBox_u,
                    BBox_v,
                    BBox_u.levels[currentLevel + 1].boundingBoxes[2*node_u.index],
                    BBox_v.levels[currentLevel + 1].boundingBoxes[2*node_v.index + 1],
                    dimensionOfInputPoints,
                    currentLevel + 1,
                    maxDepth,
                    epsilon,
                    isAdmissible);

                constructMatrixStruct_recursive(
                    HMatrixStruct,
                    BBox_u,
                    BBox_v,
                    BBox_u.levels[currentLevel + 1].boundingBoxes[2*node_u.index + 1],
                    BBox_v.levels[currentLevel + 1].boundingBoxes[2*node_v.index + 1],
                    dimensionOfInputPoints,
                    currentLevel + 1,
                    maxDepth,
                    epsilon,
                    isAdmissible);

            }
}

void constructMatrixStruct(
    HMatrixStructure *HMatrixStruct,
    ADMISSIBILITY_CONDITION admissibilityCondition,
    KDTreeBoundingBoxes BBox1,
    KDTreeBoundingBoxes BBox2,
    unsigned int numberOfInputPoints,
    unsigned int dimensionOfInputPoints,
    unsigned int bucketSize,
    float epsilon = 5) {

        unsigned int maxDepth = __builtin_ctz(upperPowerOfTwo(numberOfInputPoints)/bucketSize);

        // call recursive function
        if(admissibilityCondition == BOX_CENTER_ADMISSIBILITY) {
            constructMatrixStruct_recursive(
                HMatrixStruct,
                BBox1,
                BBox2,
                BBox1.levels[0].boundingBoxes[0],
                BBox2.levels[0].boundingBoxes[0],
                dimensionOfInputPoints,
                0,
                maxDepth,
                epsilon,
                &BBoxCenterAdmissibility);
        }
        else if(admissibilityCondition == WEAK_ADMISSIBILITY) {
            constructMatrixStruct_recursive(
                HMatrixStruct,
                BBox1,
                BBox2,
                BBox1.levels[0].boundingBoxes[0],
                BBox2.levels[0].boundingBoxes[0],
                dimensionOfInputPoints,
                0,
                maxDepth,
                epsilon,
                &weakAdmissibility);
        }
}

void constructHMatrixStructure(
    HMatrixStructure *HMatrixStruct,
    unsigned int numberOfInputPoints,
    unsigned int dimensionOfInputPoints,
    unsigned int bucketSize,
    ADMISSIBILITY_CONDITION admissibilityCondition,
    KDTreeBoundingBoxes BBox1,
    KDTreeBoundingBoxes BBox2) {

        HMatrixStruct->numLevels = __builtin_ctz(numberOfInputPoints/bucketSize) + 1;
        HMatrixStruct->numTiles = (int*)malloc((HMatrixStruct->numLevels)*sizeof(int));
        HMatrixStruct->tileIndices = (int**)malloc((HMatrixStruct->numLevels)*sizeof(int*));

        for(unsigned int level = 0; level < HMatrixStruct->numLevels; ++level) {
            HMatrixStruct->numTiles[level] = 0;
            unsigned int numTiles = 1<<(level + 1);
            HMatrixStruct->tileIndices[level] = (int*)malloc(numTiles*(numTiles - 1)*sizeof(int));
        }

        constructMatrixStruct(
            HMatrixStruct,
            admissibilityCondition,
            BBox1,
            BBox2,
            numberOfInputPoints,
            dimensionOfInputPoints,
            bucketSize);

        printf("num levels: %d\n", HMatrixStruct->numLevels);
        for(unsigned int i = 0; i < HMatrixStruct->numLevels; ++i) {
            printf("num tiles in level: %d is: %d\n", i, HMatrixStruct->numTiles[i]);
            for(unsigned int j = 0; j < HMatrixStruct->numTiles[i]; ++j) {
                printf("    %d ", HMatrixStruct->tileIndices[i][j]);
            }
            printf("\n");
        }
}

void freeHMatrixStructure(HMatrixStructure &HMatrixStruct) {
    free(HMatrixStruct.numTiles);
    for(unsigned int i = 0; i < HMatrixStruct.numLevels - 1; ++i) {
        free(HMatrixStruct.tileIndices[i]);
    }
    free(HMatrixStruct.tileIndices);
}

void allocateAndCopyToHMatrixLevel(
    HMatrixLevel &matrixLevel, 
    int* ranks, 
    HMatrixStructure HMatrixStruct, 
    unsigned int level, 
    H2Opus_Real *A, H2Opus_Real *B, 
    int maxRows, int maxRank) {
        
        // matrixLevel.numTiles = HMatrixStruct.numTiles[level - 1];
        matrixLevel.level = level;

        // scan ranks array
        hipMalloc((void**) &matrixLevel.tileScanRanks, HMatrixStruct.numTiles[level - 1]*sizeof(int));
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, ranks, matrixLevel.tileScanRanks, HMatrixStruct.numTiles[level - 1]);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, ranks, matrixLevel.tileScanRanks, HMatrixStruct.numTiles[level - 1]);

        int *scanRanks = (int*)malloc(HMatrixStruct.numTiles[level - 1]*sizeof(int));
        hipMemcpy(scanRanks, matrixLevel.tileScanRanks, HMatrixStruct.numTiles[level - 1]*sizeof(int), hipMemcpyDeviceToHost);

        // allocate U and V
        hipMalloc((void**) &matrixLevel.U, static_cast<uint64_t>(scanRanks[HMatrixStruct.numTiles[level - 1] - 1])*maxRows*sizeof(H2Opus_Real));
        hipMalloc((void**) &matrixLevel.V, static_cast<uint64_t>(scanRanks[HMatrixStruct.numTiles[level - 1] - 1])*maxRows*sizeof(H2Opus_Real));

        // copy A and B to U and V
        for(unsigned int tile = 0; tile < HMatrixStruct.numTiles[level - 1]; ++tile) {
            int tileRank = (tile == 0) ? scanRanks[tile] : scanRanks[tile] - scanRanks[tile - 1];
            hipMemcpy(&matrixLevel.U[static_cast<uint64_t>(scanRanks[tile] - tileRank)*maxRows], &A[static_cast<uint64_t>(tile)*maxRows*maxRank], static_cast<uint64_t>(tileRank)*maxRows*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
            hipMemcpy(&matrixLevel.V[static_cast<uint64_t>(scanRanks[tile] - tileRank)*maxRows], &B[static_cast<uint64_t>(tile)*maxRows*maxRank], static_cast<uint64_t>(tileRank)*maxRows*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        }

        // copy tile indices from HMatrixStruct to here
        // hipMalloc((void**) &matrixLevel.tileIndices, matrixLevel.numTiles*sizeof(int));
        // hipMemcpy(matrixLevel.tileIndices, HMatrixStruct.tileIndices[level - 1], matrixLevel.numTiles*sizeof(int), hipMemcpyHostToDevice);
}

void freeHMatrixLevel(HMatrixLevel matrixLevel){
    // hipFree(matrixLevel.tileIndices);
    hipFree(matrixLevel.tileScanRanks);
    hipFree(matrixLevel.U);
    hipFree(matrixLevel.V);
}

void allocateHMatrix(HMatrix &matrix, TLR_Matrix mortonOrderedMatrix, int segmentSize, int numSegments, unsigned int numberOfInputPoints, unsigned int bucketSize, HMatrixStructure HMatrixStruct) {
    // TODO: consolidate bucket size and segment size
    hipMalloc((void**) &matrix.diagonalBlocks, segmentSize*segmentSize*numSegments*sizeof(H2Opus_Real));
    hipMemcpy(matrix.diagonalBlocks, mortonOrderedMatrix.diagonal, segmentSize*segmentSize*numSegments*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    // matrix.matrixStructure.numLevels = __builtin_ctz(numberOfInputPoints/bucketSize) + 1;
    matrix.levels = (HMatrixLevel*)malloc((matrix.matrixStructure.numLevels - 1)*sizeof(HMatrixLevel));

    // copy tlr tiles to HMatrix bottom level
    int *h_ranks = (int*)malloc(numSegments*numSegments*sizeof(int));
    int *h_scanRanks = (int*)malloc(numSegments*numSegments*sizeof(int));
    int *h_levelRanks = (int*)malloc(HMatrixStruct.numTiles[matrix.matrixStructure.numLevels - 2]*sizeof(int));

    hipMemcpy(h_ranks, mortonOrderedMatrix.blockRanks, numSegments*numSegments*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_scanRanks, mortonOrderedMatrix.blockOffsets, numSegments*numSegments*sizeof(int), hipMemcpyDeviceToHost);
    // matrix.levels[matrix.matrixStructure.numLevels - 2].numTiles = HMatrixStruct.numTiles[matrix.matrixStructure.numLevels - 2];
    matrix.levels[matrix.matrixStructure.numLevels - 2].level = matrix.matrixStructure.numLevels - 1;

    int rankSum = 0;
    for(unsigned int i = 0; i < matrix.matrixStructure.numTiles[matrix.matrixStructure.numLevels - 2]; ++i) {
        rankSum += h_ranks[HMatrixStruct.tileIndices[matrix.matrixStructure.numLevels - 2][i]];
        h_levelRanks[i] = h_ranks[HMatrixStruct.tileIndices[matrix.matrixStructure.numLevels - 2][i]];
    }

    // hipMalloc((void**) &matrix.levels[matrix.matrixStructure.numLevels - 2].tileIndices, matrix.levels[matrix.matrixStructure.numLevels - 2].numTiles*sizeof(int));
    // hipMemcpy(matrix.levels[matrix.matrixStructure.numLevels - 2].tileIndices, HMatrixStruct.tileIndices[matrix.matrixStructure.numLevels - 2], matrix.levels[matrix.matrixStructure.numLevels - 2].numTiles*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &matrix.levels[matrix.matrixStructure.numLevels - 2].U, rankSum*bucketSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &matrix.levels[matrix.matrixStructure.numLevels - 2].V, rankSum*bucketSize*sizeof(H2Opus_Real));

    hipMalloc((void**) &matrix.levels[matrix.matrixStructure.numLevels - 2].tileScanRanks, matrix.matrixStructure.numTiles[matrix.matrixStructure.numLevels - 2]*sizeof(int));    
    int *d_levelRanks;
    hipMalloc((void**) &d_levelRanks, HMatrixStruct.numTiles[matrix.matrixStructure.numLevels - 2]*sizeof(int));
    hipMemcpy(d_levelRanks, h_levelRanks, HMatrixStruct.numTiles[matrix.matrixStructure.numLevels - 2]*sizeof(int), hipMemcpyHostToDevice);
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_levelRanks, matrix.levels[matrix.matrixStructure.numLevels - 2].tileScanRanks, matrix.matrixStructure.numTiles[matrix.matrixStructure.numLevels - 2]);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_levelRanks, matrix.levels[matrix.matrixStructure.numLevels - 2].tileScanRanks, matrix.matrixStructure.numTiles[matrix.matrixStructure.numLevels - 2]);

    int tmp = 0;
    for(unsigned int i = 0; i < matrix.matrixStructure.numTiles[matrix.matrixStructure.numLevels - 2]; ++i) {
        hipMemcpy(&matrix.levels[matrix.matrixStructure.numLevels - 2].U[tmp*bucketSize], &mortonOrderedMatrix.U[h_scanRanks[HMatrixStruct.tileIndices[matrix.matrixStructure.numLevels - 2][i]]*bucketSize], h_levelRanks[i]*bucketSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        hipMemcpy(&matrix.levels[matrix.matrixStructure.numLevels - 2].V[tmp*bucketSize], &mortonOrderedMatrix.V[h_scanRanks[HMatrixStruct.tileIndices[matrix.matrixStructure.numLevels - 2][i]]*bucketSize], h_levelRanks[i]*bucketSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);

        tmp += h_levelRanks[i];
    }
}

void freeHMatrix(HMatrix &matrix) {
    hipFree(matrix.diagonalBlocks);
    for(unsigned int level = 1; level < matrix.matrixStructure.numLevels - 1; ++level) {
        freeHMatrixLevel(matrix.levels[level - 1]);
    }
}