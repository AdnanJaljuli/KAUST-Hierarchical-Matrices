#include "hip/hip_runtime.h"

#include "cutlassHMatrixXVector.cuh"
#include "HMatrix.cuh"

#include <chrono>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/util/command_line.h>
#include <cutlass/util/distribution.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/tensor_view_io.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/host/gemm_complex.h>
#include <cutlass/util/reference/device/gemm_complex.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_copy.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/tensor_norm.h>

struct Result {
	double runtime_ms;
	double initialization_time_ms;
	double gflops;
	cutlass::Status status;
	hipError_t error;
	bool passed;

	Result(
		double runtime_ms = 0,
		double initialization_time_ms = 0,
		double gflops = 0,
		cutlass::Status status = cutlass::Status::kSuccess,
		hipError_t error = hipSuccess
	):
	runtime_ms(runtime_ms), initialization_time_ms(initialization_time_ms), gflops(gflops),
	status(status), error(error), passed(true) { }
};

void preprocessGroupedGEMM(
	unsigned int numberOfInputPoints, unsigned int level, int numLevels,
	int problemCount, unsigned int bucketSize, unsigned int vectorWidth,
	HMatrixLevel matrixLevel, 
	std::vector<cutlass::gemm::GemmCoord> *h_problemSizes, cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> *d_problemSizes,
	std::vector<int64_t> *lda_host, std::vector<int64_t> *ldb_host, std::vector<int64_t> *ldc_host,
	cutlass::DeviceAllocation<int64_t> *lda, cutlass::DeviceAllocation<int64_t> *ldb, cutlass::DeviceAllocation<int64_t> *ldc,
	H2Opus_Real *AMatrix, H2Opus_Real *BMatrix, H2Opus_Real *CMatrix,
	std::vector<H2Opus_Real*> *ptr_A_host, std::vector<H2Opus_Real*> *ptr_B_host, std::vector<H2Opus_Real*> *ptr_C_host,
	cutlass::DeviceAllocation<H2Opus_Real *> *ptr_A, cutlass::DeviceAllocation<H2Opus_Real *> *ptr_B, cutlass::DeviceAllocation<H2Opus_Real *> *ptr_C,
	unsigned int iteration) {

		int *h_tileScanRanks = (int*)malloc(problemCount*sizeof(int));
		hipMemcpy(h_tileScanRanks, matrixLevel.tileScanRanks, problemCount*sizeof(int), hipMemcpyDeviceToHost);

		int previousTileScanRank = 0;
		for (unsigned int tile = 0; tile < problemCount; ++tile) {
			int tileRank = h_tileScanRanks[tile] - previousTileScanRank;

			unsigned int tileDimension = 1<<(numLevels - (level + 1))*bucketSize;
			if(iteration == 0) {
				cutlass::gemm::GemmCoord problem(tileRank, vectorWidth, tileDimension);
				h_problemSizes->push_back(problem);
			}
			else {
				cutlass::gemm::GemmCoord problem(tileDimension, vectorWidth, tileRank);
				h_problemSizes->push_back(problem);
			}

			if(iteration == 0) {
				lda_host->at(tile) = (int64_t)tileRank;
				ldb_host->at(tile) = (int64_t)numberOfInputPoints;
			}
			else {
				lda_host->at(tile) = (int64_t)numberOfInputPoints;
				ldb_host->at(tile) = (int64_t)tileRank;
			}
			ldc_host->at(tile) = (int64_t)numberOfInputPoints;

			ptr_A_host->at(tile) = AMatrix + previousTileScanRank*tileDimension;
			ptr_B_host->at(tile) = BMatrix + tile*tileDimension;
			ptr_C_host->at(tile) = CMatrix + tile*tileDimension;

			previousTileScanRank += tileRank;
		}

		d_problemSizes->reset(problemCount);
 		d_problemSizes->copy_from_host(h_problemSizes->data());
		h_problemSizes->clear();

		lda->copy_from_host(lda_host->data());
		ldb->copy_from_host(ldb_host->data());
		ldc->copy_from_host(ldc_host->data());

		ptr_A->copy_from_host(ptr_A_host->data());
		ptr_B->copy_from_host(ptr_B_host->data());
		ptr_C->copy_from_host(ptr_C_host->data());

		free(h_tileScanRanks);
}

hipError_t cutlass_grouped_dgemm() {
}

// __global__ void printOutputMatrix(unsigned int numberOfInputPoints, unsigned int  vectorWidth, H2Opus_Real *resultVectors) {
// 	for(unsigned int i = 0; i < vectorWidth; ++i) {
// 		for(unsigned int j = 0; j < numberOfInputPoints; ++j) {
// 			printf("%lf ", resultVectors[i*numberOfInputPoints + j]);
// 		}
// 		printf("\n");
// 	}
// 	printf("\n");
// }

hipError_t cutlassHierarchicalXVec(
    unsigned int numberOfInputPoints, unsigned int  bucketSize, 
    unsigned int  numSegments, unsigned int vectorWidth, HMatrix hierarchicalMatrix,
    H2Opus_Real *inputVectors, H2Opus_Real *bufferVectors, H2Opus_Real *resultVectors) {

		using ElementInput = H2Opus_Real;
		using ElementOutput = H2Opus_Real;
		using ElementAccumulator = H2Opus_Real;

		using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
			ElementInput, 
			cutlass::layout::ColumnMajor, 
			cutlass::ComplexTransform::kNone,
			1,
			ElementInput,
			cutlass::layout::RowMajor, 
			cutlass::ComplexTransform::kNone,
			1,
			ElementOutput, cutlass::layout::ColumnMajor,
			ElementAccumulator, 
			cutlass::arch::OpClassTensorOp, 
			cutlass::arch::Sm80,
			cutlass::gemm::GemmShape<64, 64, 16>,
			cutlass::gemm::GemmShape<32, 32, 16>,
			cutlass::gemm::GemmShape<8, 8, 4>,
			cutlass::epilogue::thread::LinearCombination<
				ElementOutput, 1,
				ElementAccumulator, ElementAccumulator>,
			cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
			4>::GemmKernel;

		using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

		typename Gemm::EpilogueOutputOp::Params epilogue_1(1.0f, 0.0f);
		typename Gemm::EpilogueOutputOp::Params epilogue_2(1.0f, 1.0f);

		std::vector<cutlass::gemm::GemmCoord> h_problemSizes;
		cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> d_problemSizes;

		std::vector<int64_t> lda_host(numSegments);
		std::vector<int64_t> ldb_host(numSegments);
		std::vector<int64_t> ldc_host(numSegments);

		cutlass::DeviceAllocation<int64_t> lda;
		cutlass::DeviceAllocation<int64_t> ldb;
		cutlass::DeviceAllocation<int64_t> ldc;
		lda.reset(numSegments);
		ldb.reset(numSegments);
		ldc.reset(numSegments);

		std::vector<ElementInput*> ptr_A_host(numSegments);
		std::vector<ElementInput*> ptr_B_host(numSegments);
		std::vector<ElementAccumulator*> ptr_C_host(numSegments);

		cutlass::DeviceAllocation<ElementInput *> ptr_A;
		cutlass::DeviceAllocation<ElementInput *> ptr_B;
		cutlass::DeviceAllocation<ElementAccumulator *> ptr_C;
		ptr_A.reset(numSegments);
		ptr_B.reset(numSegments);
		ptr_C.reset(numSegments);

      	// loop over levels
		for(unsigned int level = hierarchicalMatrix.numLevels - 2; level > 0; --level) {
			// preprocess each level
			int problemCount = hierarchicalMatrix.levels[level - 1].numTiles;

			// V*inputVectors
			preprocessGroupedGEMM(numberOfInputPoints, level, hierarchicalMatrix.numLevels,
				problemCount, bucketSize, vectorWidth,
				hierarchicalMatrix.levels[level - 1],
				&h_problemSizes, &d_problemSizes,
				&lda_host, &ldb_host, &ldc_host,
				&lda, &ldb, &ldc,
				hierarchicalMatrix.levels[level - 1].V, inputVectors, bufferVectors,
				&ptr_A_host, &ptr_B_host, &ptr_C_host,
				&ptr_A, &ptr_B, &ptr_C,
				0);

			int threadblockCount = Gemm::sufficient(h_problemSizes.data(), problemCount);
			if (!threadblockCount) {
				printf("Active CUDA device lacks hardware resources to run CUTLASS Grouped GEMM kernel.");
			}

			typename Gemm::Arguments args_1(
				d_problemSizes.get(),
				problemCount,
				threadblockCount,
				epilogue_1,
				ptr_A.get(),
				ptr_B.get(),
				ptr_C.get(),
				ptr_C.get(),
				lda.get(),
				ldb.get(),
				ldc.get(),
				ldc.get(),
				h_problemSizes.data() // ptr to where data in vector starts
			);

			Gemm gemm_1;
			size_t workspace_size = gemm_1.get_workspace_size(args_1);
			cutlass::DeviceAllocation<uint8_t> workspace_1(workspace_size);
			gemm_1.initialize(args_1, workspace_1.get());
			gemm_1.run();

			// U*results
			preprocessGroupedGEMM(numberOfInputPoints, level, hierarchicalMatrix.numLevels,
				problemCount, bucketSize, vectorWidth,
				hierarchicalMatrix.levels[level - 1],
				&h_problemSizes, &d_problemSizes,
				&lda_host, &ldb_host, &ldc_host,
				&lda, &ldb, &ldc,
				hierarchicalMatrix.levels[level - 1].U, bufferVectors, resultVectors,
				&ptr_A_host, &ptr_B_host, &ptr_C_host,
				&ptr_A, &ptr_B, &ptr_C,
				1);

			typename Gemm::Arguments args_2(
				d_problemSizes.get(),
				problemCount,
				threadblockCount,
				epilogue_1,
				ptr_A.get(),
				ptr_B.get(),
				ptr_C.get(),
				ptr_C.get(),
				lda.get(),
				ldb.get(),
				ldc.get(),
				ldc.get(),
				h_problemSizes.data() // ptr to where data in vector starts
			);

			Gemm gemm_2;
			workspace_size = gemm_2.get_workspace_size(args_2);
			cutlass::DeviceAllocation<uint8_t> workspace_2(workspace_size);
			gemm_2.initialize(args_2, workspace_2.get());
			gemm_2.run();
		}

		// printOutputMatrix <<< 1, 1 >>> (numberOfInputPoints, vectorWidth, resultVectors);

		hipError_t result;
		return result;
}