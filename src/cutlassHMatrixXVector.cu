
#include "cutlassHMatrixXVector.cuh"
#include "HMatrix.cuh"

#include <chrono>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/util/command_line.h>
#include <cutlass/util/distribution.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/tensor_view_io.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/host/gemm_complex.h>
#include <cutlass/util/reference/device/gemm_complex.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_copy.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/tensor_norm.h>

struct Result {
	double runtime_ms;
	double initialization_time_ms;
	double gflops;
	cutlass::Status status;
	hipError_t error;
	bool passed;

	Result(
		double runtime_ms = 0,
		double initialization_time_ms = 0,
		double gflops = 0,
		cutlass::Status status = cutlass::Status::kSuccess,
		hipError_t error = hipSuccess
	):
	runtime_ms(runtime_ms), initialization_time_ms(initialization_time_ms), gflops(gflops),
	status(status), error(error), passed(true) { }
};

void findProblemSizes(unsigned int numberOfInputPoints, unsigned int level, int numLevels, int problemCount, unsigned int bucketSize, unsigned int vectorWidth, HMatrixLevel matrixLevel, std::vector<cutlass::gemm::GemmCoord> *problemSizes) {
	int previousTileScanRank = 0;
	
	int *h_tileScanRanks = (int*)malloc(problemCount*sizeof(int));
	hipMemcpy(h_tileScanRanks, matrixLevel.tileScanRanks, problemCount*sizeof(int), hipMemcpyDeviceToHost);

	for (unsigned int tile = 0; tile < problemCount; ++tile) {
		int tileRank = h_tileScanRanks[tile] - previousTileScanRank;
		unsigned int tileDimension = 1<<(numLevels - (level + 1))*bucketSize;

		cutlass::gemm::GemmCoord problem(tileRank, vectorWidth, tileDimension);
		problemSizes->push_back(problem);

		previousTileScanRank += tileRank;
	}

	free(h_tileScanRanks);
}

void fillLeadingDimensions(unsigned int numberOfInputPoints, int problemCount, HMatrixLevel matrixLevel, std::vector<int64_t> *lda_host, std::vector<int64_t> *ldb_host, std::vector<int64_t> *ldc_host, cutlass::DeviceAllocation<int64_t> *lda, cutlass::DeviceAllocation<int64_t> *ldb, cutlass::DeviceAllocation<int64_t> *ldc) {
	int previousTileScanRank = 0;
	int *h_tileScanRanks = (int*)malloc(problemCount*sizeof(int));
	hipMemcpy(h_tileScanRanks, matrixLevel.tileScanRanks, problemCount*sizeof(int), hipMemcpyDeviceToHost);
	for (unsigned int tile = 0; tile < problemCount; ++tile) {
		int tileRank = h_tileScanRanks[tile] - previousTileScanRank;
		lda_host->at(tile) = (int64_t)tileRank;
		ldb_host->at(tile) = (int64_t)numberOfInputPoints;
		ldc_host->at(tile) = (int64_t)numberOfInputPoints;

		previousTileScanRank += tileRank;
	}
	free(h_tileScanRanks);

	lda->copy_from_host(lda_host->data());
	ldb->copy_from_host(lda_host->data());
	ldc->copy_from_host(lda_host->data());
}

void fillMatrixPtrs(unsigned int numberOfInputPoints, int problemCount, int numLevels, unsigned int level, unsigned int bucketSize, HMatrixLevel matrixLevel, H2Opus_Real *AMatrix, H2Opus_Real *BMatrix, H2Opus_Real *CMatrix, std::vector<H2Opus_Real*> *ptr_A_host, std::vector<H2Opus_Real*> *ptr_B_host, std::vector<H2Opus_Real*> *ptr_C_host, cutlass::DeviceAllocation<H2Opus_Real *> *ptr_A, cutlass::DeviceAllocation<H2Opus_Real *> *ptr_B, cutlass::DeviceAllocation<H2Opus_Real *> *ptr_C) {
	int previousTileScanRank = 0;
	int *h_tileScanRanks = (int*)malloc(problemCount*sizeof(int));
	hipMemcpy(h_tileScanRanks, matrixLevel.tileScanRanks, problemCount*sizeof(int), hipMemcpyDeviceToHost);
	unsigned int tileDimension = 1<<(numLevels - (level + 1))*bucketSize;

	for (unsigned int tile = 0; tile < problemCount; ++tile) {
		int tileRank = h_tileScanRanks[tile] - previousTileScanRank;
		ptr_A_host->at(tile) = AMatrix + previousTileScanRank*tileDimension;
		ptr_B_host->at(tile) = BMatrix + tile*tileDimension;
		ptr_C_host->at(tile) = CMatrix + tile*tileDimension;

		previousTileScanRank += tileRank;
	}
	free(h_tileScanRanks);

	ptr_A->copy_from_host(ptr_A_host->data());
	ptr_B->copy_from_host(ptr_B_host->data());
	ptr_C->copy_from_host(ptr_C_host->data());
}

hipError_t cutlass_grouped_dgemm() {
}

hipError_t cutlassHierarchicalXVec(
    unsigned int numberOfInputPoints, unsigned int  bucketSize, 
    unsigned int  numSegments, unsigned int vectorWidth, HMatrix hierarchicalMatrix,
    H2Opus_Real *inputVectors, H2Opus_Real *resultVectors) {

		H2Opus_Real *d_bufferVectors;
		hipMalloc((void**) &d_bufferVectors, numberOfInputPoints*vectorWidth*sizeof(H2Opus_Real));
		
		std::vector<int64_t> lda_host;
		std::vector<int64_t> ldb_host;
		std::vector<int64_t> ldc_host;
		lda_host.resize(numSegments);
		ldb_host.resize(numSegments);
		ldc_host.resize(numSegments);

		cutlass::DeviceAllocation<int64_t> lda;
		cutlass::DeviceAllocation<int64_t> ldb;
		cutlass::DeviceAllocation<int64_t> ldc;
		lda.reset(numSegments);
		ldb.reset(numSegments);
		ldc.reset(numSegments);

		using ElementInput = H2Opus_Real;
		using ElementOutput = H2Opus_Real;
		using ElementAccumulator = H2Opus_Real;
		using LayoutA = cutlass::layout::ColumnMajor;
		using LayoutB = cutlass::layout::ColumnMajor;
		using LayoutC = cutlass::layout::ColumnMajor;

		using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
			ElementInput, 
			cutlass::layout::ColumnMajor, 
			cutlass::ComplexTransform::kNone,
			1,
			ElementInput,
			cutlass::layout::ColumnMajor, 
			cutlass::ComplexTransform::kNone,
			1,
			ElementOutput, cutlass::layout::ColumnMajor,
			ElementAccumulator, 
			cutlass::arch::OpClassTensorOp, 
			cutlass::arch::Sm80,
			cutlass::gemm::GemmShape<64, 64, 16>,
			cutlass::gemm::GemmShape<32, 32, 16>,
			cutlass::gemm::GemmShape<8, 8, 4>,
			cutlass::epilogue::thread::LinearCombination<
				ElementOutput, 1,
				ElementAccumulator, ElementAccumulator>,
			cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
			4>::GemmKernel;

    	using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

		std::vector<ElementInput*> ptr_A_host(numSegments);
		std::vector<ElementInput*> ptr_B_host(numSegments);
		std::vector<ElementAccumulator*> ptr_C_host(numSegments);

		cutlass::DeviceAllocation<ElementInput *> ptr_A;
		cutlass::DeviceAllocation<ElementInput *> ptr_B;
		cutlass::DeviceAllocation<ElementAccumulator *> ptr_C;
		ptr_A.reset(numSegments);
		ptr_B.reset(numSegments);
		ptr_C.reset(numSegments);

		std::vector<cutlass::gemm::GemmCoord> h_problemSizes;
		cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> d_problemSizes;

		typename Gemm::EpilogueOutputOp::Params epilogue_1(1.0f, 0.0f);
		typename Gemm::EpilogueOutputOp::Params epilogue_2(1.0f, 1.0f);

      	// loop over levels
		for(unsigned int level = hierarchicalMatrix.numLevels - 2; level > 0; --level) {
			int problemCount = hierarchicalMatrix.levels[level - 1].numTiles;
			findProblemSizes(numberOfInputPoints, level, hierarchicalMatrix.numLevels, problemCount, bucketSize, vectorWidth, hierarchicalMatrix.levels[level - 1], &h_problemSizes);
			d_problemSizes.reset(problemCount);
    		d_problemSizes.copy_from_host(h_problemSizes.data());

			fillLeadingDimensions(numberOfInputPoints, problemCount, hierarchicalMatrix.levels[level - 1], &lda_host, &ldb_host, &ldc_host, &lda, &ldb, &ldc);

			fillMatrixPtrs(numberOfInputPoints, problemCount, hierarchicalMatrix.numLevels, level, bucketSize, hierarchicalMatrix.levels[level - 1], hierarchicalMatrix.levels[level - 1].U, inputVectors, d_bufferVectors, &ptr_A_host, &ptr_B_host, &ptr_C_host, &ptr_A, &ptr_B, &ptr_C);

			// TODO: preprocess each level
			int threadblockCount = Gemm::sufficient(h_problemSizes.data(), problemCount);
			if (!threadblockCount) {
				printf("Active CUDA device lacks hardware resources to run CUTLASS Grouped GEMM kernel.");
			}

			typename Gemm::Arguments args(
				d_problemSizes.get(),
				problemCount,
				threadblockCount,
				epilogue_1,
				ptr_A.get(),
				ptr_B.get(),
				ptr_C.get(),
				ptr_C.get(),
				lda.get(),
				ldb.get(),
				ldc.get(),
				ldc.get(),
				h_problemSizes.data() // ptr to where data in vector starts
			);

			Gemm gemm;
			size_t workspace_size = gemm.get_workspace_size(args);
			cutlass::DeviceAllocation<uint8_t> workspace(workspace_size);

			gemm.initialize(args, workspace.get());

			gemm.run();
		}

		hipFree(d_bufferVectors);
		hipError_t result;
		return result;
}