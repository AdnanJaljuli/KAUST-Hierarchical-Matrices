#include "hip/hip_runtime.h"

#include "buildHMatrix_helpers.cuh"
#include "helperKernels.cuh"

#include <algorithm>
#include <vector>
#include <utility>

__global__ void fillLRARAArrays(
    int batchSize,
    int maxRows,
    int* d_rowsBatch, int* d_colsBatch,
    int* d_LDABatch, int* d_LDBBatch) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < batchSize){
        d_rowsBatch[i] = maxRows;
        d_colsBatch[i] = maxRows;
        d_LDABatch[i] = maxRows;
        d_LDBBatch[i] = maxRows;
    }
}

void generateHMatMaxRanks(unsigned int numLevels, unsigned int tileSize, std::vector<unsigned int> *maxRanks) {
    for(unsigned int i = 0; i < numLevels - 2; ++i) {
        maxRanks->push_back(tileSize*(1 << i));
    }
}

std::pair<int, int> getTilesInPiece(
    std::vector<int> tileIndices,
    unsigned int tileLevel,
    unsigned int pieceMortonIndex, unsigned int pieceLevel) {

        // unsigned int levelDiff = tileLevel/pieceLevel;
        // unsigned int numBLocksInPieceAxis = 1<<(levelDiff - 1);
        unsigned int numBLocksInPieceAxis = (1<<tileLevel)/(1<<pieceLevel);
        unsigned int left = pieceMortonIndex*numBLocksInPieceAxis*numBLocksInPieceAxis;
        unsigned int right = (pieceMortonIndex + 1)*numBLocksInPieceAxis*numBLocksInPieceAxis - 1;

        // binary search
        std::vector<int>::iterator lower = lower_bound(tileIndices.begin(), tileIndices.end(), left);
        std::vector<int>::iterator upper = upper_bound(tileIndices.begin(), tileIndices.end(), right);

        std::pair<int, int> ans;
        ans.first = upper - lower;
        ans.second = lower - tileIndices.begin();

        return ans;
}

template <class T>
__global__ void fillBatchPtrs(
    T **U, T **V,
    T *UPtr,
    T *VPtr,
    int *tileOffsets,
    int batchSize,
    int tileSize,
    int batchUnitSize,
    int *tileIndices,
    int tileLevel) {

        unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

        if(i == 0) {
            printf("batchUnitSize: %d\n", batchUnitSize);
            printf("batchSize: %d\n", batchSize);
            printf("tileSize: %d\n", tileSize);
            printf("tileLevel: %d\n", tileLevel);
            printf("tileIndices\n");
            for(unsigned int j = 0; j < batchSize; ++j) {
                printf("%d ", tileIndices[j]);
            }
            printf("\n\n");
        }

        if(i < batchSize) {
            unsigned int numTilesInAxis = 1<<tileLevel;
            // unsigned int tileCol, tileRow;
            // morton2columnMajor(tileIndices[i], tileCol, tileRow);

            if(blockIdx.y == 0) {
                // tilePtrs->d_U[i] = &UPtr[
                //     static_cast<uint64_t>(
                //         tileOffsets[tileCol*numTilesInAxis*batchUnitSize*batchUnitSize +
                //         tileRow*batchUnitSize*batchUnitSize])*
                //     tileSize];
                printf("tile indices: %d\n", tileIndices[i]);
                printf("tile offsets: %d\n", tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize);
                printf("uptr %lf\n", UPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize]);
                // T *s = &UPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize];
                // printf("r: %d\n", s);
                // H2Opus_Real p = 9;
                // tilePtrs.d_U[i] = &p;
                U[i] = &UPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize];
            }
            else {
                // tilePtrs->d_V[i] = &VPtr[
                //     static_cast<uint64_t>(
                //         tileOffsets[tileCol*numTilesInAxis*batchUnitSize*batchUnitSize + 
                //         tileRow*batchUnitSize*batchUnitSize])*
                //     tileSize];
                printf("tile indices: %d\n", tileIndices[i]);
                printf("tile offsets: %d\n", tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize);
                printf("vptr %lf\n", VPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize]);
                // T *s = &VPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize];
                // printf("r: %d\n", s);
                // H2Opus_Real p = 9;
                // tilePtrs.d_V[i] = &p;
                V[i] = &VPtr[tileOffsets[tileIndices[i]*batchUnitSize*batchUnitSize]*tileSize];
            }
        }
}

template <class T>
void allocateTilePtrs(
    int batchSize,
    int batchUnitSize,
    int tileSize,
    int tileLevel,
    int *d_tileIndices,
    LevelTilePtrs &tilePtrs,
    TLR_Matrix TLRPiece) {

        hipMalloc((void**) &tilePtrs.d_U, batchSize*sizeof(T*));
        hipMalloc((void**) &tilePtrs.d_V, batchSize*sizeof(T*));

        T *d_UPtr = thrust::raw_pointer_cast(&TLRPiece.d_U[0]);
        T *d_VPtr = thrust::raw_pointer_cast(&TLRPiece.d_V[0]);

        printf("d_Usize: %d\n", TLRPiece.d_U.size());
        printf("d_Vsize: %d\n", TLRPiece.d_V.size());
        printf("\n");

        dim3 numThreadsPerBlock(1024);
        dim3 numBlocks((batchSize + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x, 2);
        fillBatchPtrs <T> <<< numBlocks, numThreadsPerBlock >>> (
            tilePtrs.d_U,
            tilePtrs.d_V,
            d_UPtr,
            d_VPtr,
            TLRPiece.d_tileOffsets,
            batchSize,
            tileSize,
            batchUnitSize,
            d_tileIndices,
            tileLevel);
}

template void allocateTilePtrs <H2Opus_Real> (
    int batchSize,
    int batchUnitSize,
    int tileSize,
    int tileLevel,
    int *d_tileIndices,
    LevelTilePtrs &tilePtrs,
    TLR_Matrix TLRPiece);

template <class T>
void freeLevelTilePtrs(LevelTilePtrs tilePtrs) {
    hipFree(tilePtrs.d_U);
    hipFree(tilePtrs.d_V);
}

__global__ void getRanks_kernel(int *blockRanks, int *blockScanRanks, int size) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < size) {
        int prevScanRanks = (i == 0) ? 0 : blockScanRanks[i - 1];
        blockRanks[i] = blockScanRanks[i] - prevScanRanks;
    }
}

void getRanks(int *d_blockRanks, int *d_blockScanRanks, int size) {
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (size + numThreadsPerBlock - 1)/numThreadsPerBlock;
    getRanks_kernel <<< numBlocks, numThreadsPerBlock >>> (d_blockRanks, d_blockScanRanks, size);
}

__global__ void fillScanRankPtrs(int **d_scanRanksPtrs, int *d_scanRanks, int batchUnitSize, int batchSize) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < batchSize) {
        d_scanRanksPtrs[i] = &d_scanRanks[i*batchUnitSize*batchUnitSize];
    }
}

void generateScanRanks(int batchSize, int batchUnitSize, int *ranks, int *scanRanks, int **scanRanksPtrs, int *levelTileIndices) {
    // TODO: we already have a scanRanks array of all the ranks in the MOMatrix. Use that one instead of this
    for(unsigned int batch = 0; batch < batchSize; ++batch) {
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, ranks + levelTileIndices[batch]*batchUnitSize*batchUnitSize, scanRanks + batch*batchUnitSize*batchUnitSize, batchUnitSize*batchUnitSize);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, ranks + levelTileIndices[batch]*batchUnitSize*batchUnitSize, scanRanks + batch*batchUnitSize*batchUnitSize, batchUnitSize*batchUnitSize);
        hipFree(d_temp_storage);
    }

    // fillScanRanksPtrs
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (batchSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillScanRankPtrs <<< numBlocks, numThreadsPerBlock >>> (scanRanksPtrs, scanRanks, batchUnitSize, batchSize);
}