#include "hip/hip_runtime.h"

#include "buildTLRMatrixPiece.cuh"
#include "buildTLRMatrixPiece_helpers.cuh"
#include "helperKernels.cuh"
#include "kDTree.cuh"
#include "hipblas.h"
#include "kblas.h"
#include "batch_rand.h"
#include "magma_auxiliary.h"

#include <assert.h>
#include <hipcub/hipcub.hpp>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

__global__ void printK(int *ranksOutput, int batchCount) {
    for(unsigned int i = 0; i < batchCount; ++i) {
        printf("%d ", ranksOutput[i]);
    }
    printf("\n");
}

template <class T>
void buildTLRMatrixPiece(
    TLR_Matrix *matrix,
    KDTree kdtree,
    T* d_pointCloud,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis,
    T tol) {

        magma_init();

        int *d_tmpRankSum, *d_tmpRankSum2;
        hipMalloc((void**) &d_tmpRankSum, sizeof(int));
        hipMalloc((void**) &d_tmpRankSum2, sizeof(int));
        uint64_t rankSum = 0;
        int totalMem;
        int ARA_R = 10;
        matrix->tileSize = kdtree.maxLeafSize;
        matrix->numTilesInAxis = (upperPowerOfTwo(kdtree.N)/numPiecesInAxis)/matrix->tileSize;
        bool isDiagonal = isPieceDiagonal(pieceMortonIndex);
        unsigned int batchCount = isDiagonal ? matrix->numTilesInAxis - 1: matrix->numTilesInAxis;
        unsigned int maxRank = matrix->tileSize>>1;
        assert(numPiecesInAxis <= matrix->numTilesInAxis);

        printf("batch count: %d  isDiagonal: %d  tile size: %d  numTilesInAxis: %d\n", batchCount, isDiagonal, matrix->tileSize, matrix->numTilesInAxis);

        thrust::device_vector<int> d_sortBits(batchCount*maxRank*matrix->tileSize);

        int *d_rowsBatch, *d_colsBatch, *d_ranksOutput;
        int *d_LDMBatch, *d_LDABatch, *d_LDBBatch;
        T *d_UOutput, *d_VOutput;
        T **d_MPtrs, **d_UOutputPtrs, **d_VOutputPtrs;
        hipMalloc((void**) &d_rowsBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_colsBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_ranksOutput, batchCount*matrix->numTilesInAxis*sizeof(int));
        hipMalloc((void**) &d_LDMBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_LDABatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_LDBBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_UOutput, batchCount*matrix->tileSize*maxRank*sizeof(T));
        hipMalloc((void**) &d_VOutput, batchCount*matrix->tileSize*maxRank*sizeof(T));
        hipMalloc((void**) &d_MPtrs, batchCount*sizeof(T*));
        hipMalloc((void**) &d_UOutputPtrs, batchCount*sizeof(T*));
        hipMalloc((void**) &d_VOutputPtrs, batchCount*sizeof(T*));

        fillARAHelpers(
            batchCount, matrix->tileSize, 
            d_rowsBatch, d_colsBatch, 
            d_LDMBatch, d_LDABatch, d_LDBBatch);

        kblasHandle_t kblasHandle;
        kblasRandState_t randState;
        kblasCreate(&kblasHandle);
        kblasInitRandState(kblasHandle, &randState, 1 << 15, 0);
        kblasEnableMagma(kblasHandle);
        kblas_ara_batch_wsquery<T>(kblasHandle, matrix->tileSize, batchCount);
        kblasAllocateWorkspace(kblasHandle);

        T *d_denseTileCol;
        int* d_colScanRanks;
        hipMalloc((void**) &d_denseTileCol, batchCount*matrix->tileSize*matrix->tileSize*sizeof(T));
        hipMalloc((void**) &d_colScanRanks, batchCount*sizeof(int));

        for(unsigned int tileColIdx = 0; tileColIdx < matrix->numTilesInAxis; ++tileColIdx) {
            generateDenseTileCol <T> (
                d_denseTileCol,
                d_pointCloud,
                kdtree,
                tileColIdx,
                matrix->tileSize,
                batchCount,
                pieceMortonIndex, numPiecesInAxis,
                matrix->numTilesInAxis,
                isDiagonal);

            generateArrayOfPointersT<T>(d_denseTileCol, d_MPtrs, matrix->tileSize*matrix->tileSize, batchCount, 0);
            generateArrayOfPointersT<T>(d_UOutput, d_UOutputPtrs, matrix->tileSize*maxRank, batchCount, 0);
            generateArrayOfPointersT<T>(d_VOutput, d_VOutputPtrs, matrix->tileSize*maxRank, batchCount, 0);
            hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

            int kblas_ara_return = kblas_ara_batch(
                kblasHandle, d_rowsBatch, d_colsBatch, d_MPtrs, d_LDMBatch, 
                d_UOutputPtrs, d_LDABatch, d_VOutputPtrs, d_LDBBatch, d_ranksOutput + tileColIdx*batchCount,
                tol, matrix->tileSize, matrix->tileSize, maxRank, 16, ARA_R, randState, 0, batchCount);
            assert(kblas_ara_return == 1);

            unsigned int numThreadsPerBlock = 1024;
            unsigned int numBlocks = (batchCount*maxRank*matrix->tileSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
            int *d_sortBitsPtr = thrust::raw_pointer_cast(d_sortBits.data());
            fillSortBits <<< numBlocks, numThreadsPerBlock >>> 
                (batchCount*maxRank*matrix->tileSize, maxRank*matrix->tileSize, d_sortBitsPtr, d_ranksOutput + tileColIdx*batchCount);
            
            void *d_temp_storage = NULL;
            size_t temp_storage_bytes = 0;
            hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_ranksOutput + tileColIdx*batchCount, d_tmpRankSum, batchCount);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_ranksOutput + tileColIdx*batchCount, d_tmpRankSum, batchCount);
            int tmpRankSum;
            hipMemcpy(&tmpRankSum, d_tmpRankSum, sizeof(int), hipMemcpyDeviceToHost);
            printf("rank sum: %d\n", tmpRankSum);

            matrix->d_U.resize((rankSum + tmpRankSum)*matrix->tileSize);
            matrix->d_V.resize((rankSum + tmpRankSum)*matrix->tileSize);
            
            T *d_UPtr = thrust::raw_pointer_cast(matrix->d_U.data());
            d_temp_storage = NULL;
            temp_storage_bytes = 0;
            hipcub::DevicePartition::Flagged(
                d_temp_storage, temp_storage_bytes,
                d_UOutput, d_sortBitsPtr, d_UPtr + rankSum*matrix->tileSize, d_tmpRankSum2, batchCount*maxRank*matrix->tileSize);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DevicePartition::Flagged(
                d_temp_storage, temp_storage_bytes,
                d_UOutput, d_sortBitsPtr, d_UPtr + rankSum*matrix->tileSize, d_tmpRankSum2, batchCount*maxRank*matrix->tileSize);

            T *d_VPtr = thrust::raw_pointer_cast(matrix->d_V.data());
            d_temp_storage = NULL;
            temp_storage_bytes = 0;
            hipcub::DevicePartition::Flagged(
                d_temp_storage, temp_storage_bytes,
                d_VOutput, d_sortBitsPtr, d_VPtr + rankSum*matrix->tileSize, d_tmpRankSum2, batchCount*maxRank*matrix->tileSize);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
            hipcub::DevicePartition::Flagged(
                d_temp_storage, temp_storage_bytes,
                d_VOutput, d_sortBitsPtr, d_VPtr + rankSum*matrix->tileSize, d_tmpRankSum2, batchCount*maxRank*matrix->tileSize);

            printK <<< 1, 1 >>> (d_ranksOutput + tileColIdx*batchCount, batchCount);
            rankSum += tmpRankSum;
        }

        hipFree(d_denseTileCol);
        hipFree(d_rowsBatch);
        hipFree(d_colsBatch);
        hipFree(d_LDMBatch);
        hipFree(d_LDABatch);
        hipFree(d_LDBBatch);
        hipFree(d_MPtrs);
        hipFree(d_UOutputPtrs);
        hipFree(d_VOutputPtrs);
        hipFree(d_UOutput);
        hipFree(d_VOutput);
        hipFree(d_tmpRankSum);
        hipFree(d_tmpRankSum2);
        d_sortBits.clear();
        
        kblasFreeWorkspace(kblasHandle);
        kblasDestroy(&kblasHandle);
        kblasHandle = NULL;
        kblasDestroyRandState(randState);
        magma_finalize();
}

template void buildTLRMatrixPiece<H2Opus_Real>(
    TLR_Matrix *matrix,
    KDTree kdtree,
    H2Opus_Real* d_dataset,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis,
    H2Opus_Real tol);
