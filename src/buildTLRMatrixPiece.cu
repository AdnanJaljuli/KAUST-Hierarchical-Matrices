
#include "buildTLRMatrixPiece.cuh"
#include "buildTLRMatrixPiece_helpers.cuh"
#include "helperKernels.cuh"
#include "kDTree.cuh"
#include "hipblas.h"
#include "kblas.h"
#include "batch_rand.h"
#include "magma_auxiliary.h"

#include <assert.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

template <class T>
void buildTLRMatrixPiece(
    TLR_Matrix *matrix,
    KDTree kdtree,
    T* d_pointCloud,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis,
    T tol) {


        magma_init();

        uint64_t rankSum = 0;
        int totalMem;
        matrix->tileSize = kdtree.maxLeafSize;
        matrix->numTilesInAxis = (upperPowerOfTwo(kdtree.N)/numPiecesInAxis)/matrix->tileSize;
        bool isDiagonal = isPieceDiagonal(pieceMortonIndex);
        unsigned int batchCount = isDiagonal ? matrix->numTilesInAxis - 1: matrix->numTilesInAxis;
        unsigned int maxRank = matrix->tileSize>>1;

        assert(numPiecesInAxis <= matrix->numTilesInAxis);

        printf("batch count: %d  isDiagonal: %d  tile size: %d  numTilesInAxis: %d\n", batchCount, isDiagonal, matrix->tileSize, matrix->numTilesInAxis);

        int *d_rowsBatch, *d_colsBatch, *d_ranksOutput;
        int *d_LDMBatch, *d_LDABatch, *d_LDBBatch;
        T *d_UOutput, *d_VOutput;
        T **d_MPtrs, **d_UOutputPtrs, **d_VOutputPtrs;
        hipMalloc((void**) &d_rowsBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_colsBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_ranksOutput, batchCount*kdtree.numLeaves*sizeof(int));
        hipMalloc((void**) &d_LDMBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_LDABatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_LDBBatch, batchCount*sizeof(int));
        hipMalloc((void**) &d_UOutput, batchCount*matrix->tileSize*maxRank*sizeof(T));
        hipMalloc((void**) &d_VOutput, batchCount*matrix->tileSize*maxRank*sizeof(T));
        hipMalloc((void**) &d_MPtrs, batchCount*sizeof(T*));
        hipMalloc((void**) &d_UOutputPtrs, batchCount*sizeof(T*));
        hipMalloc((void**) &d_VOutputPtrs, batchCount*sizeof(T*));
        fillARAHelpers(
            batchCount, matrix->tileSize, 
            d_rowsBatch, d_colsBatch, 
            d_LDMBatch, d_LDABatch, d_LDBBatch);

        kblasHandle_t kblasHandle;
        kblasRandState_t randState;
        kblasCreate(&kblasHandle);
        kblasInitRandState(kblasHandle, &randState, 1 << 15, 0);
        kblasEnableMagma(kblasHandle);
        kblas_ara_batch_wsquery<T>(kblasHandle, matrix->tileSize, batchCount);
        kblasAllocateWorkspace(kblasHandle);

        T *d_denseTileCol;
        int* d_colScanRanks;
        hipMalloc((void**) &d_denseTileCol, batchCount*matrix->tileSize*matrix->tileSize*sizeof(T));
        hipMalloc((void**) &d_colScanRanks, batchCount*sizeof(int));

        for(unsigned int tileColIdx = 0; tileColIdx < matrix->numTilesInAxis; ++tileColIdx) {
            generateDenseTileCol <T> (
                d_denseTileCol,
                d_pointCloud,
                kdtree,
                tileColIdx,
                matrix->tileSize,
                batchCount,
                pieceMortonIndex, numPiecesInAxis,
                matrix->numTilesInAxis,
                isDiagonal);
        }

        magma_finalize();
}

template void buildTLRMatrixPiece<H2Opus_Real>(
    TLR_Matrix *matrix,
    KDTree kdtree,
    H2Opus_Real* d_dataset,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis,
    H2Opus_Real tol);
