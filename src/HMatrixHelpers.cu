#include "hip/hip_runtime.h"

#include <hipcub/hipcub.hpp>
#include "helperKernels.cuh"
#include "HMatrixHelpers.cuh"

void allocateWeakAdmissibilityStruct(WeakAdmissibility &WAStruct, unsigned int numberOfInputPoints, unsigned int bucketSize) {
    // TODO: parallelize
    WAStruct.numLevels = __builtin_ctz(numberOfInputPoints/bucketSize) + 1;
    WAStruct.numTiles = (int*)malloc((WAStruct.numLevels - 1)*sizeof(int));
    WAStruct.tileIndices = (int**)malloc((WAStruct.numLevels - 1)*sizeof(int*));

    unsigned int dim = 2;
    for(unsigned int level = 0; level < WAStruct.numLevels - 1; ++level) {
        unsigned int numTiles = 1 << (level + 1);
        WAStruct.numTiles[level] = numTiles;
        
        WAStruct.tileIndices[level] = (int*)malloc(numTiles*sizeof(int));
        for(unsigned int j = 0; j < numTiles; ++j) {
            int x;
            if(j%2 == 0) {
                x = 1;
            }
            else {
                x = -1;
            }
            unsigned int tileIndex = j*dim + j + x;
            WAStruct.tileIndices[level][j + x] = CMIndextoMOIndex(dim, tileIndex);
        }
        
        dim <<= 1;
    }
}

void freeWeakAdmissbilityStruct(WeakAdmissibility WAStruct) {
    free(WAStruct.numTiles);
    for(unsigned int i = 0; i < WAStruct.numLevels - 1; ++i) {
        free(WAStruct.tileIndices[i]);
    }
    free(WAStruct.tileIndices);
}

__global__ void fillBatchPtrs(H2Opus_Real **d_UPtrs, H2Opus_Real **d_VPtrs, TLR_Matrix mortonOrderedMatrix, int batchSize, int segmentSize, int batchUnitSize, int* tileIndices, int level) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < batchSize) {
        if(blockIdx.y == 0) {
            d_UPtrs[i] = &mortonOrderedMatrix.U[static_cast<uint64_t>(mortonOrderedMatrix.blockOffsets[tileIndices[i]*batchUnitSize*batchUnitSize])*segmentSize];
        }
        else {
            d_VPtrs[i] = &mortonOrderedMatrix.V[static_cast<uint64_t>(mortonOrderedMatrix.blockOffsets[tileIndices[i]*batchUnitSize*batchUnitSize])*segmentSize];
        }
    }
}

void allocateTilePtrs(int batchSize, int batchUnitSize, int segmentSize, int level, int *tileIndices, LevelTilePtrs &tilePtrs, TLR_Matrix mortonOrderedMatrix) {
    hipMalloc((void**) &tilePtrs.U, batchSize*sizeof(H2Opus_Real*));
    hipMalloc((void**) &tilePtrs.V, batchSize*sizeof(H2Opus_Real*));

    dim3 numThreadsPerBlock(1024);
    dim3 numBlocks((batchSize + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x, 2);
    fillBatchPtrs <<< numBlocks, numThreadsPerBlock >>> (tilePtrs.U, tilePtrs.V, mortonOrderedMatrix, batchSize, segmentSize, batchUnitSize, tileIndices, level);
}

void freeLevelTilePtrs(LevelTilePtrs tilePtrs) {
    hipFree(tilePtrs.U);
    hipFree(tilePtrs.V);
}