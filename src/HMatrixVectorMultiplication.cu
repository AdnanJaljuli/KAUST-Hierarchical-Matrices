#include "hip/hip_runtime.h"

#include "HMatrixVectorMultiplication.cuh"
#include "hipblas.h"
#include "cutlassDiagonalXVector.cuh"
#include "cutlassHMatrixXVector.cuh"
#include "HMatrix.cuh"
#include <assert.h>
#include <stdio.h>

// __global__ void checkErrorInDiagXVec(unsigned int size, H2Opus_Real *result, H2Opus_Real *originalResult, H2Opus_Real* error, H2Opus_Real* tmp) {
//     unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
//     if(i < size) {
//         H2Opus_Real x = originalResult[i];
//         H2Opus_Real y = result[i];
//         printf("x: %lf   y: %lf\n", x, y);
//         atomicAdd(tmp, x*x);
//         atomicAdd(error, (x - y)*(x - y));
//     }
// }

hipError_t HMatrixVecMult(unsigned int numberOfInputPoints, unsigned int bucketSize, unsigned int numSegments, unsigned int vectorWidth, HMatrix hierarchicalMatrix, H2Opus_Real* inpuVectors, H2Opus_Real* resultVectors) {
    
    assert((bucketSize & (bucketSize - 1)) == 0);
    assert((vectorWidth & (vectorWidth - 1)) == 0);
    hipError_t result;
    // multiply diagonal blocks first
    // TODO: replace this with cuBLAS batched gemm
    result = cutlassDiagonalXVec(numberOfInputPoints, bucketSize, numSegments, vectorWidth, hierarchicalMatrix.diagonalBlocks, inpuVectors, resultVectors);
    // TODO: add error checking and return error message if result != success

    // #if EXPAND_MATRIX
    // // run cublas on diagonalXVector and compare answers
    // H2Opus_Real *d_tempResultsVectors;
    // hipMalloc((void**) &d_tempResultsVectors, numberOfInputPoints*vectorWidth*sizeof(H2Opus_Real));
    // hipblasHandle_t handle;
    // double alpha = 1.0f;
    // double beta = 0.0f;

    // hipblasDgemmStridedBatched(handle,
    //     HIPBLAS_OP_N, HIPBLAS_OP_N,
    //     bucketSize, vectorWidth, bucketSize,
    //     &alpha,
    //     hierarchicalMatrix.diagonalBlocks, bucketSize,
    //     bucketSize*bucketSize,
    //     inpuVectors, numberOfInputPoints,
    //     bucketSize,
    //     &beta,
    //     d_tempResultsVectors, numberOfInputPoints,
    //     bucketSize,
    //     numSegments);

    // H2Opus_Real* d_error;
    // H2Opus_Real* d_tmp;
    // hipMalloc((void**) &d_error, sizeof(H2Opus_Real));
    // hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));
    // hipMemset(d_error, 0, sizeof(H2Opus_Real));
    // hipMemset(d_tmp, 0, sizeof(H2Opus_Real));
    
    // unsigned int numThreadsPerBlock = 1024;
    // unsigned int numBlocks = (numberOfInputPoints*vectorWidth + numThreadsPerBlock - 1)/numThreadsPerBlock;
    // printf("matrix size: %d\n", numberOfInputPoints*vectorWidth);
    // hipDeviceSynchronize();
    // checkErrorInDiagXVec <<< numBlocks, numThreadsPerBlock >>> (numberOfInputPoints*vectorWidth, resultVectors, d_tempResultsVectors, d_error, d_tmp);
    
    // H2Opus_Real h_error;
    // H2Opus_Real h_tmp;
    // hipMemcpy(&h_error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    // hipMemcpy(&h_tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    // printf("error in diagXVec: %lf\n", sqrt(h_error)/sqrt(h_tmp));
    // hipFree(d_tmp);
    // hipFree(d_error);
    // #endif

    #if 0
    // multiply rest of hierarchical matrix by the vector
    H2Opus_Real *d_bufferVectors;
    hipMalloc((void**) &d_bufferVectors, numberOfInputPoints*vectorWidth*sizeof(H2Opus_Real));
    result = cutlassHierarchicalXVec(numberOfInputPoints, bucketSize, numSegments, vectorWidth, hierarchicalMatrix, inpuVectors, d_bufferVectors, resultVectors);
    hipFree(d_bufferVectors);
    #endif

    return result;
}