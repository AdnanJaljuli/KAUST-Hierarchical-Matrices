
#include <hip/hip_runtime.h>
#include <algorithm>
#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <typeinfo>
#include <utility>
#include <bits/stdc++.h>
using namespace std;

__global__ void generateSamplingVectors(double *samplingVectors, int size) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < size) {
        unsigned int seed = i;
        hiprandState s;
        hiprand_init(seed, 0, 0, &s);
        samplingVectors[i] = hiprand_uniform(&s);
    }
}

int main() {
    // TODO: read a batch of n*n TLR matrices from a file
    fstream myFile("batchedMatrix.txt", ios_base::in);

    int unitSize, segmentSize, batchSize, rank;
    myFile >> unitSize >> segmentSize >> batchSize;
    printf("%d %d %d\n", unitSize, segmentSize, batchSize);
    int *ranks = (int*)malloc(batchSize*unitSize*unitSize*sizeof(int));
    int rankSum = 0;
    double *U, *V;
    V = (double*)malloc(0);
    U = (double*)malloc(0);

    for(int i = 0; i < batchSize; ++i) {
        for(int j = 0; j < unitSize*unitSize; ++j) {
            int index = i*unitSize*unitSize + j;
            myFile >> ranks[index];
            rankSum += ranks[index];
            U = (double*)realloc(U, rankSum*segmentSize*sizeof(double));
            V = (double*)realloc(V, rankSum*segmentSize*sizeof(double));

            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> U[rankSum - ranks[index] + k];
            }
            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> V[rankSum -ranks[index] + k];
            }
        }
    }
    int *d_ranks;
    double *d_U, *d_V;
    hipMalloc((void**) &d_ranks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_U, rankSum*segmentSize*sizeof(double));
    hipMalloc((void**) &d_V, rankSum*segmentSize*sizeof(double));
    hipMemcpy(d_ranks, ranks, batchSize*unitSize*unitSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);

    // TODO: generate random sampling vectors
    unsigned int samplingVectorsWidth = 16;
    double *d_samplingVectors;
    hipMalloc((void**) &d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (samplingVectorsWidth*batchSize*segmentSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    generateSamplingVectors <<< numBlocks, numThreadsPerBlock >>> (d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize);

    // TODO: launch a kernel that takes as input the TLR matrices, sampling function and multiplies them and stores them in a matrix
    // TODO: launch a kernel that checks the correctness of the multiplication
}