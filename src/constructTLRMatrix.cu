#include "hip/hip_runtime.h"

#include "constructTLRMatrix.cuh"
#include "TLRMatrix.cuh"
#include "TLRMatrixHelpers.cuh"
#include "hipblas.h"
#include "helperFunctions.cuh"
#include "helperKernels.cuh"
#include "kblas.h"
#include "batch_rand.h"

#include <assert.h>

uint64_t createColumnMajorLRMatrix(unsigned int numberOfInputPoints, unsigned int leafSize, unsigned int dimensionOfInputPoints, TLR_Matrix &matrix, KDTree kDTree, H2Opus_Real* &d_dataset, float tolerance, int ARA_R) {

    int maxRank = kDTree.leafSize/2;
    int *d_rowsBatch, *d_colsBatch, *d_ranks;
    int *d_LDMBatch, *d_LDABatch, *d_LDBBatch;
    H2Opus_Real *d_A, *d_B;
    H2Opus_Real **d_MPtrs, **d_APtrs, **d_BPtrs;
    hipMalloc((void**) &d_rowsBatch, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &d_colsBatch, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &d_ranks, (kDTree.numSegments - 1)*kDTree.numSegments*sizeof(int));
    hipMalloc((void**) &d_LDMBatch, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &d_LDABatch, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &d_LDBBatch, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &d_A, (kDTree.numSegments - 1)*kDTree.leafSize*maxRank*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_B, (kDTree.numSegments - 1)*kDTree.leafSize*maxRank*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_MPtrs, (kDTree.numSegments - 1)*sizeof(H2Opus_Real*));
    hipMalloc((void**) &d_APtrs, (kDTree.numSegments - 1)*sizeof(H2Opus_Real*));
    hipMalloc((void**) &d_BPtrs, (kDTree.numSegments - 1)*sizeof(H2Opus_Real*));

    int numThreadsPerBlock = 1024;
    int numBlocks = (kDTree.numSegments - 1 + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillARAArrays <<< numBlocks, numThreadsPerBlock >>> (kDTree.numSegments - 1, kDTree.leafSize, d_rowsBatch, d_colsBatch, d_LDMBatch, d_LDABatch, d_LDBBatch);

    kblasHandle_t kblasHandle;
    kblasRandState_t randState;
    kblasCreate(&kblasHandle);
    kblasInitRandState(kblasHandle, &randState, 1 << 15, 0);
    kblasEnableMagma(kblasHandle);
    kblas_ara_batch_wsquery<H2Opus_Real>(kblasHandle, leafSize, kDTree.numSegments - 1);
    kblasAllocateWorkspace(kblasHandle);

    float ARATotalTime = 0;
    uint64_t rankSum = 0;
    int totalMem;
    int* d_totalMem;
    hipMalloc((void**) &d_totalMem, sizeof(int));

    H2Opus_Real* d_inputMatrixSegmented;
    int* d_scanRanksSegmented;
    hipMalloc((void**) &d_inputMatrixSegmented, kDTree.leafSize*kDTree.leafSize*kDTree.numSegments*sizeof(H2Opus_Real));
    hipMalloc((void**) &d_scanRanksSegmented, (kDTree.numSegments - 1)*sizeof(int));
    hipMalloc((void**) &matrix.blockRanks, kDTree.numSegments*kDTree.numSegments*sizeof(int));
    hipMalloc((void**) &matrix.diagonal, kDTree.numSegments*kDTree.leafSize*kDTree.leafSize*sizeof(H2Opus_Real));
    H2Opus_Real **d_UTiledTemp = (H2Opus_Real**)malloc(kDTree.numSegments*sizeof(H2Opus_Real*));
    H2Opus_Real **d_VTiledTemp = (H2Opus_Real**)malloc(kDTree.numSegments*sizeof(H2Opus_Real*));

    dim3 m_numThreadsPerBlock(min(32, (int)kDTree.leafSize), min(32, (int)kDTree.leafSize));
    dim3 m_numBlocks(1, kDTree.numSegments);

    for(unsigned int segment = 0; segment < kDTree.numSegments; ++segment) {
        generateDenseBlockColumn <<< m_numBlocks, m_numThreadsPerBlock >>> (numberOfInputPoints, kDTree.leafSize, dimensionOfInputPoints, d_inputMatrixSegmented, d_dataset, kDTree, segment, matrix.diagonal);

        generateArrayOfPointersT<H2Opus_Real>(d_inputMatrixSegmented, d_MPtrs, kDTree.leafSize*kDTree.leafSize, kDTree.numSegments - 1, 0);
        generateArrayOfPointersT<H2Opus_Real>(d_A, d_APtrs, kDTree.leafSize*maxRank, kDTree.numSegments - 1, 0);
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_BPtrs, kDTree.leafSize*maxRank, kDTree.numSegments - 1, 0);
        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

        int kblas_ara_return = kblas_ara_batch(
            kblasHandle, d_rowsBatch, d_colsBatch, d_MPtrs, d_LDMBatch, 
            d_APtrs, d_LDABatch, d_BPtrs, d_LDBBatch, d_ranks + segment*(kDTree.numSegments - 1),
            tolerance, kDTree.leafSize, kDTree.leafSize, maxRank, 16, ARA_R, randState, 0, kDTree.numSegments - 1
        );
        assert(kblas_ara_return == 1);

        void* d_tempStorage = NULL;
        size_t tempStorageBytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_tempStorage, tempStorageBytes, d_ranks + segment*(kDTree.numSegments - 1), d_scanRanksSegmented, kDTree.numSegments - 1);
        hipMalloc(&d_tempStorage, tempStorageBytes);
        hipcub::DeviceScan::InclusiveSum(d_tempStorage, tempStorageBytes, d_ranks + segment*(kDTree.numSegments - 1), d_scanRanksSegmented, kDTree.numSegments - 1);
        hipFree(d_tempStorage);

        hipMemcpy(&totalMem, d_scanRanksSegmented + kDTree.numSegments - 2, sizeof(int), hipMemcpyDeviceToHost);

        hipMalloc((void**) &d_UTiledTemp[segment], kDTree.leafSize*totalMem*sizeof(H2Opus_Real));
        hipMalloc((void**) &d_VTiledTemp[segment], kDTree.leafSize*totalMem*sizeof(H2Opus_Real));
        gpuErrchk(hipPeekAtLastError());

        // TODO: optimize thread allocation here OR replace with cudaMemcpys
        int numThreadsPerBlock = kDTree.leafSize;
        int numBlocks = kDTree.numSegments - 1;
        copyTiles <<< numBlocks, numThreadsPerBlock >>> (kDTree.numSegments - 1, kDTree.leafSize, d_ranks + segment*(kDTree.numSegments - 1), d_scanRanksSegmented, d_UTiledTemp[segment], d_A, d_VTiledTemp[segment], d_B, maxRank);
        rankSum += static_cast<uint64_t>(totalMem);
    }

    kblasDestroy(&kblasHandle);
    kblasDestroyRandState(randState);

    hipFree(d_totalMem);
    hipFree(d_inputMatrixSegmented);
    hipFree(d_scanRanksSegmented);
    hipFree(d_rowsBatch);
    hipFree(d_colsBatch);
    hipFree(d_LDMBatch);
    hipFree(d_LDABatch);
    hipFree(d_LDBBatch);
    hipFree(d_MPtrs);
    hipFree(d_APtrs);
    hipFree(d_BPtrs);
    hipFree(d_A);
    hipFree(d_B);

    hipMalloc((void**) &matrix.U, rankSum*kDTree.leafSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &matrix.V, rankSum*kDTree.leafSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &matrix.blockOffsets, kDTree.numSegments*kDTree.numSegments*sizeof(int));

    numThreadsPerBlock = 1024;
    numBlocks = ((kDTree.numSegments - 1)*kDTree.numSegments + numThreadsPerBlock - 1)/numThreadsPerBlock;
    // TODO: no need for this. Instead, replace d_ranks with matrix.blockRanks
    copyRanks <<< numBlocks, numThreadsPerBlock >>> (kDTree.numSegments, kDTree.leafSize, d_ranks, matrix.blockRanks);
    hipFree(d_ranks);

    void *d_tempStorage = NULL;
    size_t tempStorageBytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_tempStorage, tempStorageBytes, matrix.blockRanks, matrix.blockOffsets, kDTree.numSegments*kDTree.numSegments);
    hipMalloc(&d_tempStorage, tempStorageBytes);
    hipcub::DeviceScan::ExclusiveSum(d_tempStorage, tempStorageBytes, matrix.blockRanks, matrix.blockOffsets, kDTree.numSegments*kDTree.numSegments);
    hipFree(d_tempStorage);

    int* h_scanRanks = (int*)malloc(kDTree.numSegments*kDTree.numSegments*sizeof(int));
    hipMemcpy(h_scanRanks, matrix.blockOffsets, kDTree.numSegments*kDTree.numSegments*sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned int segment = 0; segment < kDTree.numSegments - 1; ++segment) {
        hipMemcpy(&matrix.U[static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*segment])*kDTree.leafSize], d_UTiledTemp[segment], static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*(segment + 1)] - h_scanRanks[kDTree.numSegments*segment])*kDTree.leafSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
        hipMemcpy(&matrix.V[static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*segment])*kDTree.leafSize], d_VTiledTemp[segment], static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*(segment + 1)] - h_scanRanks[kDTree.numSegments*segment])*kDTree.leafSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(&matrix.U[static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*(kDTree.numSegments - 1)])*kDTree.leafSize], d_UTiledTemp[kDTree.numSegments - 1], static_cast<uint64_t>(rankSum - h_scanRanks[kDTree.numSegments*(kDTree.numSegments - 1)])*kDTree.leafSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    hipMemcpy(&matrix.V[static_cast<uint64_t>(h_scanRanks[kDTree.numSegments*(kDTree.numSegments - 1)])*kDTree.leafSize], d_VTiledTemp[kDTree.numSegments - 1], static_cast<uint64_t>(rankSum - h_scanRanks[kDTree.numSegments*(kDTree.numSegments - 1)])*kDTree.leafSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    free(h_scanRanks);

    for(unsigned int segment = 0; segment < kDTree.numSegments; ++segment) {
        hipFree(d_UTiledTemp[segment]);
        hipFree(d_VTiledTemp[segment]);
    }
    free(d_UTiledTemp);
    free(d_VTiledTemp);
    
    return rankSum;
}
