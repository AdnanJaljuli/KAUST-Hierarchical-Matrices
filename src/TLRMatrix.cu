
#include "TLRMatrix.cuh"

void freeTLRMatrix(TLR_Matrix *matrix){
    hipFree(matrix->d_tileOffsets);
    matrix->d_U.clear();
    matrix->d_V.clear();
}
