
#include <assert.h>
#include "TLRMatrix.cuh"

void freeMatrix(TLR_Matrix matrix){
    hipFree(matrix.blockRanks);
    hipFree(matrix.blockOffsets);
    hipFree(matrix.U);
    hipFree(matrix.V);
    hipFree(matrix.diagonal);
}
