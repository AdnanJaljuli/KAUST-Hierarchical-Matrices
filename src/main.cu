#include "hip/hip_runtime.h"

#include "config.h"
#include "counters.h"
#include "createLRMatrix.cuh"
#include "helperFunctions.cuh"
#include "hierarchicalMatrixFunctions.cuh"
#include "kblas.h"
#include "kdtreeConstruction.cuh"
#include "tlr_example.h"
#include "TLR_Matrix.h"

#include <algorithm>
#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <typeinfo>
#include <utility>
using namespace std;

int main(int argc, char *argv[]) {

    hipDeviceSynchronize();

    Config config = parseArgs(argc, argv);
    printArgs(config);

    #if USE_COUNTERS
    Counters counters;
    initCounters(&counters);
    startTime(TOTAL_TIME, &counters);
    #endif

    // Generate the points
    H2Opus_Real* d_dataset; // TODO: rename to something more representative
    gpuErrchk(hipMalloc((void**) &d_dataset, config.numberOfInputPoints*config.dimensionOfInputPoints*sizeof(H2Opus_Real)));
    generateDataset(config.numberOfInputPoints, config.dimensionOfInputPoints, d_dataset);

    // Build the KD-tree
    // TODO: consolidate the numSegments and maxNumSegments variables
    uint64_t maxNumSegments = (config.numberOfInputPoints + config.bucketSize - 1)/config.bucketSize;
    printf("max num segments: %d\n", maxNumSegments);
    uint64_t numSegments;
    // TODO: Combine into a struct that represents the KD-tree
    int  *d_valuesIn; // TODO: rename to something more representative
    int  *d_offsetsSort; // TODO: rename to something more representative
    hipMalloc((void**) &d_valuesIn, config.numberOfInputPoints*sizeof(int));
    hipMalloc((void**) &d_offsetsSort, (maxNumSegments + 1)*sizeof(int));
    createKDTree(config.numberOfInputPoints, config.dimensionOfInputPoints, config.bucketSize, &numSegments, config.divMethod, d_valuesIn, d_offsetsSort, d_dataset, maxNumSegments);

    // Build the TLR matrix
    uint64_t maxSegmentSize = config.bucketSize;
    printf("max segment size: %lu\n", maxSegmentSize);
    printf("num segments: %lu\n", numSegments);
    const int ARA_R = 10;
    int max_rows = maxSegmentSize;
    int max_cols = maxSegmentSize;
    int max_rank = max_cols;
    TLR_Matrix matrix;
    matrix.type = COLUMN_MAJOR;
    H2Opus_Real* d_denseMatrix;
    #if EXPAND_MATRIX
    // TODO: assert that this doesn't exceed memory limit
    hipMalloc((void**) &d_denseMatrix, numSegments*numSegments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));
    #endif
    // TODO: separate the code that expands the matrix from the code that doesn't; have a separate call that passes and calculates the dense matrix; this call should only be called if the EXPAND_MATRIX macro is enabled
    uint64_t kSum = createColumnMajorLRMatrix(config.numberOfInputPoints, numSegments, maxSegmentSize, config.bucketSize, config.dimensionOfInputPoints, matrix, d_denseMatrix, d_valuesIn, d_offsetsSort, d_dataset, config.lowestLevelTolerance, ARA_R, max_rows, max_cols, max_rank);
    hipFree(d_dataset);
    hipFree(d_valuesIn);
    hipFree(d_offsetsSort);
    gpuErrchk(hipPeekAtLastError());

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, maxSegmentSize, matrix, d_denseMatrix);
    #endif

    // Convert TLR matrix to morton order
    TLR_Matrix mortonMatrix;
    mortonMatrix.type = MORTON;
    ConvertColumnMajorToMorton(numSegments, maxSegmentSize, kSum, matrix, mortonMatrix); // TODO: Do not capitalize the first letter of function names    
    matrix.cudaFreeMatrix();

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, maxSegmentSize, mortonMatrix, d_denseMatrix);
    #endif

    // Build hierarchical matrix
    // TODO: move declarations not used later inside the function
    #if 0
    const int numLevels = __builtin_ctz(config.numberOfInputPoints/config.bucketSize) + 1;
    printf("numLevels: %d\n", numLevels);
    int** HMatrixExistingRanks = (int**)malloc((numLevels - 1)*sizeof(int*));
    int** HMatrixExistingTiles = (int**)malloc((numLevels - 1)*sizeof(int*));
    genereateHierarchicalMatrix(config.numberOfInputPoints, config.bucketSize, numSegments, maxSegmentSize, numLevels, mortonMatrix, HMatrixExistingRanks, HMatrixExistingTiles);
    #endif
    mortonMatrix.cudaFreeMatrix();
    gpuErrchk(hipPeekAtLastError());

    #if USE_COUNTERS
    endTime(TOTAL_TIME, &counters);
    printCountersInFile(config, &counters);
    #endif

    return 0;

}

