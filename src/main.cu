#include "hip/hip_runtime.h"

#include "config.h"
#include "counters.h"
#include "createLRMatrix.cuh"
#include "helperFunctions.cuh"
#include "hierarchicalMatrixFunctions.cuh"
#include "kblas.h"
#include "kdtreeConstruction.cuh"
#include "tlr_example.h"
#include "TLR_Matrix.h"

#include <algorithm>
#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <typeinfo>
#include <utility>
using namespace std;

// TODO: make EXPAND_MATRIX a config argument (or similar to USE_COUNTERS)
#define EXPAND_MATRIX 1

int main(int argc, char *argv[]) {

    hipDeviceSynchronize();

    Config config = parseArgs(argc, argv);
    printArgs(config, NUM_COUNTERS);

    #if USE_COUNTERS
    Counters counters;
    initCounters(&counters);
    startTime(TOTAL_TIME, &counters);
    #endif

    H2Opus_Real* d_dataset;
    gpuErrchk(hipMalloc((void**) &d_dataset, config.numberOfInputPoints*config.dimensionOfInputPoints*sizeof(H2Opus_Real)));
    generateDataset(config.numberOfInputPoints, config.dimensionOfInputPoints, d_dataset);

    uint64_t maxNumSegments = (config.numberOfInputPoints + config.bucketSize - 1)/config.bucketSize;
    printf("max num segments: %d\n", maxNumSegments);
    uint64_t numSegments;
    int  *d_valuesIn;
    int  *d_offsetsSort;
    hipMalloc((void**) &d_valuesIn, config.numberOfInputPoints*sizeof(int));
    hipMalloc((void**) &d_offsetsSort, (maxNumSegments + 1)*sizeof(int));
    // TODO: move the frees for the two mallocs above to the end of the main function
    createKDTree(config.numberOfInputPoints, config.dimensionOfInputPoints, config.bucketSize, &numSegments, config.divMethod, d_valuesIn, d_offsetsSort, d_dataset, maxNumSegments);

    uint64_t maxSegmentSize = config.bucketSize;
    printf("max segment size: %lu\n", maxSegmentSize);
    printf("num segments: %lu\n", numSegments);

    const int ARA_R = 10;
    int max_rows = maxSegmentSize;
    int max_cols = maxSegmentSize;
    int max_rank = max_cols;
    TLR_Matrix matrix;
    matrix.type = COLUMN_MAJOR;
    H2Opus_Real* d_denseMatrix;
    uint64_t kSum = createColumnMajorLRMatrix(config.numberOfInputPoints, numSegments, maxSegmentSize, config.bucketSize, config.dimensionOfInputPoints, matrix, d_denseMatrix, d_valuesIn, d_offsetsSort, d_dataset, config.lowestLevelTolerance, ARA_R, max_rows, max_cols, max_rank);
    gpuErrchk(hipPeekAtLastError());

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, maxSegmentSize, matrix, d_denseMatrix);
    #endif

    TLR_Matrix mortonMatrix;
    mortonMatrix.type = MORTON;
    ConvertColumnMajorToMorton(numSegments, maxSegmentSize, kSum, matrix, mortonMatrix); // TODO: Do not capitalize the first letter of function names
    
    matrix.cudaFreeMatrix();

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, maxSegmentSize, mortonMatrix, d_denseMatrix);
    #endif

    #if 0
    const int numLevels = __builtin_ctz(config.numberOfInputPoints/config.bucketSize) + 1;
    printf("numLevels: %d\n", numLevels);
    int** HMatrixExistingRanks = (int**)malloc((numLevels - 1)*sizeof(int*));
    int** HMatrixExistingTiles = (int**)malloc((numLevels - 1)*sizeof(int*));
    genereateHierarchicalMatrix(config.numberOfInputPoints, config.bucketSize, numSegments, maxSegmentSize, numLevels, mortonMatrix, HMatrixExistingRanks, HMatrixExistingTiles);
    #endif

    mortonMatrix.cudaFreeMatrix();

    #if USE_COUNTERS
    endTime(TOTAL_TIME, &counters);
    printCountersInFile(config, &counters);
    #endif

    hipFree(d_dataset);
    gpuErrchk(hipPeekAtLastError());
    return 0; // XXX: XXX XXX XXX
}

