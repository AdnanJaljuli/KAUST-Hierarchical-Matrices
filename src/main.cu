#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "TLR_Matrix.cuh"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "config.h"
#include "kdtreeConstruction.cuh"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>
#include <algorithm>
#include <string.h>
#include <stdio.h>
#include "hipblas.h"

#include "kblas.h"
#include "batch_rand.h"
#include "batch_pstrf.h"
#include "batch_block_copy.h"
#include "batch_ara.h"
#include "magma_auxiliary.h"

#define EXPAND_MATRIX 1
#define DENSE_CALC 1
#define BLOCK_SIZE 32
#define PRINT_OUTPUT 0
#define KBLAS_ARA 1
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

// TODO: make sure that non powers of two work
// TODO: create a struct for the tiled matrix that has u_tiled, v_tiled, k, k_scan
// TODO: generate pointcloud and copy values of the pointcloud to ptr on GPU
// TODO: fix makefile so main.cu depends on helperKerlens.cuh
// TODO: make sure that everything that is malloced is freed
// TODO: move generate input matrix to its own file
// TODO: move the expand matrix function to outside the for loop

int main(int argc, char *argv[]){

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice name: %s\n\n", prop.name);

    hipEvent_t startCode, stopCode;
    hipEventCreate(&startCode);
    hipEventCreate(&stopCode);
    hipEventRecord(startCode);

    Config config = parseArgs(argc, argv);
    printf("n: %d\n", config.n);
    printf("bucket size: %d\n", config.bucket_size);
    printf("epsilon: %f\n", config.tol);
    printf("dim: %d\n", config.dim);

    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    timer_arr[0] = (float)config.n;
    timer_arr[1] = (float)config.bucket_size;
    timer_arr[2] = (float)config.dim;
    timer_arr[3] = (float)config.tol;

    hipError_t cudaErr;
    H2Opus_Real *d_dataset;
    gpuErrchk(hipMalloc((void**) &d_dataset, config.n*config.dim*(uint64_t)sizeof(H2Opus_Real)));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (config.n+numThreadsPerBlock-1)/numThreadsPerBlock;
    generateDataset<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, d_dataset);
    hipDeviceSynchronize();

    uint64_t num_segments = 1;
    int max_num_segments;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        max_num_segments = 1<<(getMaxSegmentSize(config.n, config.bucket_size).second);
    } else {
        max_num_segments = (config.n+config.bucket_size-1)/config.bucket_size;
    }

    printf("max num segments: %d\n", max_num_segments);

    int  *d_values_in;
    int  *d_offsets_sort;
    gpuErrchk(hipMalloc((void**) &d_offsets_sort, (max_num_segments + 1)*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_values_in, config.n*sizeof(int)));

    hipEvent_t startKDtree, stopKDtree;
    hipEventCreate(&startKDtree);
    hipEventCreate(&stopKDtree);
    hipEventRecord(startKDtree);
    createKDTree(config.n, config.dim, config.bucket_size, num_segments, config.div_method, d_values_in, d_offsets_sort, d_dataset, max_num_segments);
    hipEventRecord(stopKDtree);
    hipEventSynchronize(stopKDtree);
    hipEventElapsedTime(&timer_arr[4], startKDtree, stopKDtree);
    hipEventDestroy(startKDtree);
    hipEventDestroy(stopKDtree);

    uint64_t maxSegmentSize;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        maxSegmentSize = getMaxSegmentSize(config.n, config.bucket_size).first;
    } else {
        maxSegmentSize = config.bucket_size;
    }
    printf("max segment size: %lu\n", maxSegmentSize);

    H2Opus_Real* d_input_matrix_segmented;

    printf("mem allocated to input matrix: %lu\n", maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    gpuErrchk(hipMalloc((void**) &d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real)));

    int* d_scan_K_segmented;
    gpuErrchk(hipMalloc((void**) &d_scan_K_segmented, num_segments*sizeof(int)));

    H2Opus_Real** d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    H2Opus_Real** d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    TLR_Matrix matrix;
    gpuErrchk(hipMalloc((void**) &matrix.blockRanks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &matrix.diagonal, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));

    printf("ARA begins\n");
    magma_init();

    const int ARA_R = 10;
    const int max_rows = maxSegmentSize;
    const int max_cols = maxSegmentSize;
    const int max_rank = max_cols;

    int *d_rows_batch, *d_cols_batch, *d_ranks;
    int *d_ldm_batch, *d_lda_batch, *d_ldb_batch;
    H2Opus_Real *d_A, *d_B;
    H2Opus_Real** d_M_ptrs, **d_A_ptrs, **d_B_ptrs;

    gpuErrchk(hipMalloc((void**) &d_rows_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_cols_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ranks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldm_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_lda_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_A, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_B, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_M_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_segments*sizeof(H2Opus_Real*)));

    numThreadsPerBlock = 1024;
    numBlocks = (num_segments + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillARAArrays<<<1, 1>>>(num_segments, max_rows, max_cols, d_rows_batch, d_cols_batch, d_ldm_batch, d_lda_batch, d_ldb_batch);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasHandle_t kblas_handle;
    kblasRandState_t rand_state;
    kblasCreate(&kblas_handle);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasInitRandState(kblas_handle, &rand_state, 1<<15, 0);
    gpuErrchk(hipPeekAtLastError());

    kblasEnableMagma(kblas_handle);
    kblas_gesvj_batch_wsquery<H2Opus_Real>(kblas_handle, max_rows, max_cols, num_segments);
    kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle, config.bucket_size, num_segments);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    kblasAllocateWorkspace(kblas_handle);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    
    float ARATotalTime = 0;
    int k_sum = 0;

    #if EXPAND_MATRIX
    H2Opus_Real* d_error;
    H2Opus_Real* error = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_error, sizeof(H2Opus_Real));

    H2Opus_Real* d_tmp;
    H2Opus_Real* tmp = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));

    *error = 0;
    *tmp = 0;
    hipMemcpy(d_error, error, sizeof(H2Opus_Real), hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, tmp, sizeof(H2Opus_Real), hipMemcpyHostToDevice);

    H2Opus_Real* d_expMatrix;
    gpuErrchk(hipMalloc((void**) &d_expMatrix, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));
    #endif

    hipEvent_t startGenerateInputMatrix, stopGenerateInputMatrix;
    hipEventCreate(&startGenerateInputMatrix);
    hipEventCreate(&stopGenerateInputMatrix);
    hipEventRecord(startGenerateInputMatrix);

    dim3 m_numThreadsPerBlock(min(32, (int)maxSegmentSize), min(32, (int)maxSegmentSize));
    dim3 m_numBlocks(1, num_segments);
    
    for(unsigned int segment = 0; segment < num_segments; ++segment){
        generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(config.n, num_segments, maxSegmentSize, config.dim, d_values_in, d_input_matrix_segmented, d_dataset, d_offsets_sort, segment, matrix.diagonal);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        #if 1
        H2Opus_Real* input_matrix_segmented = (H2Opus_Real*)malloc(maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
        hipMemcpy(input_matrix_segmented, d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        char filename[100] = "results/inputmatrix.txt";
        FILE *output_file = fopen(filename, "a");
        for(unsigned int i=0; i<num_segments; ++i){
            for(unsigned int j=0; j<maxSegmentSize; ++j){
                for(unsigned int k=0; k<maxSegmentSize; ++k){
                    fprintf(output_file,"%lf ", input_matrix_segmented[i*maxSegmentSize*maxSegmentSize + k*maxSegmentSize + j]);
                }
                fprintf(output_file, "\n");
            }
            fprintf(output_file, "\n");
        }
        fprintf(output_file, "\n");
        fclose(output_file);
        free(input_matrix_segmented);
        #endif

        int* totalMem = (int*)malloc(sizeof(int));
        generateArrayOfPointersT<H2Opus_Real>(d_input_matrix_segmented, d_M_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
        hipDeviceSynchronize();

        hipEvent_t startARA, stopARA;
        hipEventCreate(&startARA);
        hipEventCreate(&stopARA);
        hipEventRecord(startARA);
        kblas_ara_batch(
                            kblas_handle, d_rows_batch, d_cols_batch, d_M_ptrs, d_ldm_batch, 
                            d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks + segment*num_segments, 
                            config.tol, max_rows, max_cols, max_rank, 32, ARA_R, rand_state, 0, num_segments
                        );
        hipEventRecord(stopARA);
        hipEventSynchronize(stopARA);
        float ARA_time = 0;
        hipEventElapsedTime(&ARA_time, startARA, stopARA);
        ARATotalTime += ARA_time;
        hipEventDestroy(startARA);
        hipEventDestroy(stopARA);
        hipDeviceSynchronize();

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_totalMem);

        #if 1
        printARAOutput<<<1, 1>>>(d_A, d_B, d_ranks + segment*num_segments, num_segments, max_rows, max_rank);
        #endif
        gpuErrchk(hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));
        gpuErrchk(hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));

        numThreadsPerBlock = maxSegmentSize;
        numBlocks = num_segments;
        copyTiles<<<numBlocks, numThreadsPerBlock>>>(num_segments, maxSegmentSize, d_ranks + segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_A, d_V_tiled_temp[segment], d_B);
        hipDeviceSynchronize();

        #if EXPAND_MATRIX
        expandMatrix<<<m_numBlocks, m_numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_ranks + segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_V_tiled_temp[segment], d_expMatrix);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments*maxSegmentSize*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
        calcError<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_expMatrix, d_input_matrix_segmented, d_error, d_tmp);
        hipDeviceSynchronize();
        #endif

        k_sum += (*totalMem);
        free(totalMem);
    }
    // printf("total mem %d\n", k_sum);
    timer_arr[5] = k_sum;
    hipDeviceSynchronize();
    hipEventRecord(stopGenerateInputMatrix);
    hipEventSynchronize(stopGenerateInputMatrix);
    hipEventElapsedTime(&timer_arr[6], startGenerateInputMatrix, stopGenerateInputMatrix);
    hipEventDestroy(startGenerateInputMatrix);
    hipEventDestroy(stopGenerateInputMatrix);

    hipFree(d_scan_K_segmented);
    hipFree(d_values_in);
    hipFree(d_offsets_sort);
    hipFree(d_dataset);

    #if KBLAS_ARA
    hipFree(d_rows_batch);
    hipFree(d_cols_batch);
    hipFree(d_ldm_batch);
    hipFree(d_lda_batch);
    hipFree(d_ldb_batch);
    hipFree(d_M_ptrs);
    hipFree(d_A_ptrs);
    hipFree(d_B_ptrs);
    hipFree(d_A);
    hipFree(d_B);
    #endif

    #if EXPAND_MATRIX
    hipMemcpy(error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipMemcpy(tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipFree(d_error);
    hipFree(d_tmp);
    printf("error: %lf\n", sqrt(*error)/sqrt(*tmp));
    timer_arr[12] = sqrt(*error)/sqrt(*tmp);
    free(tmp);
    free(error);
    hipFree(d_expMatrix);
    #endif

    printf("k sum: %d\n", k_sum);
    gpuErrchk(hipMalloc((void**) &matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.blockOffsets, num_segments*num_segments*sizeof(int)));

    gpuErrchk(hipMemcpy(matrix.blockRanks, d_ranks, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToDevice));
    hipFree(d_ranks);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipDeviceSynchronize();
    hipFree(d_temp_storage);

    int* h_scan_K = (int*)malloc(num_segments*num_segments*sizeof(int));
    gpuErrchk(hipMemcpy(h_scan_K, matrix.blockOffsets, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToHost));

    for(unsigned int segment = 0; segment < num_segments-1; ++segment){
        gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*segment]*maxSegmentSize], d_U_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*segment]*maxSegmentSize], d_V_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    }
    gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_U_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_V_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    free(h_scan_K);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        hipFree(d_U_tiled_temp[segment]);
        hipFree(d_V_tiled_temp[segment]);
    }
    free(d_U_tiled_temp);
    free(d_V_tiled_temp);

    H2Opus_Real* d_buffer_vector;
    H2Opus_Real* d_input_vector;
    H2Opus_Real* d_output_vector;
    H2Opus_Real* d_output_vector_org;
    gpuErrchk(hipMalloc((void**) &d_buffer_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_input_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_output_vector, maxSegmentSize*num_segments*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_output_vector_org, maxSegmentSize*num_segments*sizeof(H2Opus_Real)));

    numThreadsPerBlock = 1024;
    numBlocks = (num_segments*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
    fillVector<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_input_vector, d_output_vector, d_output_vector_org);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    hipEvent_t startGEMV, stopGEMV;
    hipEventCreate(&startGEMV);
    hipEventCreate(&stopGEMV);
    hipEventRecord(startGEMV);
    numThreadsPerBlock = maxSegmentSize;
    numBlocks = num_segments;
    GEMV<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, matrix.blockRanks, matrix.blockOffsets, matrix.U, matrix.V, matrix.diagonal, d_input_vector, d_output_vector, d_buffer_vector);
    hipEventRecord(stopGEMV);
    hipEventSynchronize(stopGEMV);
    gpuErrchk(hipPeekAtLastError());
    hipEventElapsedTime(&timer_arr[7], startGEMV, stopGEMV);
    hipEventDestroy(startGEMV);
    hipEventDestroy(stopGEMV);

    hipFree(d_buffer_vector);
    hipFree(d_input_vector);
    hipFree(d_output_vector);
    hipFree(d_output_vector_org);

    #if DENSE_CALC
    H2Opus_Real* d_denseMatrix;
    H2Opus_Real* d_vector;
    gpuErrchk(hipMalloc((void**) &d_denseMatrix, config.n*config.n*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_vector, config.n*sizeof(H2Opus_Real)));

    numThreadsPerBlock = 1024;
    numBlocks = (config.n + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix);
    filltmpVector<<<numBlocks, numThreadsPerBlock>>>(config.n, d_vector);
    hipDeviceSynchronize();

    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    H2Opus_Real alfa=1, beta=0;
    
    hipEvent_t startDenseGEMV, stopDenseGEMV;
    hipEventCreate(&startDenseGEMV);
    hipEventCreate(&stopDenseGEMV);
    hipEventRecord(startDenseGEMV);
    hipblasDgemv(handle, HIPBLAS_OP_T,
                           config.n, config.n,
                           &alfa,
                           d_denseMatrix, config.n,
                           d_vector, 1,
                           &beta,
                           d_vector, 1);
    hipDeviceSynchronize();
    hipEventRecord(stopDenseGEMV);
    hipEventSynchronize(stopDenseGEMV);
    hipEventElapsedTime(&timer_arr[8], startDenseGEMV, stopDenseGEMV);
    hipEventDestroy(startDenseGEMV);
    hipEventDestroy(stopDenseGEMV);

    hipblasDestroy(handle);
    hipFree(d_denseMatrix);
    hipFree(d_vector);
    #endif

    #if 1
    TLR_Matrix matrix2;
    gpuErrchk(hipMalloc((void**) &matrix2.blockRanks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &matrix2.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix2.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix2.diagonal, maxSegmentSize*maxSegmentSize*num_segments*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix2.blockOffsets, num_segments*num_segments*sizeof(int)));

    hipMemcpy(matrix2.blockRanks, matrix.blockRanks, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.blockOffsets, matrix.blockOffsets, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.U, matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.V, matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);
    hipMemcpy(matrix2.diagonal, matrix.diagonal, maxSegmentSize*maxSegmentSize*num_segments*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice);

    int* d_scan_K_output;

    gpuErrchk(hipMalloc((void**) &d_scan_K_output, num_segments*num_segments*sizeof(unsigned int)));

    H2Opus_Real* d_gemm_matrix_segmented;
    gpuErrchk(hipMalloc((void**) &d_gemm_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real)));

    d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    gpuErrchk(hipMalloc((void**) &d_scan_K_segmented, num_segments*sizeof(int)));
    k_sum = 0;
    ARATotalTime = 0;

    gpuErrchk(hipMalloc((void**) &d_rows_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_cols_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ranks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldm_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_lda_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_A, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_B, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_M_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_segments*sizeof(H2Opus_Real*)));

    numThreadsPerBlock = maxSegmentSize;
    numBlocks = num_segments;
    fillARAArrays<<<1, 1>>>(num_segments, max_rows, max_cols, d_rows_batch, d_cols_batch, d_ldm_batch, d_lda_batch, d_ldb_batch);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    hipEvent_t startGEMM, stopGEMM;
    hipEventCreate(&startGEMM);
    hipEventCreate(&stopGEMM);
    hipEventRecord(startGEMM);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        GEMM<<<m_numBlocks, m_numThreadsPerBlock, 2*config.bucket_size*config.bucket_size*sizeof(H2Opus_Real)>>>(num_segments, maxSegmentSize, matrix.U, matrix.V, matrix.diagonal, matrix.blockRanks, matrix.blockOffsets, matrix2.U, matrix2.V, matrix2.diagonal, matrix2.blockRanks, matrix2.blockOffsets, d_gemm_matrix_segmented, segment, config.bucket_size);
        hipDeviceSynchronize();

        int* totalMem = (int*)malloc(sizeof(int));
        generateArrayOfPointersT<H2Opus_Real>(d_gemm_matrix_segmented, d_M_ptrs, max_rows*max_cols, num_segments, 0);
        generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols, num_segments, 0);
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
        hipDeviceSynchronize();

        hipEvent_t startARA, stopARA;
        hipEventCreate(&startARA);
        hipEventCreate(&stopARA);
        hipEventRecord(startARA);
        kblas_ara_batch(
                            kblas_handle, d_rows_batch, d_cols_batch, d_M_ptrs, d_ldm_batch, 
                            d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks + segment*num_segments, 
                            config.tol, max_rows, max_cols, max_rank, 32, ARA_R, rand_state, 0, num_segments
                        );
        hipEventRecord(stopARA);
        hipEventSynchronize(stopARA);
        float ARA_time = 0;
        hipEventElapsedTime(&ARA_time, startARA, stopARA);
        ARATotalTime += ARA_time;
        hipEventDestroy(startARA);
        hipEventDestroy(stopARA);
        hipDeviceSynchronize();

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        gpuErrchk(hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost));
        hipFree(d_totalMem);

        #if 1
        printARAOutput<<<1, 1>>>(d_A, d_B, d_ranks + segment*num_segments, num_segments, max_rows, max_rank);
        #endif

        gpuErrchk(hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));
        gpuErrchk(hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));

        // copy d_A and d_B to d_U_tiled_segmented
        numThreadsPerBlock = maxSegmentSize;
        numBlocks = num_segments;
        copyTiles<<<numBlocks, numThreadsPerBlock>>>(num_segments, maxSegmentSize, d_ranks+ segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_A, d_V_tiled_temp[segment], d_B);
        hipDeviceSynchronize();

        k_sum += (*totalMem);
        free(totalMem);
    }

    hipDeviceSynchronize();
    hipEventRecord(stopGEMM);
    hipEventSynchronize(stopGEMM);
    float GEMM_time = 0;
    hipEventElapsedTime(&GEMM_time, startGEMM, stopGEMM);
    hipEventDestroy(startGEMM);
    hipEventDestroy(stopGEMM);
    printf("GEMM time: %f\n", GEMM_time);
    timer_arr[9] = GEMM_time;
    magma_finalize();
    // TODO: copy from double pointer array to a single pointer 

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        hipFree(d_U_tiled_temp[segment]);
        hipFree(d_V_tiled_temp[segment]);
    }
    free(d_U_tiled_temp);
    free(d_V_tiled_temp);

    #if DENSE_CALC
    H2Opus_Real* d_denseMatrix1;
    cudaErr = hipMalloc((void**) &d_denseMatrix1, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error dense matrix 1: %s\n", hipGetErrorString(cudaErr)); }
    H2Opus_Real* d_denseMatrix2;
    cudaErr = hipMalloc((void**) &d_denseMatrix2, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error dense matrix 2: %s\n", hipGetErrorString(cudaErr)); }
    H2Opus_Real* d_denseMatrix3;
    cudaErr = hipMalloc((void**) &d_denseMatrix3, config.n*config.n*sizeof(H2Opus_Real));
    if ( cudaErr != hipSuccess ){ printf("CUDA Error dense matrix3: %s\n", hipGetErrorString(cudaErr)); }

    numThreadsPerBlock = 1024;
    numBlocks = (config.n + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix1);
    fillDenseMatrix<<<numBlocks, numThreadsPerBlock>>>(config.n, d_denseMatrix2);
    hipDeviceSynchronize();
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    hipEvent_t startDenseGEMM, stopDenseGEMM;
    hipEventCreate(&startDenseGEMM);
    hipEventCreate(&stopDenseGEMM);
    hipEventRecord(startDenseGEMM);
    hipDeviceSynchronize();
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                           config.n, config.n, config.n,
                           &alfa,
                           d_denseMatrix1, config.n,
                           d_denseMatrix2, config.n,
                           &beta,
                           d_denseMatrix3, config.n);
    hipDeviceSynchronize();
    hipEventRecord(stopDenseGEMM);
    hipEventSynchronize(stopDenseGEMM);
    float DenseGEMM_time = 0;
    hipEventElapsedTime(&DenseGEMM_time, startDenseGEMM, stopDenseGEMM);
    printf("DenseGEMM time: %f\n", DenseGEMM_time);
    timer_arr[10] = DenseGEMM_time;
    hipEventDestroy(startDenseGEMM);
    hipEventDestroy(stopDenseGEMM);

    hipblasDestroy(handle);
    hipFree(d_denseMatrix1);
    hipFree(d_denseMatrix2);
    hipFree(d_denseMatrix3);
    #endif

    hipFree(d_scan_K_segmented);
    hipFree(d_gemm_matrix_segmented);
    
    cudaFreeMatrix(matrix);
    cudaFreeMatrix(matrix2);

    #endif
    hipDeviceSynchronize();
    hipEventRecord(stopCode);
    hipEventSynchronize(stopCode);
    float code_time = 0;
    float Code_time=0;
    hipEventElapsedTime(&Code_time, startCode, stopCode);
    hipEventDestroy(startCode);
    hipEventDestroy(stopCode);
    printf("total time: %f\n", Code_time);
    timer_arr[11] = Code_time;
    printCountersInFile(timer_arr);
    free(timer_arr);
}