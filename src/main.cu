#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "TLR_Matrix.cuh"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "config.h"
#include "kdtreeConstruction.cuh"
#include "magma_auxiliary.h"

#include "kblas.h"
#include "batch_rand.h"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>
#include <algorithm>
#include <string.h>
#include <stdio.h>

#define EXPAND_MATRIX 0
#define BLOCK_SIZE 32
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

int main(int argc, char *argv[]){

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice name: %s\n\n", prop.name);

    hipEvent_t startCode, stopCode;
    hipEventCreate(&startCode);
    hipEventCreate(&stopCode);
    hipEventRecord(startCode);

    Config config = parseArgs(argc, argv);
    printf("n: %d\n", config.n);
    printf("bucket size: %d\n", config.bucket_size);
    printf("epsilon: %f\n", config.tol);
    printf("dim: %d\n", config.dim);

    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    timer_arr[0] = (float)config.n;
    timer_arr[1] = (float)config.bucket_size;
    timer_arr[2] = (float)config.dim;
    timer_arr[3] = (float)config.tol;

    hipError_t cudaErr;
    H2Opus_Real *d_dataset;
    gpuErrchk(hipMalloc((void**) &d_dataset, config.n*config.dim*(uint64_t)sizeof(H2Opus_Real)));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (config.n+numThreadsPerBlock-1)/numThreadsPerBlock;
    generateDataset<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, d_dataset);
    hipDeviceSynchronize();

    uint64_t num_segments = 1;
    int max_num_segments;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        max_num_segments = 1<<(getMaxSegmentSize(config.n, config.bucket_size).second);
    } else {
        max_num_segments = (config.n+config.bucket_size-1)/config.bucket_size;
    }

    printf("max num segments: %d\n", max_num_segments);

    int  *d_values_in;
    int  *d_offsets_sort;
    gpuErrchk(hipMalloc((void**) &d_offsets_sort, (max_num_segments + 1)*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_values_in, config.n*sizeof(int)));

    hipEvent_t startKDtree, stopKDtree;
    hipEventCreate(&startKDtree);
    hipEventCreate(&stopKDtree);
    hipEventRecord(startKDtree);
    createKDTree(config.n, config.dim, config.bucket_size, num_segments, config.div_method, d_values_in, d_offsets_sort, d_dataset, max_num_segments);
    hipEventRecord(stopKDtree);
    hipEventSynchronize(stopKDtree);
    hipEventElapsedTime(&timer_arr[4], startKDtree, stopKDtree);
    hipEventDestroy(startKDtree);
    hipEventDestroy(stopKDtree);

    uint64_t maxSegmentSize;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        maxSegmentSize = getMaxSegmentSize(config.n, config.bucket_size).first;
    } else {
        maxSegmentSize = config.bucket_size;
    }
    printf("max segment size: %lu\n", maxSegmentSize);

    H2Opus_Real* d_input_matrix_segmented;

    printf("mem allocated to input matrix: %lu\n", maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    gpuErrchk(hipMalloc((void**) &d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real)));

    int* d_scan_K_segmented;
    gpuErrchk(hipMalloc((void**) &d_scan_K_segmented, (num_segments-1)*sizeof(int)));

    H2Opus_Real** d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    H2Opus_Real** d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    TLR_Matrix matrix;
    gpuErrchk(hipMalloc((void**) &matrix.blockRanks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &matrix.diagonal, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));

    magma_init();

    const int ARA_R = 10;
    const int max_rows = maxSegmentSize;
    const int max_cols = maxSegmentSize;
    const int max_rank = max_cols;

    int *d_rows_batch, *d_cols_batch, *d_ranks;
    int *d_ldm_batch, *d_lda_batch, *d_ldb_batch;
    H2Opus_Real *d_A, *d_B;
    H2Opus_Real** d_M_ptrs, **d_A_ptrs, **d_B_ptrs;

    // TODO: fix memory allocation. Change num_segments to num_segments-1
    gpuErrchk(hipMalloc((void**) &d_rows_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_cols_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ranks, (num_segments-1)*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldm_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_lda_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_A, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_B, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_M_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_segments*sizeof(H2Opus_Real*)));

    numThreadsPerBlock = 1024;
    numBlocks = ((num_segments-1) + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillARAArrays<<<1, 1>>>(num_segments-1, max_rows, max_cols, d_rows_batch, d_cols_batch, d_ldm_batch, d_lda_batch, d_ldb_batch);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasHandle_t kblas_handle;
    kblasRandState_t rand_state;
    kblasCreate(&kblas_handle);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasInitRandState(kblas_handle, &rand_state, 1<<15, 0);
    gpuErrchk(hipPeekAtLastError());

    kblasEnableMagma(kblas_handle);
    kblas_gesvj_batch_wsquery<H2Opus_Real>(kblas_handle, max_rows, max_cols, num_segments-1);
    kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle, config.bucket_size, num_segments-1);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    kblasAllocateWorkspace(kblas_handle);
    hipDeviceSynchronize();
    
    float ARATotalTime = 0;
    int k_sum = 0;

    #if EXPAND_MATRIX
    H2Opus_Real* d_error;
    H2Opus_Real* error = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_error, sizeof(H2Opus_Real));

    H2Opus_Real* d_tmp;
    H2Opus_Real* tmp = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));

    *error = 0;
    *tmp = 0;
    hipMemcpy(d_error, error, sizeof(H2Opus_Real), hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, tmp, sizeof(H2Opus_Real), hipMemcpyHostToDevice);

    H2Opus_Real* d_expMatrix;
    gpuErrchk(hipMalloc((void**) &d_expMatrix, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));
    #endif

    hipEvent_t startGenerateInputMatrix, stopGenerateInputMatrix;
    hipEventCreate(&startGenerateInputMatrix);
    hipEventCreate(&stopGenerateInputMatrix);
    hipEventRecord(startGenerateInputMatrix);

    dim3 m_numThreadsPerBlock(min(32, (int)maxSegmentSize), min(32, (int)maxSegmentSize));
    dim3 m_numBlocks(1, num_segments);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(config.n, num_segments, maxSegmentSize, config.dim, d_values_in, d_input_matrix_segmented, d_dataset, d_offsets_sort, segment, matrix.diagonal);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        #if 0
        H2Opus_Real* input_matrix_segmented = (H2Opus_Real*)malloc(maxSegmentSize*maxSegmentSize*(num_segments-1)*(uint64_t)sizeof(H2Opus_Real));
        hipMemcpy(input_matrix_segmented, d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*(num_segments-1)*(uint64_t)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        char filename[100] = "results/inputmatrix.txt";
        FILE *output_file = fopen(filename, "a");
        for(unsigned int i=0; i<num_segments-1; ++i){
            for(unsigned int j=0; j<maxSegmentSize; ++j){
                for(unsigned int k=0; k<maxSegmentSize; ++k){
                    // fprintf(output_file,"%lf ", input_matrix_segmented[i*maxSegmentSize*maxSegmentSize + k*maxSegmentSize + j]);
                    printf("%lf ", input_matrix_segmented[i*maxSegmentSize*maxSegmentSize + k*maxSegmentSize + j]);
                }
                printf("\n");
            }
            printf("\n");
        }
        printf("\n");
        fclose(output_file);
        free(input_matrix_segmented);
        #endif

        int* totalMem = (int*)malloc(sizeof(int));
        generateArrayOfPointersT<H2Opus_Real>(d_input_matrix_segmented, d_M_ptrs, max_rows*max_cols, num_segments-1, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols, num_segments-1, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols, num_segments-1, 0);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
        hipDeviceSynchronize();

        hipEvent_t startARA, stopARA;
        hipEventCreate(&startARA);
        hipEventCreate(&stopARA);
        hipEventRecord(startARA);

        int kblas_ara_return = kblas_ara_batch(
            kblas_handle, d_rows_batch, d_cols_batch, d_M_ptrs, d_ldm_batch, 
            d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks + segment*(num_segments-1),
            config.tol, max_rows, max_cols, max_rank, 32, ARA_R, rand_state, 0, num_segments-1
        );
        printf("kblas_ara_return: %d\n", kblas_ara_return);

        hipEventRecord(stopARA);
        hipEventSynchronize(stopARA);
        float ARA_time = 0;
        hipEventElapsedTime(&ARA_time, startARA, stopARA);
        ARATotalTime += ARA_time;
        hipEventDestroy(startARA);
        hipEventDestroy(stopARA);
        hipDeviceSynchronize();

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*(num_segments-1), d_scan_K_segmented, num_segments-1);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*(num_segments-1), d_scan_K_segmented, num_segments-1);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, d_ranks + segment*(num_segments-1), d_scan_K_segmented, num_segments-1);
        hipDeviceSynchronize();
        hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_totalMem);

        #if 1
        printARAOutput<<<1, 1>>>(d_A, d_B, d_ranks + segment*(num_segments-1), num_segments-1, max_rows, max_rank);
        #endif
        gpuErrchk(hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));
        gpuErrchk(hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));

        numThreadsPerBlock = maxSegmentSize;
        numBlocks = num_segments-1;
        copyTiles<<<numBlocks, numThreadsPerBlock>>>(num_segments-1, maxSegmentSize, d_ranks + segment*(num_segments-1), d_scan_K_segmented, d_U_tiled_temp[segment], d_A, d_V_tiled_temp[segment], d_B);
        hipDeviceSynchronize();

        #if EXPAND_MATRIX
        expandMatrix<<<m_numBlocks, m_numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_ranks + segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_V_tiled_temp[segment], d_expMatrix);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments*maxSegmentSize*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
        calcError<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_expMatrix, d_input_matrix_segmented, d_error, d_tmp);
        hipDeviceSynchronize();
        #endif

        k_sum += (*totalMem);
        free(totalMem);
    }
    timer_arr[5] = k_sum;
    hipDeviceSynchronize();
    hipEventRecord(stopGenerateInputMatrix);
    hipEventSynchronize(stopGenerateInputMatrix);
    hipEventElapsedTime(&timer_arr[6], startGenerateInputMatrix, stopGenerateInputMatrix);
    hipEventDestroy(startGenerateInputMatrix);
    hipEventDestroy(stopGenerateInputMatrix);

    hipFree(d_scan_K_segmented);
    hipFree(d_values_in);
    hipFree(d_offsets_sort);
    hipFree(d_dataset);

    hipFree(d_rows_batch);
    hipFree(d_cols_batch);
    hipFree(d_ldm_batch);
    hipFree(d_lda_batch);
    hipFree(d_ldb_batch);
    hipFree(d_M_ptrs);
    hipFree(d_A_ptrs);
    hipFree(d_B_ptrs);
    hipFree(d_A);
    hipFree(d_B);

    #if EXPAND_MATRIX
    hipMemcpy(error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipMemcpy(tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipFree(d_error);
    hipFree(d_tmp);
    printf("error: %lf\n", sqrt(*error)/sqrt(*tmp));
    timer_arr[12] = sqrt(*error)/sqrt(*tmp);
    free(tmp);
    free(error);
    hipFree(d_expMatrix);
    #endif

    printf("k sum: %d\n", k_sum);
    gpuErrchk(hipMalloc((void**) &matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.blockOffsets, num_segments*num_segments*sizeof(int)));

    numThreadsPerBlock = 1024;
    numBlocks = ((num_segments-1)*num_segments + numThreadsPerBlock - 1)/numThreadsPerBlock;
    copyRanks<<<numBlocks, numThreadsPerBlock>>>(num_segments, maxSegmentSize, d_ranks, matrix.blockRanks);
    hipDeviceSynchronize();
    hipFree(d_ranks);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipDeviceSynchronize();
    hipFree(d_temp_storage);
    gpuErrchk(hipPeekAtLastError());

    int* h_scan_K = (int*)malloc(num_segments*num_segments*sizeof(int));
    gpuErrchk(hipMemcpy(h_scan_K, matrix.blockOffsets, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipPeekAtLastError());

    for(unsigned int segment = 0; segment < num_segments-1; ++segment){
        gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*segment]*maxSegmentSize], d_U_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*segment]*maxSegmentSize], d_V_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    }
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_U_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_V_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    free(h_scan_K);
    gpuErrchk(hipPeekAtLastError());

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        hipFree(d_U_tiled_temp[segment]);
        hipFree(d_V_tiled_temp[segment]);
    }
    free(d_U_tiled_temp);
    free(d_V_tiled_temp);
    gpuErrchk(hipPeekAtLastError());

    TLR_Matrix mortonMatrix;
    hipMalloc((void**) &mortonMatrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &mortonMatrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real));
    hipMalloc((void**) &mortonMatrix.blockOffsets, num_segments*num_segments*sizeof(int));
    hipMalloc((void**) &mortonMatrix.blockRanks, num_segments*num_segments*sizeof(int));
    hipMalloc((void**) &mortonMatrix.diagonal, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real));
    ColumnMajorToMorton(num_segments, maxSegmentSize, k_sum, matrix, mortonMatrix);
    gpuErrchk(hipPeekAtLastError());
    printf("k_sum %d\n", k_sum);


    const int num_ops = (num_segments/2)*(num_segments/2) - (num_segments/2);

    int* d_ranks_output;
    gpuErrchk(hipMalloc((void**) &d_ranks_output, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_rows_batch, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_cols_batch, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_lda_batch, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_ops*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_ops*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_A, num_ops*2*max_rows*2*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_B, num_ops*2*max_rows*2*max_rank*sizeof(H2Opus_Real)));

    gpuErrchk(hipPeekAtLastError());
    fillARAArrays_mod<<<1, 1>>>(num_ops, max_rows, max_cols, d_rows_batch, d_cols_batch, d_lda_batch, d_ldb_batch);
    gpuErrchk(hipPeekAtLastError());
    printf("fillARAArrays_mod\n");
    generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols*4, num_ops, 0);
    printf("first\n");
    generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols*4, num_ops, 0);
    printf("second\n");
    gpuErrchk(hipPeekAtLastError());

    kblasHandle_t kblas_handle_2;
    kblasRandState_t rand_state_2;
    kblasCreate(&kblas_handle_2);
    hipDeviceSynchronize();
    printf("kblas handle 2\n");
    gpuErrchk(hipPeekAtLastError());

    int kblasrandstate_value = kblasInitRandState(kblas_handle_2, &rand_state_2, 1<<15, 0);
    printf("kblasinitrandstate: %d\n", kblasrandstate_value);
    gpuErrchk(hipPeekAtLastError());

    kblasEnableMagma(kblas_handle_2);
    kblas_gesvj_batch_wsquery<H2Opus_Real>(kblas_handle_2, 2*max_rows, 2*max_cols, num_ops);
    kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle_2, config.bucket_size, num_ops);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    kblasAllocateWorkspace(kblas_handle_2);
    hipDeviceSynchronize();

    int* d_activeArrays;
    int* d_tmpArray;
    int* d_ranks_1;
    gpuErrchk(hipMalloc((void**) &d_ranks_1, num_ops*4*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_activeArrays, (num_segments*num_segments)*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_tmpArray, (num_segments*num_segments)*sizeof(int)));
    fillActiveArrays<<<1, 1>>>(num_segments, d_activeArrays, d_tmpArray, d_ranks_1, mortonMatrix.blockRanks);
    hipFree(d_tmpArray);

    printf("num ops: %d\n", num_ops);
    gpuErrchk(hipPeekAtLastError());

    int kblas_mod_ans = kblas_ara_batch_mod(kblas_handle_2, d_rows_batch, d_cols_batch, mortonMatrix.U, mortonMatrix.V, mortonMatrix.blockRanks, mortonMatrix.blockOffsets, d_activeArrays,
        d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks_output,
        config.tol, 2*max_rows, 2*max_cols, 2*max_rank, 32, ARA_R, rand_state_2, 0, ((num_segments/2)*(num_segments/2) - (num_segments/2))
    );
    printf("kblas mod ans : %d\n", kblas_mod_ans);
    
    printK<<<1, 1>>>(d_ranks_output, num_ops);
    hipDeviceSynchronize();
    printK<<<1, 1>>>(mortonMatrix.blockRanks, num_segments*num_segments);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    #if 0
    int* d_bit_vector;
    int* d_bit_vector_scan;
    gpuErrchk(hipMalloc((void**) &d_bit_vector, num_ops*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_bit_vector_scan, num_ops*sizeof(int)));

    numThreadsPerBlock = 1024;
    numBlocks = (num_ops + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillBitVector<<<numBlocks, numThreadsPerBlock>>>(num_ops, 32, d_ranks_output, d_ranks_1, d_bit_vector);
    hipDeviceSynchronize();

    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_bit_vector, d_bit_vector_scan, num_ops);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_bit_vector, d_bit_vector_scan, num_ops);
    hipDeviceSynchronize();
    hipFree(d_temp_storage);

    int* d_new_ranks;
    int* d_new_active_tiles;
    hipMalloc((void**) &d_new_ranks, num_ops*sizeof(int));
    hipMalloc((void**) &d_new_active_tiles, num_ops*sizeof(int));

    numThreadsPerBlock = 1024;
    numBlocks = (num_ops + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillNewLevel<<<numBlocks, numThreadsPerBlock>>>(num_ops, d_bit_vector, d_bit_vector_scan, d_ranks_output, d_new_ranks, d_new_active_tiles);
    #endif 

    hipEventRecord(stopCode);
    hipEventSynchronize(stopCode);
    float Code_time=0;
    hipEventElapsedTime(&Code_time, startCode, stopCode);
    hipEventDestroy(startCode);
    hipEventDestroy(stopCode);
    printf("total time: %f\n", Code_time);
    timer_arr[11] = Code_time;
    printCountersInFile(timer_arr);
    free(timer_arr);
}