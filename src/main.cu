
#include "admissibilityFunctions.cuh"
#include "buildTLRMatrixPiece.cuh"
#include "config.h"
#include "counters.h"
#include "generateDataset.cuh"
#include "helperFunctions.cuh"
#include "HMatrix.cuh"
#include "HMatrixStructure.cuh"
#include "kDTree.cuh"
#include "kDTreeHelpers.cuh"
#include "kDTreeConstruction.cuh"
#include "TLRMatrix.cuh"

#include <algorithm>
#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <typeinfo>
#include <utility>
using namespace std;

// TODO: template all functions that deal with H2Opus_Real
int main(int argc, char *argv[]) {

    hipDeviceSynchronize();

    Config config = parseArgs(argc, argv);
    printArgs(config);

    #if USE_COUNTERS
    Counters counters;
    initCounters(&counters);
    startTime(TOTAL_TIME, &counters);
    #endif

    // Generate the points
    #if USE_COUNTERS
    startTime(GENERATE_DATASET, &counters);
    #endif
    H2Opus_Real* d_pointCloud;
    hipMalloc((void**) &d_pointCloud, config.N*config.nDim*sizeof(H2Opus_Real));
    generateDataset(config.N, config.nDim, d_pointCloud);
    #if USE_COUNTERS
    endTime(GENERATE_DATASET, &counters);
    #endif

    #if EXPAND_MATRIX
    printPointCloud(config.N, config.nDim, d_pointCloud);
    #endif

    // Build the KD-tree
    #if USE_COUNTERS
    startTime(KDTREE, &counters);
    #endif
    KDTree kDTree;
    allocateKDTree(
        kDTree,
        config.N,
        config.nDim,
        config.leafSize,
        config.divMethod);

    constructKDTree(
        kDTree,
        d_pointCloud,
        config.divMethod); // TODO: pass a reference to kdtree
    printf("segment size: %lu\n", kDTree.maxLeafSize);
    printf("num segments: %lu\n", kDTree.numLeaves);
    printf("num levels: %d\n", kDTree.numLevels);
    #if USE_COUNTERS
    endTime(KDTREE, &counters);
    #endif

    // create HMatrixStructure
    #if USE_COUNTERS
    startTime(HMATRIX_STRUCTURE, &counters);
    #endif
    HMatrix hierarchicalMatrix;
    allocateHMatrixStructure(&hierarchicalMatrix.matrixStructure, kDTree.numLevels);
    if(config.admissibilityCondition == BOX_CENTER_ADMISSIBILITY) {
        H2Opus_Real eta = 1;
        BBoxCenterAdmissibility <H2Opus_Real> admissibility(eta, kDTree.nDim);
        constructHMatrixStructure<H2Opus_Real>(
            &hierarchicalMatrix.matrixStructure,
            admissibility,
            kDTree,
            kDTree);
    }
    else if(config.admissibilityCondition == WEAK_ADMISSIBILITY) {
        WeakAdmissibility <H2Opus_Real> admissibility;
        constructHMatrixStructure<H2Opus_Real>(
            &hierarchicalMatrix.matrixStructure,
            admissibility,
            kDTree,
            kDTree);
    }
    #if USE_COUNTERS
    endTime(HMATRIX_STRUCTURE, &counters);
    #endif

    #if EXPAND_MATRIX
    printKDTree(config.N, config.nDim, config.divMethod, config.leafSize, kDTree, d_pointCloud);
    printMatrixStructure(hierarchicalMatrix.matrixStructure);
    #endif

    // build TLR piece
    int numPiecesInAxis = 2;
    for(unsigned int piece = 0; piece < numPiecesInAxis*numPiecesInAxis; ++piece) {
        TLR_Matrix TLRMatrix;
        TLRMatrix.ordering = COLUMN_MAJOR;

        buildTLRMatrixPiece <H2Opus_Real> (
            &TLRMatrix,
            kDTree,
            d_pointCloud,
            piece, numPiecesInAxis,
            config.lowestLevelTolerance);

        checkErrorInTLRPiece <H2Opus_Real> (
            TLRMatrix,
            kDTree,
            d_pointCloud,
            piece, numPiecesInAxis);

        freeTLRMatrix(&TLRMatrix);
    }
    freeKDTree(kDTree);

    #if USE_COUNTERS
    endTime(TOTAL_TIME, &counters);
    printCountersInFile(config, &counters);
    #endif

    printf("done :)\n");

    return 0;

}