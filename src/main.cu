#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "TLR_Matrix.cuh"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "config.h"
#include "kdtreeConstruction.cuh"
#include "sampleBatch.cuh"
#include "kblas_ara.cuh"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>
#include <algorithm>
#include <string.h>
#include <stdio.h>

#define EXPAND_MATRIX 1
#define DENSE_CALC 1
#define BLOCK_SIZE 32
#define PRINT_OUTPUT 0
#define KBLAS_ARA 1
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

// TODO: write a GEMM functionce to replace with the kblas_ara_batch one
// TODO: change the ordering of the LR tiles into a morton or z ordering

int main(int argc, char *argv[]){

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice name: %s\n\n", prop.name);

    hipEvent_t startCode, stopCode;
    hipEventCreate(&startCode);
    hipEventCreate(&stopCode);
    hipEventRecord(startCode);

    Config config = parseArgs(argc, argv);
    printf("n: %d\n", config.n);
    printf("bucket size: %d\n", config.bucket_size);
    printf("epsilon: %f\n", config.tol);
    printf("dim: %d\n", config.dim);

    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    timer_arr[0] = (float)config.n;
    timer_arr[1] = (float)config.bucket_size;
    timer_arr[2] = (float)config.dim;
    timer_arr[3] = (float)config.tol;

    hipError_t cudaErr;
    H2Opus_Real *d_dataset;
    gpuErrchk(hipMalloc((void**) &d_dataset, config.n*config.dim*(uint64_t)sizeof(H2Opus_Real)));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (config.n+numThreadsPerBlock-1)/numThreadsPerBlock;
    generateDataset<<<numBlocks, numThreadsPerBlock>>> (config.n, config.dim, d_dataset);
    hipDeviceSynchronize();

    uint64_t num_segments = 1;
    int max_num_segments;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        max_num_segments = 1<<(getMaxSegmentSize(config.n, config.bucket_size).second);
    } else {
        max_num_segments = (config.n+config.bucket_size-1)/config.bucket_size;
    }

    printf("max num segments: %d\n", max_num_segments);

    int  *d_values_in;
    int  *d_offsets_sort;
    gpuErrchk(hipMalloc((void**) &d_offsets_sort, (max_num_segments + 1)*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_values_in, config.n*sizeof(int)));

    hipEvent_t startKDtree, stopKDtree;
    hipEventCreate(&startKDtree);
    hipEventCreate(&stopKDtree);
    hipEventRecord(startKDtree);
    createKDTree(config.n, config.dim, config.bucket_size, num_segments, config.div_method, d_values_in, d_offsets_sort, d_dataset, max_num_segments);
    hipEventRecord(stopKDtree);
    hipEventSynchronize(stopKDtree);
    hipEventElapsedTime(&timer_arr[4], startKDtree, stopKDtree);
    hipEventDestroy(startKDtree);
    hipEventDestroy(stopKDtree);

    uint64_t maxSegmentSize;
    if(config.div_method != POWER_OF_TWO_ON_LEFT){
        maxSegmentSize = getMaxSegmentSize(config.n, config.bucket_size).first;
    } else {
        maxSegmentSize = config.bucket_size;
    }
    printf("max segment size: %lu\n", maxSegmentSize);

    H2Opus_Real* d_input_matrix_segmented;

    printf("mem allocated to input matrix: %lu\n", maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
    gpuErrchk(hipMalloc((void**) &d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real)));

    int* d_scan_K_segmented;
    gpuErrchk(hipMalloc((void**) &d_scan_K_segmented, num_segments*sizeof(int)));

    H2Opus_Real** d_U_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));
    H2Opus_Real** d_V_tiled_temp = (H2Opus_Real**)malloc(num_segments*sizeof(H2Opus_Real*));

    TLR_Matrix matrix;
    gpuErrchk(hipMalloc((void**) &matrix.blockRanks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &matrix.diagonal, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));

    printf("ARA begins\n");
    magma_init();

    const int ARA_R = 10;
    const int max_rows = maxSegmentSize;
    const int max_cols = maxSegmentSize;
    const int max_rank = max_cols;

    int *d_rows_batch, *d_cols_batch, *d_ranks;
    int *d_ldm_batch, *d_lda_batch, *d_ldb_batch;
    H2Opus_Real *d_A, *d_B;
    H2Opus_Real** d_M_ptrs, **d_A_ptrs, **d_B_ptrs;

    gpuErrchk(hipMalloc((void**) &d_rows_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_cols_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ranks, num_segments*num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldm_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_lda_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_segments*sizeof(int)));
    gpuErrchk(hipMalloc((void**) &d_A, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_B, num_segments*max_rows*max_rank*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &d_M_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_segments*sizeof(H2Opus_Real*)));
    gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_segments*sizeof(H2Opus_Real*)));

    numThreadsPerBlock = 1024;
    numBlocks = (num_segments + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillARAArrays<<<1, 1>>>(num_segments, max_rows, max_cols, d_rows_batch, d_cols_batch, d_ldm_batch, d_lda_batch, d_ldb_batch);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasHandle_t kblas_handle;
    kblasRandState_t rand_state;
    kblasCreate(&kblas_handle);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    kblasInitRandState(kblas_handle, &rand_state, 1<<15, 0);
    gpuErrchk(hipPeekAtLastError());

    kblasEnableMagma(kblas_handle);
    kblas_gesvj_batch_wsquery<H2Opus_Real>(kblas_handle, max_rows, max_cols, num_segments);
    kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle, config.bucket_size, num_segments);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    kblasAllocateWorkspace(kblas_handle);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    
    float ARATotalTime = 0;
    int k_sum = 0;

    #if EXPAND_MATRIX
    H2Opus_Real* d_error;
    H2Opus_Real* error = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_error, sizeof(H2Opus_Real));

    H2Opus_Real* d_tmp;
    H2Opus_Real* tmp = (H2Opus_Real*) malloc(sizeof(H2Opus_Real));
    hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));

    *error = 0;
    *tmp = 0;
    hipMemcpy(d_error, error, sizeof(H2Opus_Real), hipMemcpyHostToDevice);
    hipMemcpy(d_tmp, tmp, sizeof(H2Opus_Real), hipMemcpyHostToDevice);

    H2Opus_Real* d_expMatrix;
    gpuErrchk(hipMalloc((void**) &d_expMatrix, num_segments*maxSegmentSize*maxSegmentSize*sizeof(H2Opus_Real)));
    #endif

    hipEvent_t startGenerateInputMatrix, stopGenerateInputMatrix;
    hipEventCreate(&startGenerateInputMatrix);
    hipEventCreate(&stopGenerateInputMatrix);
    hipEventRecord(startGenerateInputMatrix);

    dim3 m_numThreadsPerBlock(min(32, (int)maxSegmentSize), min(32, (int)maxSegmentSize));
    dim3 m_numBlocks(1, num_segments);
    
    for(unsigned int segment = 0; segment < num_segments; ++segment){
        generateInputMatrix<<<m_numBlocks, m_numThreadsPerBlock>>>(config.n, num_segments, maxSegmentSize, config.dim, d_values_in, d_input_matrix_segmented, d_dataset, d_offsets_sort, segment, matrix.diagonal);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        #if 1
        H2Opus_Real* input_matrix_segmented = (H2Opus_Real*)malloc(maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real));
        hipMemcpy(input_matrix_segmented, d_input_matrix_segmented, maxSegmentSize*maxSegmentSize*num_segments*(uint64_t)sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        char filename[100] = "results/inputmatrix.txt";
        FILE *output_file = fopen(filename, "a");
        for(unsigned int i=0; i<num_segments; ++i){
            for(unsigned int j=0; j<maxSegmentSize; ++j){
                for(unsigned int k=0; k<maxSegmentSize; ++k){
                    fprintf(output_file,"%lf ", input_matrix_segmented[i*maxSegmentSize*maxSegmentSize + k*maxSegmentSize + j]);
                }
                fprintf(output_file, "\n");
            }
            fprintf(output_file, "\n");
        }
        fprintf(output_file, "\n");
        fclose(output_file);
        free(input_matrix_segmented);
        #endif

        int* totalMem = (int*)malloc(sizeof(int));
        generateArrayOfPointersT<H2Opus_Real>(d_input_matrix_segmented, d_M_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols, num_segments, 0);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
        hipDeviceSynchronize();

        hipEvent_t startARA, stopARA;
        hipEventCreate(&startARA);
        hipEventCreate(&stopARA);
        hipEventRecord(startARA);

        kblas_ara_batch(
                            kblas_handle, d_rows_batch, d_cols_batch, d_M_ptrs, d_ldm_batch, 
                            d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks + segment*num_segments, 
                            config.tol, max_rows, max_cols, max_rank, 32, ARA_R, rand_state, 0, num_segments
                        );
                        
        hipEventRecord(stopARA);
        hipEventSynchronize(stopARA);
        float ARA_time = 0;
        hipEventElapsedTime(&ARA_time, startARA, stopARA);
        ARATotalTime += ARA_time;
        hipEventDestroy(startARA);
        hipEventDestroy(stopARA);
        hipDeviceSynchronize();

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* d_totalMem;
        hipMalloc((void**) &d_totalMem, sizeof(int));
        getTotalMem<<<1, 1>>> (d_totalMem, d_ranks + segment*num_segments, d_scan_K_segmented, num_segments);
        hipDeviceSynchronize();
        hipMemcpy(totalMem, d_totalMem, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_totalMem);

        #if 1
        printARAOutput<<<1, 1>>>(d_A, d_B, d_ranks + segment*num_segments, num_segments, max_rows, max_rank);
        #endif
        gpuErrchk(hipMalloc((void**) &d_U_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));
        gpuErrchk(hipMalloc((void**) &d_V_tiled_temp[segment], maxSegmentSize*(*totalMem)*sizeof(H2Opus_Real)));

        numThreadsPerBlock = maxSegmentSize;
        numBlocks = num_segments;
        copyTiles<<<numBlocks, numThreadsPerBlock>>>(num_segments, maxSegmentSize, d_ranks + segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_A, d_V_tiled_temp[segment], d_B);
        hipDeviceSynchronize();

        #if EXPAND_MATRIX
        expandMatrix<<<m_numBlocks, m_numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_ranks + segment*num_segments, d_scan_K_segmented, d_U_tiled_temp[segment], d_V_tiled_temp[segment], d_expMatrix);
        hipDeviceSynchronize();

        numThreadsPerBlock = 1024;
        numBlocks = (num_segments*maxSegmentSize*maxSegmentSize + numThreadsPerBlock-1)/numThreadsPerBlock;
        calcError<<<numBlocks, numThreadsPerBlock>>> (num_segments, maxSegmentSize, d_expMatrix, d_input_matrix_segmented, d_error, d_tmp);
        hipDeviceSynchronize();
        #endif

        k_sum += (*totalMem);
        free(totalMem);
    }
    // printf("total mem %d\n", k_sum);
    timer_arr[5] = k_sum;
    hipDeviceSynchronize();
    hipEventRecord(stopGenerateInputMatrix);
    hipEventSynchronize(stopGenerateInputMatrix);
    hipEventElapsedTime(&timer_arr[6], startGenerateInputMatrix, stopGenerateInputMatrix);
    hipEventDestroy(startGenerateInputMatrix);
    hipEventDestroy(stopGenerateInputMatrix);

    hipFree(d_scan_K_segmented);
    hipFree(d_values_in);
    hipFree(d_offsets_sort);
    hipFree(d_dataset);

    #if KBLAS_ARA
    hipFree(d_rows_batch);
    hipFree(d_cols_batch);
    hipFree(d_ldm_batch);
    hipFree(d_lda_batch);
    hipFree(d_ldb_batch);
    hipFree(d_M_ptrs);
    hipFree(d_A_ptrs);
    hipFree(d_B_ptrs);
    hipFree(d_A);
    hipFree(d_B);
    #endif

    #if EXPAND_MATRIX
    hipMemcpy(error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipMemcpy(tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
    hipFree(d_error);
    hipFree(d_tmp);
    printf("error: %lf\n", sqrt(*error)/sqrt(*tmp));
    timer_arr[12] = sqrt(*error)/sqrt(*tmp);
    free(tmp);
    free(error);
    hipFree(d_expMatrix);
    #endif

    printf("k sum: %d\n", k_sum);
    gpuErrchk(hipMalloc((void**) &matrix.U, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.V, k_sum*maxSegmentSize*sizeof(H2Opus_Real)));
    gpuErrchk(hipMalloc((void**) &matrix.blockOffsets, num_segments*num_segments*sizeof(int)));

    gpuErrchk(hipMemcpy(matrix.blockRanks, d_ranks, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToDevice));
    hipFree(d_ranks);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, matrix.blockRanks, matrix.blockOffsets, num_segments*num_segments);
    hipDeviceSynchronize();
    hipFree(d_temp_storage);

    int* h_scan_K = (int*)malloc(num_segments*num_segments*sizeof(int));
    gpuErrchk(hipMemcpy(h_scan_K, matrix.blockOffsets, num_segments*num_segments*sizeof(int), hipMemcpyDeviceToHost));

    for(unsigned int segment = 0; segment < num_segments-1; ++segment){
        gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*segment]*maxSegmentSize], d_U_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*segment]*maxSegmentSize], d_V_tiled_temp[segment], (h_scan_K[num_segments*(segment+1)] - h_scan_K[num_segments*segment])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    }
    gpuErrchk(hipMemcpy(&matrix.U[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_U_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(&matrix.V[h_scan_K[num_segments*(num_segments-1)]*maxSegmentSize], d_V_tiled_temp[num_segments-1], (k_sum - h_scan_K[num_segments*(num_segments-1)])*maxSegmentSize*sizeof(H2Opus_Real), hipMemcpyDeviceToDevice));
    free(h_scan_K);

    for(unsigned int segment = 0; segment < num_segments; ++segment){
        hipFree(d_U_tiled_temp[segment]);
        hipFree(d_V_tiled_temp[segment]);
    }
    free(d_U_tiled_temp);
    free(d_V_tiled_temp);

    hipDeviceSynchronize();
    hipEventRecord(stopCode);
    hipEventSynchronize(stopCode);
    float code_time = 0;
    float Code_time=0;
    hipEventElapsedTime(&Code_time, startCode, stopCode);
    hipEventDestroy(startCode);
    hipEventDestroy(stopCode);
    printf("total time: %f\n", Code_time);
    timer_arr[11] = Code_time;
    printCountersInFile(timer_arr);
    free(timer_arr);
}