#include "hip/hip_runtime.h"
#include "tlr_example.h"
#include "TLR_Matrix.cuh"
#include "helperFunctions.h"
#include "helperKernels.cuh"
#include "config.h"
#include "kdtreeConstruction.cuh"
#include "createLRMatrix.cuh"

#include <iostream>
#include <utility>
#include <time.h>
#include <assert.h>
#include <math.h>
#include <typeinfo>
#include <algorithm>
#include <string.h>
#include <stdio.h>
#include <inttypes.h>

// TODO: make all header files independent
// TODO: make EXPAND_MATRIX a config argument
#define EXPAND_MATRIX 1
#define BLOCK_SIZE 32
using namespace std;

int main(int argc, char *argv[]){

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice name: %s\n\n", prop.name);

    hipEvent_t startCode, stopCode;
    hipEventCreate(&startCode);
    hipEventCreate(&stopCode);
    hipEventRecord(startCode);

    Config config = parseArgs(argc, argv);
    printf("n: %d\n", config.n);
    printf("bucket size: %d\n", config.bucket_size);
    printf("epsilon: %f\n", config.tol);
    printf("dim: %d\n", config.dim);
    float tolerance = config.tol;

    float* timer_arr = (float*)malloc(numTimers*sizeof(float));
    timer_arr[0] = (float)config.n;
    timer_arr[1] = (float)config.bucket_size;
    timer_arr[2] = (float)config.dim;
    timer_arr[3] = (float)config.tol;

    H2Opus_Real *d_dataset;
    generateDataset_h(config.n, config.dim, d_dataset);

    uint64_t numSegments = 1;
    uint64_t max_num_segments = (config.n+config.bucket_size-1)/config.bucket_size;
    printf("max num segments: %d\n", max_num_segments);

    int  *d_values_in;
    int  *d_offsets_sort;
    createKDTree(config.n, config.dim, config.bucket_size, numSegments, config.div_method, d_values_in, d_offsets_sort, d_dataset, max_num_segments);

    uint64_t max_segment_size = config.bucket_size;
    printf("max segment size: %lu\n", max_segment_size);
    printf("num segments: %lu\n", numSegments);

    const int ARA_R = 10;
    int max_rows = max_segment_size;
    int max_cols = max_segment_size;
    int max_rank = max_cols;

    TLR_Matrix matrix;
    matrix.type = COLUMN_MAJOR;
    H2Opus_Real* d_denseMatrix;

    uint64_t k_sum = createColumnMajorLRMatrix(config.n, numSegments, max_segment_size, config.bucket_size, config.dim, matrix, d_denseMatrix, d_values_in, d_offsets_sort, d_dataset, tolerance, ARA_R, max_rows, max_cols, max_rank);
    gpuErrchk(hipPeekAtLastError());

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, max_segment_size, matrix, d_denseMatrix);
    #endif

    TLR_Matrix mortonMatrix;
    mortonMatrix.type = MORTON;
    ConvertColumnMajorToMorton(numSegments, max_segment_size, k_sum, matrix, mortonMatrix);

    #if EXPAND_MATRIX
    checkErrorInLRMatrix(numSegments, max_segment_size, mortonMatrix, d_denseMatrix);
    #endif
    #if 0
    const int num_levels = __builtin_ctz(config.n) - __builtin_ctz(config.bucket_size) + 1;
    printf("num_levels: %d\n", num_levels);
    int** HMatrixRanks = (int**)malloc((num_levels - 1)*sizeof(int*));
    int** HMatrixCandidateTiles = (int**)malloc((num_levels - 1)*sizeof(int*));
    int numCandidateTiles = numSegments*(numSegments-1);

    int *d_rows_batch, *d_cols_batch, *d_ranks;
    int *d_lda_batch, *d_ldb_batch;
    H2Opus_Real *d_A, *d_B;
    H2Opus_Real **d_A_ptrs, **d_B_ptrs;

    hipMalloc((void**) &HMatrixRanks[num_levels - 2], num_existing_tiles*sizeof(int));
    hipMalloc((void**) &HMatrixExistingTiles[num_levels - 2], num_existing_tiles*sizeof(int));

    // TODO: parallelize
    fillFirstLevelExistingArrays<<<1, 1>>>(numSegments, HMatrixExistingTiles[num_levels - 2], HMatrixRanks[num_levels - 2], mortonMatrix.blockRanks);
    unsigned int tile_size = config.bucket_size;
    bool stop = false;

    H2Opus_Real h_error;
    H2Opus_Real h_tmp;
    H2Opus_Real* d_error;
    H2Opus_Real* d_tmp;
    hipMalloc((void**) &d_tmp, sizeof(H2Opus_Real));
    hipMalloc((void**) &d_error, sizeof(H2Opus_Real));


    // TODO: fix the number of iterations.
    for(unsigned int level = num_levels - 1; level > 0; --level){
        // TODO: set hipMalloc and cudaFrees to outside the loop
        int* d_num_ops;
        hipMalloc((void**) &d_num_ops, sizeof(int));
        int num_ops;
        hipMemset(d_num_ops, 0, sizeof(int));
        unsigned int numThreadsPerBlock = 1024;
        unsigned int numBlocks = (num_existing_tiles + numThreadsPerBlock - 1)/numThreadsPerBlock;
        // TODO: instead of using atmoicAdds, let each thread write to a bit vector and then do a reduce
        calcNumOps<<<numBlocks, numThreadsPerBlock>>> (num_existing_tiles, d_num_ops, HMatrixExistingTiles[level - 1]);        
        hipMemcpy(&num_ops, d_num_ops, sizeof(int), hipMemcpyDeviceToHost);
        printf("level: %d   num ops: %d\n", level, num_ops);
        hipFree(d_num_ops);

        int* d_activeTiles;
        int* d_activeRanks;
        hipMalloc((void**) &d_activeTiles, 4*num_ops*sizeof(int));
        hipMalloc((void**) &d_activeRanks, 4*num_ops*sizeof(int));

        // TODO: parallelize
        fillActiveTiles<<<1, 1>>>(num_existing_tiles, d_activeTiles, HMatrixExistingTiles[level - 1], d_activeRanks, HMatrixRanks[level - 1]);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        printK<<<1, 1>>>(d_activeTiles, num_ops*4);

        max_rows <<= 1;
        max_cols <<= 1;
        max_rank <<= 1;
        // tolerance *= 2;
        printf("max rows: %d\n", max_rows);
        printf("tolerance: %f\n", tolerance);

        // TODO: find a tight upper limit and malloc and free before and after the loop
        gpuErrchk(hipMalloc((void**) &d_ranks, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_rows_batch, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_cols_batch, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_lda_batch, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_ldb_batch, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_A_ptrs, num_ops*sizeof(H2Opus_Real*)));
        gpuErrchk(hipMalloc((void**) &d_B_ptrs, num_ops*sizeof(H2Opus_Real*)));
        gpuErrchk(hipMalloc((void**) &d_A, num_ops*max_rows*max_rank*sizeof(H2Opus_Real)));
        gpuErrchk(hipMalloc((void**) &d_B, num_ops*max_rows*max_rank*sizeof(H2Opus_Real)));

        numThreadsPerBlock = 1024;
        numBlocks = (num_ops + numThreadsPerBlock - 1)/numThreadsPerBlock;
        fillLRARAArrays<<<numBlocks, numThreadsPerBlock>>>(num_ops, max_rows, max_cols, d_rows_batch, d_cols_batch, d_lda_batch, d_ldb_batch);

        generateArrayOfPointersT<H2Opus_Real>(d_A, d_A_ptrs, max_rows*max_cols, num_ops, 0);
        generateArrayOfPointersT<H2Opus_Real>(d_B, d_B_ptrs, max_rows*max_cols, num_ops, 0);
        gpuErrchk(hipPeekAtLastError());

        kblasHandle_t kblas_handle_2;
        kblasRandState_t rand_state_2;
        kblasCreate(&kblas_handle_2);

        kblasInitRandState(kblas_handle_2, &rand_state_2, 1<<15, 0);
        gpuErrchk(hipPeekAtLastError());

        kblasEnableMagma(kblas_handle_2);
        kblas_gesvj_batch_wsquery<H2Opus_Real>(kblas_handle_2, max_rows, max_cols, num_ops);
        kblas_ara_batch_wsquery<H2Opus_Real>(kblas_handle_2, config.bucket_size, num_ops);
        kblasAllocateWorkspace(kblas_handle_2);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        // TODO: write a unit test for the lr_kblas_ara_batch function. 
        // TODO: optimize max_cols. max_cols shouldn't be equal to max_rows, instead, its values should depend on the ranks of the tiles
        int lr_ARA_return = lr_kblas_ara_batch(kblas_handle_2, d_rows_batch, d_cols_batch, mortonMatrix.U, mortonMatrix.V, d_activeRanks, mortonMatrix.blockOffsets, d_activeTiles,
            d_A_ptrs, d_lda_batch, d_B_ptrs, d_ldb_batch, d_ranks,
            tolerance, max_rows, max_cols, max_rank, 32, ARA_R, rand_state_2, 0, num_ops
        );
        hipDeviceSynchronize();
        assert(lr_ARA_return == 1);
        gpuErrchk(hipPeekAtLastError());

        // TODO: move this error checking to its own function
        #if EXPAND_MATRIX
        hipDeviceSynchronize();
        H2Opus_Real* expandedHMatrix;
        hipMalloc((void**) &expandedHMatrix, num_ops*max_rows*max_cols*sizeof(H2Opus_Real));
        dim3 hm_numBlocks(2, 2*num_ops);
        dim3 hm_numThreadsPerBlock(32, 32);
        expandHMatrixLevel<<<hm_numBlocks, hm_numThreadsPerBlock>>>(num_ops, 64, 64, d_A, d_B, d_ranks, expandedHMatrix);

        hipMemset(d_error, 0, sizeof(H2Opus_Real));
        hipMemset(d_tmp, 0, sizeof(H2Opus_Real));
        errorInHMatrix<<<hm_numBlocks, hm_numThreadsPerBlock>>>(numSegments, max_segment_size, num_ops, max_rows, max_cols, expandedHMatrix, d_denseMatrix, d_activeTiles, d_error, d_tmp);
        hipDeviceSynchronize();
        hipMemcpy(&h_error, d_error, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        hipMemcpy(&h_tmp, d_tmp, sizeof(H2Opus_Real), hipMemcpyDeviceToHost);
        printf("h matrix error: %lf\n", sqrt(h_error)/sqrt(h_tmp));
        hipFree(expandedHMatrix);
        #endif
        break;

        // TODO: optimize the bit vector: use an array of longs instead.
        int* d_old_bit_vector;
        int* d_new_bit_vector;
        int* d_old_bit_vector_scan;
        int* d_new_bit_vector_scan;
        gpuErrchk(hipMalloc((void**) &d_old_bit_vector, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_new_bit_vector, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_old_bit_vector_scan, num_ops*sizeof(int)));
        gpuErrchk(hipMalloc((void**) &d_new_bit_vector_scan, num_ops*sizeof(int)));

        numThreadsPerBlock = 1024;
        numBlocks = (num_ops + numThreadsPerBlock - 1)/numThreadsPerBlock;
        fillBitVector<<<numBlocks, numThreadsPerBlock>>>(num_ops, tile_size, d_ranks, d_activeRanks, d_new_bit_vector, d_old_bit_vector);
        hipDeviceSynchronize();
        tile_size <<= 1;

        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_old_bit_vector, d_old_bit_vector_scan, num_ops);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_old_bit_vector, d_old_bit_vector_scan, num_ops);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        d_temp_storage = NULL;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_new_bit_vector, d_new_bit_vector_scan, num_ops);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_new_bit_vector, d_new_bit_vector_scan, num_ops);
        hipDeviceSynchronize();
        hipFree(d_temp_storage);

        int* d_newLevelCount;
        int* newLevelCount = (int*)malloc(sizeof(int));
        hipMalloc((void**) &d_newLevelCount, sizeof(int));
        getNewLevelCount<<<1, 1>>>(num_ops, d_new_bit_vector, d_new_bit_vector_scan, d_newLevelCount);
        hipMemcpy(newLevelCount, d_newLevelCount, sizeof(int), hipMemcpyDeviceToHost);
        num_existing_tiles = *newLevelCount;
        printf("new level count %d\n", num_existing_tiles);

        if(*newLevelCount == 0) {
            stop = true;
        }
        else {
            gpuErrchk(hipMalloc((void**) &HMatrixRanks[level - 1], *newLevelCount*sizeof(int)));
            gpuErrchk(hipMalloc((void**) &HMatrixExistingTiles[level - 1], *newLevelCount*sizeof(int)));

            numThreadsPerBlock = 1024;
            numBlocks = (num_ops + numThreadsPerBlock - 1)/numThreadsPerBlock;
            fillNewLevel<<<numBlocks, numThreadsPerBlock>>>(num_ops, d_new_bit_vector, d_new_bit_vector_scan, d_ranks, HMatrixRanks[level - 1], d_activeTiles, HMatrixExistingTiles[level - 1]);
            copyTilesToNewLevel<<<numBlocks, numThreadsPerBlock>>>(num_ops, d_new_bit_vector, mortonMatrix, d_A, d_B, d_ranks, d_activeTiles, max_rows, max_cols);
            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            // TODO: clean previous ranks and active tiles arrays
        }
        kblasDestroy(&kblas_handle_2);
        kblasDestroyRandState(rand_state_2);
        free(newLevelCount);
        hipFree(d_newLevelCount);

        hipFree(d_ranks);
        hipFree(d_rows_batch);
        hipFree(d_cols_batch);
        hipFree(d_lda_batch);
        hipFree(d_ldb_batch);
        hipFree(d_A_ptrs);
        hipFree(d_B_ptrs);
        hipFree(d_A);
        hipFree(d_B);
        if(stop){
            break;
        }
        break;
    }
    // hipEventRecord(stopCode);
    // hipEventSynchronize(stopCode);
    // float Code_time=0;
    // hipEventElapsedTime(&Code_time, startCode, stopCode);
    // hipEventDestroy(startCode);
    // hipEventDestroy(stopCode);
    // printf("total time: %f\n", Code_time);
    // timer_arr[11] = Code_time;
    // printCountersInFile(timer_arr);
    // free(timer_arr);
    #endif
}
