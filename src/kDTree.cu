#include "kDTree.cuh"
#include "boundingBoxes.h"
#include "config.h"
#include "kDTreeHelpers.cuh"

void allocateKDTree(
    KDTree &tree, 
    unsigned int numberOfInputPoints, 
    unsigned int dimensionOfInputPoints, 
    unsigned int bucketSize, 
    DIVISION_METHOD divMethod) {

        tree.numSegments = 1;
        tree.segmentSize = bucketSize;
        int maxNumSegments;
        if(divMethod == FULL_TREE) {
            maxNumSegments = 1<<(getMaxSegmentSize(numberOfInputPoints, bucketSize).second);
        }
        else {
            maxNumSegments = (numberOfInputPoints + bucketSize - 1)/bucketSize;
        }
        
        hipMalloc((void**) &tree.segmentIndices, numberOfInputPoints*sizeof(int)); // TODO: rename to indexMap
        hipMalloc((void**) &tree.segmentOffsets, (maxNumSegments + 1)*sizeof(int));

        allocateKDTreeBoundingBoxes(
            &tree.boundingBoxes,
            numberOfInputPoints,
            bucketSize,
            dimensionOfInputPoints);
}

void freeKDTree(KDTree tree){
    hipFree(tree.segmentIndices);
    hipFree(tree.segmentOffsets);
}