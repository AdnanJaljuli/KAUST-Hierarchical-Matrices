#include "kDTree.cuh"

void allocateKDTree(KDTree &tree, unsigned int numberOfInputPoints, unsigned int bucketSize){
    // tree.numSegments = (numberOfInputPoints + bucketSize - 1)/bucketSize;
    tree.numSegments = 1;
    tree.segmentSize = bucketSize;
    int maxNumSegments = (numberOfInputPoints + bucketSize - 1)/bucketSize;
    hipMalloc((void**) &tree.segmentIndices, numberOfInputPoints*sizeof(int)); // TODO: rename to indexMap
    hipMalloc((void**) &tree.segmentOffsets, (maxNumSegments + 1)*sizeof(int));
}

void freeKDTree(KDTree tree){
    hipFree(tree.segmentIndices);
    hipFree(tree.segmentOffsets);
}