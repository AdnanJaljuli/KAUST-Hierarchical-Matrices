
#include "buildHMatrix.cuh"
#include "HMatrix.cuh"
#include "TLRMatrix.cuh"
#include "precision.h"
#include "kblas.h"
#include "batch_rand.h"

#include <algorithm> 
#include <hipcub/hipcub.hpp>
#include <vector>

// TODO: add TLRtolerance as a parameter. if eps(TLR) == eps(HMatrix) skip finest level, else: do kblas_ara on finest level to produce a finer tile
template <class T>
void buildHMatrixPiece (
    HMatrix <T> hierarchicalMatrix,
    TLR_Matrix TLRMatrix,
    std::vector<unsigned int> maxRanks,
    float lowestLevelTolerance,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis) {

        kblasHandle_t kblasHandle;
        kblasRandState_t randState;
        kblasCreate(&kblasHandle);
        kblasInitRandState(kblasHandle, &randState, 1<<15, 0);
        kblasEnableMagma(kblasHandle);

        // TODO: get rid of maxCols
        int maxRows;
        int maxCols;
        int *d_rowsBatch, *d_colsBatch, *d_ranks;
        int *d_LDABatch, *d_LDBBatch;
        H2Opus_Real *d_A, *d_B;
        H2Opus_Real **d_APtrs, **d_BPtrs;
        // TODO: allocate memory outside the loop
        // TODO: use multiple streams

        for(unsigned int tileLevel = hierarchicalMatrix.structure.numLevels - 2; tileLevel > 0; --tileLevel) {
            assert(pieceLevel <= tileLevel);
            pair<int, int> tilesInPiece = getTilesInPiece(
                hierarchicalMatrix.structure.tileIndices[tileLevel],
                tileLevel,
                pieceMortonIndex, pieceLevel);

            if(tilesInPiece.first != 0) {

                unsigned int batchUnitSize = 1 << (hierarchicalMatrix.structure.numLevels - (tileLevel + 1));
                int* d_tileIndices;
                hipMalloc((void**) &d_tileIndices, tilesInPiece.first*sizeof(int));
                hipMemcpy(
                    d_tileIndices,
                    hierarchicalMatrix.structure.tileIndices[tileLevel - 1][tilesInPiece.second],
                    tilesInPiece.first*sizeof(int),
                    hipMemcpyHostToDevice);

                LevelTilePtrs tilePtrs;
                allocateTilePtrs(
                    batchSize,
                    batchUnitSize,
                    segmentSize,
                    level,
                    d_tileIndices,
                    tilePtrs,
                    mortonOrderedMatrix);
            }
        }
}

template void buildHMatrixPiece <H2Opus_Real> (
    HMatrix <H2Opus_Real> hierarchicalMatrix,
    TLR_Matrix TLRMatrix,
    std::vector<unsigned int> maxRanks,
    float lowestLevelTolerance,
    unsigned int pieceMortonIndex, unsigned int numPiecesInAxis);

