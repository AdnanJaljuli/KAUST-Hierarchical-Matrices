
#include "HMatrixVectorMultiplication.cuh"
#include "hipblas.h"
#include "cutlassDiagonalXVector.cuh"
#include "cutlassHMatrixXVector.cuh"
#include "HMatrix.cuh"
#include <assert.h>
#include <stdio.h>

hipError_t HMatrixVecMult(unsigned int numberOfInputPoints, unsigned int leafSize, unsigned int numSegments, unsigned int vectorWidth, HMatrix hierarchicalMatrix, H2Opus_Real* inpuVectors, H2Opus_Real* resultVectors) {

    assert((leafSize & (leafSize - 1)) == 0);
    assert((vectorWidth & (vectorWidth - 1)) == 0);
    hipError_t result;
    // multiply diagonal blocks first
    // TODO: replace this with cuBLAS batched gemm
    result = cutlassDiagonalXVec(numberOfInputPoints, leafSize, numSegments, vectorWidth, hierarchicalMatrix.diagonalBlocks, inpuVectors, resultVectors);
    // TODO: add error checking and return error message if result != success

    // multiply rest of hierarchical matrix by the vector
    H2Opus_Real *d_bufferVectors;
    hipMalloc((void**) &d_bufferVectors, numberOfInputPoints*vectorWidth*sizeof(H2Opus_Real));
    result = cutlassHierarchicalXVec(numberOfInputPoints, leafSize, numSegments, vectorWidth, hierarchicalMatrix, inpuVectors, d_bufferVectors, resultVectors);
    hipFree(d_bufferVectors);

    return result;
}