#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <typeinfo>
#include <utility>
#include <bits/stdc++.h>
using namespace std;

__global__ void generateSamplingVectors(double *samplingVectors, int size) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < size) {
        unsigned int seed = i;
        hiprandState s;
        hiprand_init(seed, 0, 0, &s);
        samplingVectors[i] = hiprand_uniform(&s);
    }
}

int main() {
    // TODO: read a batch of n*n TLR matrices from a file
    fstream myFile("batchedMatrix.txt", ios_base::in);

    int unitSize, segmentSize, batchSize;
    myFile >> unitSize >> segmentSize >> batchSize;
    printf("%d %d %d\n", unitSize, segmentSize, batchSize);
    int *ranks = (int*)malloc(batchSize*unitSize*unitSize*sizeof(int));
    int rankSum = 0;
    double *U, *V;
    V = (double*)malloc(0);
    U = (double*)malloc(0);

    for(int i = 0; i < batchSize; ++i) {
        for(int j = 0; j < unitSize*unitSize; ++j) {
            int index = i*unitSize*unitSize + j;
            myFile >> ranks[index];
            rankSum += ranks[index];
            U = (double*)realloc(U, rankSum*segmentSize*sizeof(double));
            V = (double*)realloc(V, rankSum*segmentSize*sizeof(double));

            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> U[rankSum - ranks[index] + k];
            }
            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> V[rankSum -ranks[index] + k];
            }
        }
    }
    int *d_ranks, *d_scanRanks;
    double *d_U, *d_V;
    hipMalloc((void**) &d_ranks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_scanRanks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_U, rankSum*segmentSize*sizeof(double));
    hipMalloc((void**) &d_V, rankSum*segmentSize*sizeof(double));
    hipMemcpy(d_ranks, ranks, batchSize*unitSize*unitSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);

    double **d_UBatchPtrs, **d_VBatchPtrs;
    hipMalloc((void**) &d_UBatchPtrs, batchSize*sizeof(double*));
    hipMalloc((void**) &d_VBatchPtrs, batchSize*sizeof(double*));

    fillBatchedPtrs <<< numBlocks, numThreadsPerBlock >>> (d_UBatchPtrs, d_VBatchPtrs, d_U, d_V, batchSize, segmentSize, unitSize, rankSum);

    // TODO: generate random sampling vectors
    unsigned int samplingVectorsWidth = 16;
    double *d_samplingVectors;
    hipMalloc((void**) &d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (samplingVectorsWidth*batchSize*segmentSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    generateSamplingVectors <<< numBlocks, numThreadsPerBlock >>> (d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize);


    // TODO: launch a kernel that takes as input the TLR matrices, sampling function and multiplies them and stores them in a matrix


    // TODO: launch a kernel that checks the correctness of the multiplication
}