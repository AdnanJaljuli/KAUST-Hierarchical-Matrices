#include "hip/hip_runtime.h"

#include "batchedSampling.cuh"
#include <algorithm>
#include <assert.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <typeinfo>
#include <utility>
#include <bits/stdc++.h>
using namespace std;

__global__ void generateSamplingVectors(double *samplingVectors, int size) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < size) {
        unsigned int seed = i;
        hiprandState s;
        hiprand_init(seed, 0, 0, &s);
        // samplingVectors[i] = hiprand_uniform(&s);
        samplingVectors[i] = 1;
    }
}

__global__ void fillBatchedPtrs(double** d_UBatchPtrs, double** d_VBatchPtrs, double* d_U, double* d_V, int* d_scanRanks, int batchSize, int segmentSize, int unitSize) {
    int sumRanks = 0;
    for(int i = 0; i < batchSize; ++i) {
        d_UBatchPtrs[i] = &d_U[sumRanks*segmentSize];
        d_VBatchPtrs[i] = &d_V[sumRanks*segmentSize];
        sumRanks = d_scanRanks[(i + 1)*unitSize*unitSize - 1];
    }
}

__global__ void fillBatchSegments(int *batchSegments, int unitSize, int batchSize) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < batchSize*unitSize*unitSize) {
        batchSegments[i] = i/(unitSize*unitSize);
    }
}

__global__ void printArray(int* output, int size) {
    for(int i = 0; i < size; ++i) {
        printf("%d\n", output[i]);
    }
}

__global__ void printOutput(double* output, int size) {
    for(int i = 0; i < size; ++i) {
        printf("%lf\n", output[i]);
    }
}

__global__ void denseMatrixSampling(int batchSize, int matrixDim, double* denseMatrix, double* denseMatrixOutput, double* samplingVectors, int samplingVectorDim) {
    // unsigned int batch = blockIdx.y/(matrixDim/32);
    // unsigned int blockInBatch = blockIdx.y%(matrixDim/32);
    if(threadIdx.x < samplingVectorDim) {
        double sum = 0;
        for(unsigned int i = 0; i < matrixDim; ++i) {
            sum += denseMatrix[i*matrixDim + blockIdx.y*32 + threadIdx.y]*samplingVectors[threadIdx.x*matrixDim + i];
        }
        denseMatrixOutput[threadIdx.x*matrixDim + blockIdx.y*32 + threadIdx.y] = sum;
    }
}

__global__ void compareResults(double* denseMatrixOutput, double* output, int size, double* error, double* tmp) {
    for(unsigned int i = 0; i < size; ++ i) {
        double x = denseMatrixOutput[i];
        double y = output[i];
        atomicAdd(tmp, x*x);
        atomicAdd(error, (x - y)*(x - y));
    }
}

int main() {
    // read a batch of n*n TLR matrices from a file
    fstream myFile("batchedMatrix.txt", ios_base::in);
    int unitSize, segmentSize, batchSize;
    myFile >> unitSize >> segmentSize >> batchSize;
    printf("%d %d %d\n", unitSize, segmentSize, batchSize);
    int *ranks = (int*)malloc(batchSize*unitSize*unitSize*sizeof(int));
    int rankSum = 0;
    double *U, *V;
    V = (double*)malloc(0);
    U = (double*)malloc(0);

    for(int i = 0; i < batchSize; ++i) {
        for(int j = 0; j < unitSize*unitSize; ++j) {
            int index = i*unitSize*unitSize + j;
            myFile >> ranks[index];
            rankSum += ranks[index];
            U = (double*)realloc(U, rankSum*segmentSize*sizeof(double));
            V = (double*)realloc(V, rankSum*segmentSize*sizeof(double));

            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> U[(rankSum - ranks[index])*segmentSize + k];
            }
            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> V[(rankSum - ranks[index])*segmentSize + k];
            }
        }
    }

    int *d_ranks, *d_scanRanks;
    double *d_U, *d_V;
    hipMalloc((void**) &d_ranks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_scanRanks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_U, rankSum*segmentSize*sizeof(double));
    hipMalloc((void**) &d_V, rankSum*segmentSize*sizeof(double));
    hipMemcpy(d_ranks, ranks, batchSize*unitSize*unitSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);

    int *d_batchSegments;
    hipMalloc((void**) &d_batchSegments, batchSize*unitSize*unitSize*sizeof(int));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (batchSize*unitSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillBatchSegments <<< numBlocks, numThreadsPerBlock >>> (d_batchSegments, unitSize, batchSize);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_batchSegments, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_batchSegments, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);

    double **d_UBatchPtrs, **d_VBatchPtrs;
    hipMalloc((void**) &d_UBatchPtrs, batchSize*sizeof(double*));
    hipMalloc((void**) &d_VBatchPtrs, batchSize*sizeof(double*));

    numBlocks = 1;
    numThreadsPerBlock = 1;
    fillBatchedPtrs <<< numBlocks, numThreadsPerBlock >>> (d_UBatchPtrs, d_VBatchPtrs, d_U, d_V, d_scanRanks, batchSize, segmentSize, unitSize);

    // generate random sampling vectors
    unsigned int samplingVectorsWidth = 32;
    double *d_output;
    double *d_bufferMemory;
    double *d_samplingVectors;
    hipMalloc((void**) &d_output, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    hipMalloc((void**) &d_bufferMemory, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    hipMalloc((void**) &d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    
    numThreadsPerBlock = 1024;
    numBlocks = (samplingVectorsWidth*batchSize*segmentSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    generateSamplingVectors <<< numBlocks, numThreadsPerBlock >>> (d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize);

    // launch a kernel that takes as input the TLR matrices, sampling function and multiplies them and stores them in a matrix
    dim3 m_numThreadsPerBlock(32, 32);
    dim3 m_numBlocks(batchSize*unitSize, 1);
    batchedSampling <<< m_numBlocks, m_numThreadsPerBlock >>> (segmentSize, batchSize, unitSize, d_UBatchPtrs, d_VBatchPtrs, d_scanRanks, d_samplingVectors, samplingVectorsWidth, d_output, d_bufferMemory);

    // read the batched dense tiles form the txt file
    fstream denseMatrixFile("denseMatrix.txt", ios_base::in);
    double* denseMatrix = (double*)malloc(batchSize*unitSize*segmentSize*unitSize*segmentSize*sizeof(double));
    for(unsigned int batch = 0; batch < batchSize; ++batch) {
        for(unsigned int row = 0; row < unitSize*segmentSize; ++row) {
            for(unsigned int col = 0; col < unitSize*segmentSize; ++col) {
                denseMatrixFile >> denseMatrix[batch*unitSize*segmentSize*unitSize*segmentSize + col*unitSize*segmentSize + row];
            }
        }
    }

    double* d_denseMatrix;
    hipMalloc((void**) &d_denseMatrix, batchSize*unitSize*segmentSize*unitSize*segmentSize*sizeof(double));    
    hipMemcpy(d_denseMatrix, denseMatrix, batchSize*unitSize*segmentSize*unitSize*segmentSize*sizeof(double), hipMemcpyHostToDevice);
    double *d_denseMatrixOutput;
    hipMalloc((void**) &d_denseMatrixOutput, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));

    // multiply the dense matrix by the sampling vectors
    dim3 dm_numThreadsPerBlock(samplingVectorsWidth, 32);
    dim3 dm_numBlocks(1, (unitSize*segmentSize)/32);
    denseMatrixSampling <<< dm_numBlocks, dm_numThreadsPerBlock >>> (batchSize, unitSize*segmentSize, d_denseMatrix, d_denseMatrixOutput, d_samplingVectors, samplingVectorsWidth);

    // compare the results
    double *d_error, *d_tmp;
    hipMalloc((void**) &d_error, sizeof(double));
    hipMalloc((void**) &d_tmp, sizeof(double));
    hipMemset(d_error, 0, sizeof(double));
    hipMemset(d_tmp, 0, sizeof(double));
    compareResults <<< 1, 1 >>> (d_denseMatrixOutput, d_output, samplingVectorsWidth*batchSize*segmentSize*unitSize, d_error, d_tmp);
    double h_error;
    double h_tmp;
    hipMemcpy(&h_error, d_error, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_tmp, d_tmp, sizeof(double), hipMemcpyDeviceToHost);
    printf("error in matrix: %lf\n", sqrt(h_error)/sqrt(h_tmp));
    hipFree(d_tmp);
    hipFree(d_error);
    hipDeviceSynchronize();
}