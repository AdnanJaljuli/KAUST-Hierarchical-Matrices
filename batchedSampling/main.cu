#include "hip/hip_runtime.h"

#include "batchedSampling.cuh"
#include <algorithm>
#include <assert.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include <inttypes.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <typeinfo>
#include <utility>
#include <bits/stdc++.h>
using namespace std;

__global__ void generateSamplingVectors(double *samplingVectors, int size) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < size) {
        unsigned int seed = i;
        hiprandState s;
        hiprand_init(seed, 0, 0, &s);
        samplingVectors[i] = hiprand_uniform(&s);
    }
}

__global__ void fillBatchedPtrs(double** d_UBatchPtrs, double** d_VBatchPtrs, double* d_U, double* d_V, int* d_scanRanks, int batchSize, int segmentSize, int unitSize) {
    int sumRanks = 0;
    for(int i = 0; i < batchSize; ++i) {
        d_UBatchPtrs[i] = &d_U[sumRanks*segmentSize];
        d_VBatchPtrs[i] = &d_V[sumRanks*segmentSize];
        sumRanks += d_scanRanks[(i + 1)*unitSize*unitSize - 1];
    }
}

__global__ void fillBatchSegments(int *batchSegments, int unitSize, int batchSize) {
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < batchSize*unitSize*unitSize) {
        batchSegments[i] = i/(unitSize*unitSize);
    }
}

__global__ void printOutput(double* output, int size) {
    for(int i = 0; i < size; ++i) {
        printf("%lf\n", output[i]);
    }
}

int main() {
    // read a batch of n*n TLR matrices from a file
    fstream myFile("batchedMatrix.txt", ios_base::in);
    int unitSize, segmentSize, batchSize;
    myFile >> unitSize >> segmentSize >> batchSize;
    printf("%d %d %d\n", unitSize, segmentSize, batchSize);
    int *ranks = (int*)malloc(batchSize*unitSize*unitSize*sizeof(int));
    int rankSum = 0;
    double *U, *V;
    V = (double*)malloc(0);
    U = (double*)malloc(0);

    for(int i = 0; i < batchSize; ++i) {
        for(int j = 0; j < unitSize*unitSize; ++j) {
            int index = i*unitSize*unitSize + j;
            myFile >> ranks[index];
            rankSum += ranks[index];
            U = (double*)realloc(U, rankSum*segmentSize*sizeof(double));
            V = (double*)realloc(V, rankSum*segmentSize*sizeof(double));

            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> U[(rankSum - ranks[index])*segmentSize + k];
            }
            for(int k = 0; k < ranks[index]*segmentSize; ++k) {
                myFile >> V[(rankSum - ranks[index])*segmentSize + k];
            }
        }
    }

    int *d_ranks, *d_scanRanks;
    double *d_U, *d_V;
    hipMalloc((void**) &d_ranks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_scanRanks, batchSize*unitSize*unitSize*sizeof(int));
    hipMalloc((void**) &d_U, rankSum*segmentSize*sizeof(double));
    hipMalloc((void**) &d_V, rankSum*segmentSize*sizeof(double));
    hipMemcpy(d_ranks, ranks, batchSize*unitSize*unitSize*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_U, U, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, rankSum*segmentSize*sizeof(double), hipMemcpyHostToDevice);

    int *d_batchSegments;
    hipMalloc((void**) &d_batchSegments, batchSize*unitSize*unitSize*sizeof(int));
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (batchSize*unitSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    fillBatchSegments <<< numBlocks, numThreadsPerBlock >>> (d_batchSegments, unitSize, batchSize);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_batchSegments, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSumByKey(d_temp_storage, temp_storage_bytes, d_batchSegments, d_ranks, d_scanRanks, batchSize*unitSize*unitSize);

    double **d_UBatchPtrs, **d_VBatchPtrs;
    hipMalloc((void**) &d_UBatchPtrs, batchSize*sizeof(double*));
    hipMalloc((void**) &d_VBatchPtrs, batchSize*sizeof(double*));

    numBlocks = 1;
    numThreadsPerBlock = 1;
    fillBatchedPtrs <<< numBlocks, numThreadsPerBlock >>> (d_UBatchPtrs, d_VBatchPtrs, d_U, d_V, d_scanRanks, batchSize, segmentSize, unitSize);

    // generate random sampling vectors
    unsigned int samplingVectorsWidth = 16;
    double *d_output;
    double *d_bufferMemory;
    double *d_samplingVectors;
    hipMalloc((void**) &d_output, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    hipMalloc((void**) &d_bufferMemory, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    hipMalloc((void**) &d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    numThreadsPerBlock = 1024;
    numBlocks = (samplingVectorsWidth*batchSize*segmentSize*unitSize + numThreadsPerBlock - 1)/numThreadsPerBlock;
    generateSamplingVectors <<< numBlocks, numThreadsPerBlock >>> (d_samplingVectors, samplingVectorsWidth*batchSize*segmentSize*unitSize);

    // launch a kernel that takes as input the TLR matrices, sampling function and multiplies them and stores them in a matrix
    dim3 m_numThreadsPerBlock(32, 32);
    dim3 m_numBlocks(batchSize*unitSize, 1);
    batchedSampling <<< m_numBlocks, m_numThreadsPerBlock >>> (segmentSize, batchSize, unitSize, d_UBatchPtrs, d_VBatchPtrs, d_scanRanks, d_samplingVectors, samplingVectorsWidth, d_output, d_bufferMemory);

    hipDeviceSynchronize();
    // printOutput <<< 1, 1 >>> (d_output, samplingVectorsWidth*batchSize*segmentSize*unitSize);
    double* output = (double*)malloc(samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double));
    hipMemcpy(output, d_output, samplingVectorsWidth*batchSize*segmentSize*unitSize*sizeof(double), hipMemcpyDeviceToHost);
    char fileName[100] = "output.txt";
    FILE *outputFile = fopen(fileName, "w");
    for(int i = 0; i < samplingVectorsWidth*batchSize*segmentSize*unitSize; ++i) {
        fprintf(outputFile, "%lf\n", output[i]);
    }
    printf("done\n");

    // TODO: launch a kernel that checks the correctness of the multiplication

}